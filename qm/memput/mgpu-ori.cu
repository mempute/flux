#include "hip/hip_runtime.h"

#include "mgpu.h"
#include "matrix.h"
#include <stdio.h>

__device__ intt didx2offset(intt ndim, intt *srank, intt *idx)
{
	intt off = 0, j = 0;

	for(intt i = 1;i < ndim; i++) {
		off += MRANK_SIZE(srank, i) * *(idx + j++);
	}
	off += *(idx + j);

	return off;
}
__device__ void doffset2idx2(intt out_axis[], intt n_preout_axis, intt rdim, intt rrank[], intt off, intt pidx[], intt sidx[])
{
	intt j = 0, k;

	for(intt i = 1;i < rdim; i++, j++) {//�ش� �ε����� �ϳ� �Ʒ� ��ũ�� ������� ���� ���̹Ƿ� i�� 1����
		k = out_axis[j];
		if(j < n_preout_axis) {
			if(rrank[j] < 0) pidx[k] = 0;
			else {
				pidx[k] = off / MRANK_SIZE(rrank, i);
				off %= MRANK_SIZE(rrank, i);
			}
		} else {
			if(rrank[j] < 0) sidx[k] = 0;
			else {
				sidx[k] = off / MRANK_SIZE(rrank, i);
				off %= MRANK_SIZE(rrank, i);
			}
		}
	}
	sidx[out_axis[j]] = off;//������ �ε����� suf matrix�� ���� �ǰ� ���� �ɼ��� �ȴ�.
}
__device__ intt dsparse_idx2offset(intt ndim, intt *srank, intt *idx, intt *axis)
{
	intt off = 0, j = 0;

	for(intt i = 1;i < ndim; i++) {
		off += MRANK_SIZE(srank, i) * *(idx + *(axis + j++));
	}
	off += *(idx + *(axis + j));

	return off;
}
__device__ void doffset2idx(intt ndim, intt *srank, intt off, intt *idx)
{
	intt j = 0;

	for(intt i = 1;i < ndim; i++) {
		if(*(srank + j) < 0) *(idx + j++) = 0;
		else {
			*(idx + j++) = off / MRANK_SIZE(srank, i);
			off %= MRANK_SIZE(srank, i);
		}
	}
	*(idx + j) = off;
}
__device__ intt dmoff2soff(intt mdim, intt *mrank, intt sdim, intt *srank, intt moff, intt *idx)
{
	intt i = mdim - 1, j = sdim - 1;

	doffset2idx(mdim, mrank, moff, idx);
	for(;j >= 0; i--, j--) {
		if(*(srank + j) < 0) *(idx + i) = 0;
	}
	return didx2offset(sdim, srank, idx + ++i);
}
__device__ void dlead_offset2idx(intt nbro, intt cdim, intt ndim, intt *srank, intt off, intt cidx[])
{
	if(nbro) {
		intt i = 0;
		for(;i < cdim - ndim; i++) cidx[i] = 0;
		doffset2idx(ndim, srank, off, &cidx[i]);
	} else cidx[0] = off;

	cidx[MX_DIM - 1] = 0;//bro_offset�� ��)���� ����üũ�� ���
}
__device__ intt dbro_offset(intt nbro, intt bro_dim[], intt bro_idx[], intt cdim, intt *crank, intt cidx[])
{
	intt i = nbro - 1, off;

	if(cidx[MX_DIM - 1] == 1) return -1;//��.

	if(nbro) {
		off = didx2offset(cdim, crank, cidx);

		for(;i >= 0; i--) {
			if(++cidx[bro_idx[i]] == bro_dim[i]) cidx[bro_idx[i]] = 0;
			else break;
		}
		if(i < 0) cidx[MX_DIM - 1] = 1;
		return off;
	} else {
		cidx[MX_DIM - 1] = 1;
		return cidx[0];
	}
}
/*__device__ intt count_over_axis(intt idx[], intt axis)
{
	if(axis == 0) return -1;

	intt n = 1;
	for(intt i = 0;i < axis; i++) n *= idx[i];
	return n;
}*/
template<typename T>
__global__ void ksplit_f(void *pcxt, T *m_split, intt pdim, intt sdim, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt *prank;
	T *rsplit_mdev, *rsplit_mdevs;
	ConcatVar *ccv = (ConcatVar *)pcxt;

	prank = P_LINK_VAR2(intt, pcxt, ccv->szRankPrimary);
	rsplit_mdevs = P_LINK_VAR2(T, pcxt, ccv->mptrDevSecondary);
	intt outer_sz = MRANK_SIZE(prank, axis), inner_sz = (axis == pdim - 1 ? 1 : MRANK_SIZE(prank, axis + 1)) * nstep;
	intt si, soff;

	soff = roff % outer_sz;//split�� ��ũ���� ���� �ɼ�
	si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
	soff = (roff / outer_sz) * inner_sz + soff % inner_sz;//���ҹ�°���� �ɼ�
	rsplit_mdev = *((T **)rsplit_mdevs + si);//�̹� ���� �޸�

	if(bw) *(rsplit_mdev + soff) += *(m_split + roff);
	else *(rsplit_mdev + soff) = *(m_split + roff);
	//printf("(%p) %d %d %f %f\n", rsplit_mdev, roff, soff, *(rsplit_mdev + soff), *(m_split + roff));
}

template<typename T>
intt gsplit_t(void *pcxt, T *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rsize > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rsize);

	ksplit_f<T> << <grid, block >> > (pcxt, m_split, pdim, sdim, idx_origin, idx_width, nsplit, nstep, axis, bw, n);
	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gsplit_f(void *pcxt, floatt *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw)
{
	return gsplit_t<floatt>(pcxt, m_split, pdim, sdim, rsize, idx_origin, idx_width, nsplit, nstep, axis, bw);
}
intt gsplit_f(void *pcxt, intt *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width, 
	intt nsplit, intt nstep, intt axis, bool bw)
{
	return gsplit_t<intt>(pcxt, m_split, pdim, sdim, rsize, idx_origin, idx_width, nsplit, nstep, axis, bw);
}

template<typename T>
__global__ void kconcat_f(void *pcxt, T *m_rcat, intt pdim, intt sdim, intt idx_origin, intt idx_width, 
	intt ncat, intt nstep, intt axis, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt *prank;
	T *pcat_mdev, *pcat_mdevs;
	ConcatVar *ccv = (ConcatVar *)pcxt;

	prank = P_LINK_VAR2(intt, pcxt, ccv->szRankPrimary);
	pcat_mdevs = P_LINK_VAR2(T, pcxt, ccv->mptrDevSecondary);
	intt outer_sz = MRANK_SIZE(prank, axis), inner_sz = (axis == pdim - 1 ? 1 : MRANK_SIZE(prank, axis + 1)) * nstep;
	intt si, soff;

	soff = roff % outer_sz;//split�� ��ũ���� ���� �ɼ�
	si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
	soff = (roff / outer_sz) * inner_sz + soff % inner_sz;//���ҹ�°���� �ɼ�
	pcat_mdev = *((T **)pcat_mdevs + si);

	if(bw) *(m_rcat + roff) += *(pcat_mdev + soff);
	else *(m_rcat + roff) = *(pcat_mdev + soff);
	//printf("[%d](%p) %d %d %d %d %f %f\n", roff, pcat_mdev, i, is, idx[axis], cat_off, *(pcat_mdev + cat_off), *(m_rcat + roff));
}
template<typename T>
intt gconcat_t(void *pcxt, T *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width, 
	intt ncat, intt nstep, intt axis, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rsize > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rsize);

	kconcat_f<T> << <grid, block >> > (pcxt, m_rcat, pdim, sdim, idx_origin, idx_width, ncat, nstep, axis, bw, n);
	hipDeviceSynchronize();

	return n - idx_origin * idx_width;
}
intt gconcat_f(void *pcxt, floatt *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt ncat, intt nstep, intt axis, bool bw)
{
	return gconcat_t<floatt>(pcxt, m_rcat, pdim, sdim, rsize, idx_origin, idx_width, ncat, nstep, axis, bw);
}
intt gconcat_f(void *pcxt, intt *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt ncat, intt nstep, intt axis, bool bw)
{
	return gconcat_t<intt>(pcxt, m_rcat, pdim, sdim, rsize, idx_origin, idx_width, ncat, nstep, axis, bw);
}
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt range, intt n)
{//������� ���� ���� ��� ���� ������ ����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x * range + threadIdx.x * range;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;
	if(n > roff + range) n = roff + range;
	
	intt *out_rank = dotv->outRank, nout = dotv->noutRank;
	intt njo_pre = dotv->njoPre, njo_suf = dotv->njoSuf;
	const bool bw_get_ori = dotv->bwGetOri, jdim_equal = dotv->jdimEqual;
	SparseRank *spr_out = dotv->sprPreOut, *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], ret_idx[MX_DIM], i;
	register T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx(nout, out_rank, roff, ret_idx);
	for(i = 0;i < nout; i++) {
		if(spr_out[i].rkPref) m_pdot += (ret_idx[i] * spr_out[i].rksz);
		else m_sdot += (ret_idx[i] * spr_out[i].rksz);
	}
	if(jdim_equal) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < njo_pre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < njo_pre; i++) pj_idx[i] = 0;
		for(i = 0;i < njo_suf; i++) sj_idx[i] = 0;
	}
	nout--;
	const intt njo_pre2 = njo_pre - 1, njo_suf2 = njo_suf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	const intt podim = spr_out[nout].rkdim, posz = spr_out[nout].rksz;
	bool lastout_is_pref = spr_out[nout].rkPref;
	intt i_po = ret_idx[nout], i_pj = 0, i_sj = 0;
	
	for(;roff < n; roff++) {
		for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%f %f\n", *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(jdim_equal) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J0:;					m_pdot -= spr_pre_jo[i].rktsz;
						m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {//�߰� ���� ����
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J0;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								m_sdot += spr_suf_jo[i].rksz;
								break;
							}
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J1:;					m_pdot -= spr_pre_jo[i].rktsz;
						if(--i < 0) break;
						else {
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J1;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						i_sj = 0;
						i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {
							if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
								sj_idx[i] = 0;
								goto J2;
							} else {
								m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(bw_get_ori) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
		for(;;) {//��� ���� �ε��� ����(pre�� suf�� ��� �� �����Ͽ�)
			if(podim == ++i_po) {//podim�� pre�� suf�� ���� �������� �� ������ ��ġ�ϴ� ���� �����, i_po, posz, lastout_is_pref�� ��������)
				i_po = 0;
				i = nout;
O2:				if(spr_out[i].rkPref) m_pdot -= spr_out[i].rktsz;
				else m_sdot -= spr_out[i].rktsz;
				if(--i < 0) goto LB2;
				else {
					if(spr_out[i].rkdim == ++ret_idx[i]) {
						ret_idx[i] = 0;
						goto O2;
					} else {
						if(spr_out[i].rkPref) m_pdot += spr_out[i].rksz;//���� ���� ����(����)�� ����
						else m_sdot += spr_out[i].rksz;
						break;//suf out �߰� ���� ���� ����
					}
				}
			} else {
				if(lastout_is_pref) m_pdot += posz;
				else m_sdot += posz;
				break;
			}
		}
	}
LB2:;
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
			intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));//�� dot�� BLOCK_SIZE �̴� idx_width�̴� 512�� �Ѿ�� �� ���� ���� ������ ������
	intt range = idx_width / block.x;//512�� �Ѿ�� ���� �޸�(����) ����� �Ѱ� �ʰ��Ǿ� ������� �ʴ´�. 
	dim3 grid(idx_width % (block.x * range) ? 2 : 1);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, range, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//�������� one thread one out����, ���� grid����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], i;
	register T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx(dotv->noutRank, dotv->outRank, roff, pj_idx);
	for(i = 0;i < dotv->noutRank; i++) {
		if(dotv->sprPreOut[i].rkPref) m_pdot += (pj_idx[i] * dotv->sprPreOut[i].rksz);
		else m_sdot += (pj_idx[i] * dotv->sprPreOut[i].rksz);
	}
	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt i_pj = 0, i_sj = 0;

	for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
		//printf("%f %f\n", *m_pdot, *m_sdot);
		sum += *m_pdot * *m_sdot;
		if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J0:;				m_pdot -= spr_pre_jo[i].rktsz;
					m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {//�߰� ���� ����
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J0;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							m_sdot += spr_suf_jo[i].rksz;
							break;
						}
					}
				} else {//���� ���� ����
					m_pdot += pjsz;//�� ����(����)�� ����
					m_sdot += sjsz;
					break;
				}
			}
		} else {
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J1:;				m_pdot -= spr_pre_jo[i].rktsz;
					if(--i < 0) break;
					else {
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J1;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_pdot += pjsz;//�� ����(����)�� ����
					break;
				}
			}
			for(;;) {//suf �������� �ε��� ����
				if(sjdim == ++i_sj) {
					i_sj = 0;
					i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {
						if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
							sj_idx[i] = 0;
							goto J2;
						} else {
							m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_sdot += sjsz;
					break;
				}
			}
		}
	}
LB1:;
	if(dotv->bwGetOri) {//������
		if(rplus != 1) *(m_rdot + roff) *= rplus;
		*(m_rdot + roff) += sum;
	} else {
		if(rplus == 0) *(m_rdot + roff) = 0;
		else *(m_rdot + roff) *= rplus;
		*(m_rdot + roff) += sum;
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//���� �� �޸� ���� ������ ����: ȿ������, ���߿� Ȥ�� ����ҷ��� jdimEqual�ƴҶ� ��� Ʋ���Ƿ� jdimEqual�϶��� 
	//�������� �����ϰ� jdimEqual�ƴҶ��� ���� ��ȷ���ϰ� ���� �ڵ� ����, ����� ������ �׽�Ʈ ������.
	//__shared__ T cache_bank[SM_SIZE];
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], tmp_idx[MX_DIM], i, i_pj, i_sj;
	bool pover = 0, sover = 0;
	register T sum;

	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	T *m_pdot, *m_sdot;
	for(;roff < n; roff += blockDim.x) {
		//__syncthreads();
		_offset2idx(dotv->noutRank, dotv->outRank, roff, tmp_idx);
		for(i = 0, m_pdot = _m_pdot, m_sdot = _m_sdot;i < dotv->noutRank; i++) {
			if(dotv->sprPreOut[i].rkPref) m_pdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
			else m_sdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
		}
		i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
		if(i_pj == 0) pover = 1;
		if(dotv->jdimEqual) i_sj = i_pj;
		else {
			i_sj = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
			if(i_sj == 0) sover = 1;
		}
		m_pdot += i_pj * pjsz;
		m_sdot += i_sj * sjsz;
		for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//if(threadIdx.x == 2) printf("%d %d %d %f %f\n", threadIdx.x, i_pj, pjdim, *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						if(pover) {
							m_pdot += (pjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							m_sdot += (sjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							pjdim = spr_pre_jo[njo_pre2].rkdim;
							pover = 0;
							i = njo_pre2;
J0:;						m_pdot -= spr_pre_jo[i].rktsz;
							m_sdot -= spr_suf_jo[i].rktsz;
							if(--i < 0) goto LB1;
							else {//�߰� ���� ����
								if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
									pj_idx[i] = 0;
									goto J0;
								} else {
									i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
									if(i_pj == 0) pover = 1;
									m_pdot += (spr_pre_jo[i].rksz + i_pj * pjsz);//���� ���� ����(����)�� ����
									m_sdot += (spr_suf_jo[i].rksz + i_pj * sjsz);
									break;
								}
							}
						} else {
							i_pj = 0;
							pover = 1;
							pjdim = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
							m_pdot -= spr_pre_jo[njo_pre2].rktsz;//m_pdot -= (spr_pre_jo[njo_pre2].rktsz - (pjdim * pjsz));
							m_sdot -= spr_suf_jo[njo_pre2].rktsz;//m_sdot -= (spr_suf_jo[njo_pre2].rktsz - (pjdim * sjsz));
							break;
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						if(pover) {
							m_pdot += (pjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							pjdim = spr_pre_jo[njo_pre2].rkdim;
							pover = 0;
							i = njo_pre2;
J1:;						m_pdot -= spr_pre_jo[i].rktsz;
							if(--i < 0) break;
							else {
								if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
									pj_idx[i] = 0;
									goto J1;
								} else {
									i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
									if(i_pj == 0) pover = 1;
									m_pdot += (spr_pre_jo[i].rksz + i_pj * pjsz);//���� ���� ����(����)�� ����
									break;
								}
							}
						} else {
							i_pj = 0;
							pover = 1;
							pjdim = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
							m_pdot -= spr_pre_jo[njo_pre2].rktsz;//m_pdot -= (spr_pre_jo[njo_pre2].rktsz - (pjdim * pjsz));
							break;
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						if(sover) {
							m_sdot += (sjsz * (spr_suf_jo[njo_suf2].rkdim - sjdim));
							sjdim = spr_suf_jo[njo_suf2].rkdim;
							sover = 0;
							i = njo_suf2;
J2:;						m_sdot -= spr_suf_jo[i].rktsz;
							if(--i < 0) goto LB1;
							else {
								if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
									sj_idx[i] = 0;
									goto J2;
								} else {
									i_sj = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
									if(i_sj == 0) sover = 1;
									m_sdot += (spr_suf_jo[i].rksz + i_sj * sjsz);//���� ���� ����(����)�� ����
									break;
								}
							}
						} else {
							i_sj = 0;
							sover = 1;
							sjdim = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
							m_sdot -= spr_suf_jo[njo_suf2].rktsz;//m_sdot -= (spr_suf_jo[njo_suf2].rktsz - (sjdim * sjsz));
							break;
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ�� ����.
	//__shared__ T cache_bank[SM_SIZE];
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], tmp_idx[MX_DIM], i, i_pj, i_sj;
	register T sum;

	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	T *m_pdot, *m_sdot;
	for(;roff < n; roff += blockDim.x) {
		//__syncthreads();
		_offset2idx(dotv->noutRank, dotv->outRank, roff, tmp_idx);
		for(i = 0, m_pdot = _m_pdot, m_sdot = _m_sdot;i < dotv->noutRank; i++) {
			if(dotv->sprPreOut[i].rkPref) m_pdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
			else m_sdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
		}
		for(sum = 0, i_pj = i_sj = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%f %f\n", *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J0:;					m_pdot -= spr_pre_jo[i].rktsz;
						m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {//�߰� ���� ����
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J0;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								m_sdot += spr_suf_jo[i].rksz;
								break;
							}
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J1:;					m_pdot -= spr_pre_jo[i].rktsz;
						if(--i < 0) break;
						else {
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J1;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						i_sj = 0;
						i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {
							if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
								sj_idx[i] = 0;
								goto J2;
							} else {
								m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ��, �����޸𸮻��, �Ѱ� �� ���� ����.
	__shared__ T _cache_bank[SM_SIZE];
	T *cache_bank;
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	intt roff_ori = roff;
	//printf("qq %d %d %d %d %d %d\n", idx_width, idx_origin, blockIdx.x, blockDim.x, threadIdx.x, roff);
	if(roff >= n) return;

	SharePoint *sp = (SharePoint *)&_cache_bank[0];
	if(threadIdx.x == 0) {
		memcpy(sp->spr_out, dotv->sprPreOut, sizeof(SparseRank) * dotv->noutRank);
		sp->cache_axis = dotv->axisCache;
		sp->okfit = dotv->fitOutKernel;
		sp->jkfit = dotv->fitJoKernel;
		sp->oksz = blockDim.x;//��� ��Ʈ���� ũ�Ⱑ ��� Ŀ�� ������� ������ ��� ��Ʈ���� ũ��� ������.
		sp->jksz = dotv->szJoKernel;
		sp->nrecycc = dotv->nrecycCache;
		sp->ncycj = dotv->ncycJo;
		sp->njoint = dotv->nJointAxis;
		sp->nout = dotv->noutRank;
		memcpy(sp->out_rank, dotv->outRank, sizeof(intt) * dotv->noutRank);
		if(sp->spr_out[sp->cache_axis].rkPref) {//�����ϴ� �� �������� �ٷ����� (���)������ �� ������ ���� ���� ���ҵ��� 
			memcpy(sp->spr_jcache, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);	//�����޸𸮿� ĳ���Ͽ� �� ����(���)���� �Ѱ� ���Ҹ��� ���� ������ �����Ѵ�.
			memcpy(sp->jrank_cache, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_cache = dotv->njoPre;
			memcpy(sp->spr_jleaf, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);//��¿��� �����ϴ� �� ���� �ܸ� ����(�������� ������ pre�� suf�� �Ѱ� ����)
			memcpy(sp->jrank_leaf, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_leaf = dotv->njoSuf;
		} else {
			memcpy(sp->spr_jcache, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);
			memcpy(sp->jrank_cache, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_cache = dotv->njoSuf;
			memcpy(sp->spr_jleaf, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);
			memcpy(sp->jrank_leaf, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_leaf = dotv->njoPre;
		}
	}
	__syncthreads();
	//printf("zz %d %d\n", sp->jksz, dotv->szJoKernel);
	cache_bank = (T *)((bytet *)_cache_bank + sizeof(SharePoint));
	cache_bank = (T *)ALIGN_UNIT((divadx)cache_bank);
	const intt njo_leaf2 = sp->njo_leaf - 1;
	intt ljdim = sp->spr_jleaf[njo_leaf2].rkdim;
	const intt ljdim_ori = ljdim, ljsz = sp->spr_jleaf[njo_leaf2].rksz;
	intt leaf_idx[MX_DIM], tmp_idx[MX_DIM], i, i_lj, j;
	register T sum;

	T *m_pdot, *m_sdot, *m_cache, *m_leaf;
	intt inc = blockDim.x, itime_ok = sp->nrecycc, itime_jk = 0;//�ʱ� Ƚ���� ����.
	//intt iloop = 0;
	for(;; roff += inc) {
LP:;	//printf("aa %d %d %d\n", roff, itime_ok, sp->nrecycc);
		cache_load(n, roff_ori, roff, sp->okfit, sp->jkfit, sp->oksz, sp->jksz, sp->nrecycc, sp->ncycj, 
			sp->njoint, itime_ok, itime_jk, threadIdx.x, blockDim.x, inc, sp->cache_axis, sp->nout, sp->out_rank, tmp_idx,
			sp->spr_out, sp->spr_jcache, sp->spr_jleaf, sp->njo_cache, sp->jrank_cache, sp->njo_leaf, 
			sp->jrank_leaf,	_m_pdot, m_pdot, _m_sdot, m_sdot, m_cache, m_leaf, leaf_idx, cache_bank, LP);
		i_lj = leaf_idx[njo_leaf2];//�̹� ���� Ŀ�� �ָ��� ���� �ε���
		//printf("sss %d: %d %d %d\n", threadIdx.x, ljdim, i_lj);
		if(sp->jksz < ljdim - i_lj) i_lj = ljdim - sp->jksz;//���εǴ� ������ ����Ŀ�� ������(jksz)���� ���� ���� ��)���� üũ�ǹǷ� 
		//��������ʰ� ���λ���� Ŀ�� ���ҵǾ� jksz�� ���ϴܸ� �������� ������� ��)���� üũ�ǰ� ����.
		for(sum = 0, j = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%d: %d %d %f %f\n", threadIdx.x, sp->jksz, ljsz, *m_cache, *m_leaf);
			sum += *m_cache * *m_leaf;
			//printf("ss %d: %d %d\n", threadIdx.x, ljdim, i_lj);
			if(ljdim == ++i_lj) {//��.���� �ܸ� ���� �ε��� ����
				if(sp->jksz + leaf_idx[njo_leaf2] < (++j + 1) * ljdim) {//����Ŀ�λ���� �ʱ���� �������� �����Ͽ� ������ ���ϸ���
					i_lj = (j +1) * ljdim - (sp->jksz + leaf_idx[njo_leaf2]);//üũ������� ������ ���ϸ��� ������� 
					if(i_lj >= ljdim) break;									//������ŭ �����Ͽ� ������ ��)���� üũ�ǰ� �Ѵ�.
				} else i_lj = 0;
				i = njo_leaf2;
J2:;			m_leaf -= sp->spr_jleaf[i].rktsz;
				if(--i < 0) {
					//printf("vv %d: %d %d\n", threadIdx.x, ljdim, i_lj);
					goto LB1;//��.
				} else {
					if(sp->spr_jleaf[i].rkdim == ++leaf_idx[i]) {
						leaf_idx[i] = 0;
						goto J2;
					} else m_leaf += sp->spr_jleaf[i].rksz;//���� ���� ����(����)�� ����
				}
			} else m_leaf += ljsz;
			m_cache++;
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(itime_jk == 0 && rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(itime_jk == 0) {
				if(rplus == 0) *(m_rdot + roff) = 0;
				else *(m_rdot + roff) *= rplus;
			}
			*(m_rdot + roff) += sum;
			//printf("%f\n", sum);
		}
		//iloop++;
	}
	//printf("## %d\n", iloop);
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(oksz));//������ ��� Ŀ�� ������� �� ����� �� ũ�� �������� �ʴ´�.
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
template<typename T>
__global__ void kdot_f(void *pcxt, intt share_unit, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ��, �����޸𸮻��, ���� �� ���� ����.
	__shared__ T _cache_bank[SM_SIZE];//���߿� �����޸� �뷮�� Ŀ���� m_leaf��Ʈ���� ��ä�� leaf_idx, tmp_idx�� �����޸𸮿� �����Ѵ�.
	T *cache_bank;
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * share_unit + threadIdx.x;
	intt roff_ori = roff;
	//printf("qq %d %d %d %d %d %d\n", idx_width, idx_origin, blockIdx.x, blockDim.x, threadIdx.x, roff);
	if(roff >= n) return;
	
	share_unit = roff - threadIdx.x + share_unit;//share_unit�� �ٷ� �Ʒ��� ����ҷ��� ����, ���Ŀ��� ������ �����Ƿ�
	if(n > share_unit) n = share_unit;

	SharePoint *sp = (SharePoint *)&_cache_bank[0];
	if(threadIdx.x == 0) {
		memcpy(sp->spr_out, dotv->sprPreOut, sizeof(SparseRank) * dotv->noutRank);
		sp->cache_axis = dotv->axisCache;
		sp->okfit = dotv->fitOutKernel;
		sp->jkfit = dotv->fitJoKernel;
		sp->oksz = blockDim.x;//��� ��Ʈ���� ũ�Ⱑ ��� Ŀ�� ������� ������ ��� ��Ʈ���� ũ��� ������.
		sp->jksz = dotv->szJoKernel;
		sp->nrecycc = dotv->nrecycCache;
		sp->ncycj = dotv->ncycJo;
		sp->njoint = dotv->nJointAxis;
		sp->nout = dotv->noutRank;
		memcpy(sp->out_rank, dotv->outRank, sizeof(intt) * dotv->noutRank);
		if(sp->spr_out[sp->cache_axis].rkPref) {//�����ϴ� �� �������� �ٷ����� (���)������ �� ������ ���� ���� ���ҵ��� 
			memcpy(sp->spr_jcache, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);	//�����޸𸮿� ĳ���Ͽ� �� ����(���)���� �Ѱ� ���Ҹ��� ���� ������ �����Ѵ�.
			memcpy(sp->jrank_cache, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_cache = dotv->njoPre;
			memcpy(sp->spr_jleaf, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);//��¿��� �����ϴ� �� ���� �ܸ� ����(�������� ������ pre�� suf�� �Ѱ� ����)
			memcpy(sp->jrank_leaf, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_leaf = dotv->njoSuf;
		} else {
			memcpy(sp->spr_jcache, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);
			memcpy(sp->jrank_cache, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_cache = dotv->njoSuf;
			memcpy(sp->spr_jleaf, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);
			memcpy(sp->jrank_leaf, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_leaf = dotv->njoPre;
		}
	}
	__syncthreads();
	//printf("zz %d %d\n", sp->jksz, dotv->szJoKernel);
	cache_bank = (T *)((bytet *)_cache_bank + sizeof(SharePoint));
	cache_bank = (T *)ALIGN_UNIT((divadx)cache_bank);
	const intt njo_leaf2 = sp->njo_leaf - 1;
	intt ljdim = sp->spr_jleaf[njo_leaf2].rkdim;
	const intt ljsz = sp->spr_jleaf[njo_leaf2].rksz;
	intt leaf_idx[MX_DIM], tmp_idx[MX_DIM], i, i_lj, j;
	bool bw_get_ori = dotv->bwGetOri;
	register T sum;

	T *m_pdot, *m_sdot, *m_cache, *m_leaf;
	intt inc = blockDim.x, itime_ok = sp->nrecycc, itime_jk = 0;//�ʱ� Ƚ���� ����.
	//intt iloop = 0;
	for(;; roff += inc) {
LP:;	//printf("aa %d %d %d\n", roff, itime_ok, sp->nrecycc);
		cache_load(n, roff_ori, roff, sp->okfit, sp->jkfit, sp->oksz, sp->jksz, sp->nrecycc, sp->ncycj,
			sp->njoint, itime_ok, itime_jk, threadIdx.x, blockDim.x, inc, sp->cache_axis, sp->nout, sp->out_rank, tmp_idx,
			sp->spr_out, sp->spr_jcache, sp->spr_jleaf, sp->njo_cache, sp->jrank_cache, sp->njo_leaf,
			sp->jrank_leaf, _m_pdot, m_pdot, _m_sdot, m_sdot, m_cache, m_leaf, leaf_idx, cache_bank, LP);
		i_lj = leaf_idx[njo_leaf2];//�̹� ���� Ŀ�� �ָ��� ���� �ε���
		//printf("sss %d: %d %d %d\n", threadIdx.x, ljdim, i_lj);
		if(sp->jksz < ljdim - i_lj) i_lj = ljdim - sp->jksz;//���εǴ� ������ ����Ŀ�� ������(jksz)���� ���� ���� ��)���� üũ�ǹǷ� 
		//��������ʰ� ���λ���� Ŀ�� ���ҵǾ� jksz�� ���ϴܸ� �������� ������� ��)���� üũ�ǰ� ����.
		for(sum = 0, j = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%d: %d %d %f %f\n", threadIdx.x, sp->jksz, ljsz, *m_cache, *m_leaf);
			sum += *m_cache * *m_leaf;
			//printf("ss %d: %d %d\n", threadIdx.x, ljdim, i_lj);
			if(ljdim == ++i_lj) {//��.���� �ܸ� ���� �ε��� ����
				if(sp->jksz + leaf_idx[njo_leaf2] < (++j + 1) * ljdim) {//����Ŀ�λ���� �ʱ���� �������� �����Ͽ� ������ ���ϸ���
					i_lj = (j + 1) * ljdim - (sp->jksz + leaf_idx[njo_leaf2]);//üũ������� ������ ���ϸ��� ������� 
					if(i_lj >= ljdim) break;									//������ŭ �����Ͽ� ������ ��)���� üũ�ǰ� �Ѵ�.
				} else i_lj = 0;
				i = njo_leaf2;
J2:;			m_leaf -= sp->spr_jleaf[i].rktsz;
				if(--i < 0) {
					//printf("vv %d: %d %d\n", threadIdx.x, ljdim, i_lj);
					goto LB1;//��.
				} else {
					if(sp->spr_jleaf[i].rkdim == ++leaf_idx[i]) {
						leaf_idx[i] = 0;
						goto J2;
					} else m_leaf += sp->spr_jleaf[i].rksz;//���� ���� ����(����)�� ����
				}
			} else m_leaf += ljsz;
			m_cache++;
		}
LB1:;
		if(itime_jk) *(m_rdot + roff) += sum;
		else {
			if(rplus) *(m_rdot + roff) += sum;
			else *(m_rdot + roff) = sum;
		}
		//iloop++;
	}
	//printf("## %d\n", iloop);
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(oksz));//������ ��� Ŀ�� ������� �� ����� �� ũ�� �������� �ʴ´�.
	dim3 grid((idx_width + share_unit - 1) / share_unit);//�׸��� ����� ���Ϸ��� �Ʒ����� share_unit ����� 
	//grid((idx_width + share_unit *2 - 1) / share_unit *2);//���(x����� �����ϸ� ���������� share_unit x���� ó���ȴ�.)�ϸ� �ǳ� policyTrack������ ���� ó���Ǿ� �ϹǷ� �� share_unit�� x�谡 �ǰ��Ѵ�.
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <grid, block >> > (pcxt_dev, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gdot_f(void *pcxt_dev, intt oksz, intt share_unit, floatt *pdot_mdev, floatt *sdot_mdev, floatt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, floatt rplus)
{
	return gdot_t<floatt>(pcxt_dev, oksz, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
intt gdot_f(void *pcxt_dev, intt oksz, intt share_unit, intt *pdot_mdev, intt *sdot_mdev, intt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, intt rplus)
{
	return gdot_t<intt>(pcxt_dev, oksz, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
template<typename T>
__global__ void karith_f(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt *mrank = arv->arRankMast, *prank = arv->arRankPre, *srank = arv->arRankSuf, *rrank = arv->arRankRet;
	intt npre = arv->narPre, nsuf = arv->narSuf, nmast = arv->narMast;
	intt off, cidx[MX_DIM], tmp_idx[MX_DIM], coff;
	T rval, *ppre, *psuf;

	if(arv->bwGetOri) {
		dlead_offset2idx(arv->narBro, npre, arv->narRet, rrank, roff, cidx);
		for(;;) {//rrank�� �Ѱ� ���ҿ� ���Ͽ� rrank�� ������ 1�� ��ũ���� ���������� ��ȸ�Ͽ� ��ε�ĳ���õ� �͵��� pre�� suf ����
			coff = dbro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx);
			if(coff < 0) break;
			if(m_pari) {//�����Ŀ��� pref�� �����Ŀ��� ���ϵǴ� ��ε�ĳ��Ʈ�� ��Ʈ������ ������ mrank�� �ɼ°��ǹǷ�
				ppre = m_pari + coff;//�ٷ� ���.
				//printf("%d ", coff);
			} else ppre = &sval;
			if(m_sari) {
				off = dmoff2soff(nmast, mrank, nsuf, srank, coff, tmp_idx);
				psuf = m_sari + off;
				//printf("%d\n", off);
			} else psuf = &sval;
			switch(aop) {
			case AOP_MUL:
				rval = *ppre * *psuf;
				break;
			case AOP_PLUS:
				rval = *ppre;
				break;
			case AOP_DIV:
				break;
			case AOP_MINUS:
				break;
			case ABP_MINUS_PREF:
				rval = *ppre;
				break;
			case ABP_MINUS_SUFF:
				rval = *ppre * -1;
				break;
			case ABP_DIV_PREF:
				rval = *ppre * (1 / *psuf);
				break;
			case ABP_DIV_SUFF:
				rval = *ppre * (1 / (*psuf * *psuf) * -1);
				break;
			case ABP_BWTEST:
				rval = *ppre / *psuf;
				break;
			}
			if(aop == ABP_BWTEST) {
				if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
				else if(*(m_rari + roff) != rval) printf("xxx\n");
			} else {
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
		}
	} else {
		if(m_pari) {
			off = dmoff2soff(nmast, mrank, npre, prank, roff, tmp_idx);
			ppre = m_pari + off;
			//printf("%d ", off);
		} else ppre = &sval;
		if(m_sari) {
			off = dmoff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx);
			psuf = m_sari + off;
			//printf("%d\n", off);
		} else psuf = &sval;
		switch(aop) {
		case AOP_MUL:
			rval = *ppre * *psuf;
			break;
		case AOP_PLUS:
			rval = *ppre + *psuf;
			break;
		case AOP_DIV:
			rval = *ppre / *psuf;
			break;
		case AOP_MINUS:
			rval = *ppre - *psuf;
			break;
		}
		if(rplus) {
			*(m_rari + roff) *= rplus;
			*(m_rari + roff) += rval;
		} else *(m_rari + roff) = rval;
	}
}
template<typename T>
__global__ void karith_f1(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��ε� �ɽ�Ʈ�� ���� ���� ��Ʈ���� �ϴ��� ���� 
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;//pre�� suf�� ret�� ��ġ�� pre�� suf�� ���Ǳ����� �����Ǳ� ���� ���µǹǷ� ���� �Ի��ϰ� �����ϱ�����
	if(arv->bwGetOri) {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * *(m_sari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF:
			rval = *(m_pari + roff) * -1;
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			rval = *(m_pari + roff) * (1 / *(m_sari + roff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			rval = *(m_pari + roff) * (1 / (*(m_sari + roff) * *(m_sari + roff)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = *(m_pari + roff) / *(m_sari + roff);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff) + *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff) / *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff) - *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_bwprem(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��� ������ ���� ��ε� �ɽ�Ʈ(������� 1�� ������ ����) �����, m_par����� m_rari������ ���� Ŀ�� m_par�� �׸��������
	intt poff = blockIdx.x*blockDim.x + threadIdx.x;
	ArithVar *arv = (ArithVar *)pcxt;
	intt osz = arv->zarOut, ssz = arv->zarSuf, psz = arv->zarPre;
	intt roff = poff % osz;

	if(roff < idx_width * idx_origin || roff >= n) return;//���� ��Ʈ������ ���ҵǾ��� ��� pre�κ��� ������ roff�� ���� üũ

	T rval;
	switch(aop) {
	case AOP_MUL:
		rval = *(m_pari + poff) * *(m_sari + poff % ssz);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_PLUS:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_DIV:
		break;
	case AOP_MINUS:
		break;
	case ABP_MINUS_PREF:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_MINUS_SUFF:
		rval = *(m_pari + poff) * -1;
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_PREF:
		rval = *(m_pari + poff) * (1 / *(m_sari + poff % ssz));
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_SUFF:
		rval = *(m_pari + poff) * (1 / (*(m_sari + poff % ssz) * *(m_sari + poff % ssz)) * -1);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_BWTEST:
		rval = *(m_pari + poff) / *(m_sari + poff % ssz);
		if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
		else if(*(m_rari + roff) != rval) printf("xxx\n");
		break;
	}
}
template<typename T>
__global__ void karith_f2(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��� ������ ���� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt osz = arv->zarOut, ssz = arv->zarSuf, psz = arv->zarPre, poff;
	T rval;
	if(arv->bwGetOri) {//pre�� out�� ����� ���� ���, pre����� out���� ū ���� �� �Լ����� ����, pre�� �����Ŀ���
		switch(aop) {	//���� ��Ʈ�����̹Ƿ� pre�� �� ���� ���� ����.
		case AOP_MUL:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff) * *(m_sari + poff % ssz);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case AOP_PLUS:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_MINUS_SUFF:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff) * -1;
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_DIV_PREF:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff) * (1 / *(m_sari + poff % ssz));
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_DIV_SUFF:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff) * (1 / (*(m_sari + poff % ssz) * *(m_sari + poff % ssz)) * -1);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_BWTEST:
			for(poff = roff;poff < psz; poff += osz) {
				rval = *(m_pari + poff) / *(m_sari + poff % ssz);
				if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
				else if(*(m_rari + roff) != rval) printf("xxx\n");
			}
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff % psz) * *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff % psz) + *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff % psz) / *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff % psz) - *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_lc(void *pcxt, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{//������ ����� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;
	if(arv->bwGetOri) {
		switch(aop) {
		case AOP_MUL:
			rval = (sval * *(m_sari + roff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			if(rplus) *(m_rari + roff) += sval;
			else *(m_rari + roff) = sval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			if(rplus) *(m_rari + roff) += sval;
			else *(m_rari + roff) = sval;
			break;
		case ABP_MINUS_SUFF:
			if(rplus) *(m_rari + roff) += sval * -1;
			else *(m_rari + roff) = sval * -1;
			break;
		case ABP_DIV_PREF:
			rval = sval * (1 / *(m_sari + roff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			rval = sval * (1 / (*(m_sari + roff) * *(m_sari + roff)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = sval / *(m_sari + roff);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = (sval * *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = (sval + *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = (sval / *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = (sval - *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_rc(void *pcxt, T *m_pari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{//������ ����� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;
	if(arv->bwGetOri) {
		switch(aop) {
		case AOP_MUL:
			rval = (*(m_pari + roff) * sval);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF:
			rval = *(m_pari + roff) * -1;
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			rval = *(m_pari + roff) * (1 / sval);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			rval = *(m_pari + roff) * (1 / (sval * sval) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = *(m_pari + roff) / sval;
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff) + sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff) / sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff) - sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f3(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//������� 1�� ������ �����ϴ� ��ε� �ɽ�Ʈ, ��� ������ ����̸� ����� ������� 1�� ������ �ǹ̰� �������Ƿ� Ÿ�� 2�� ���̽��� �����.
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt *mrank = arv->arRankMast, *prank = arv->arRankPre, *srank = arv->arRankSuf, *rrank = arv->arRankRet;
	intt npre = arv->narPre, nsuf = arv->narSuf, nmast = arv->narMast, nret = arv->narRet;
	intt cidx[MX_DIM], tmp_idx[MX_DIM], poff, soff;
	T rval;
	bool end_check;

	if(arv->bwGetOri) {
		_lead_offset2idx(arv->narBro, npre, nret, rrank, roff, cidx, end_check);
		switch(aop) {
		case AOP_MUL:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
				rval = *(m_pari + poff) * *(m_sari + soff);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case AOP_PLUS:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				rval = *(m_pari + poff);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				rval = *(m_pari + poff);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_MINUS_SUFF:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				rval = *(m_pari + poff) * -1;
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_DIV_PREF:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
				rval = *(m_pari + poff) * (1 / *(m_sari + soff));
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_DIV_SUFF:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
				rval = *(m_pari + poff) * (1 / (*(m_sari + soff) * *(m_sari + soff)) * -1);
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
			break;
		case ABP_BWTEST:
			for(;;) {
				_bro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx, poff, end_check);
				_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
				rval = *(m_pari + poff) / *(m_sari + soff);
				if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
				else if(*(m_rari + roff) != rval)  printf("xxx\n");
			}
			break;
		}
	} else {
		_moff2soff(nmast, mrank, npre, prank, roff, tmp_idx, poff);
		_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
		//printf("11111111 %d %d %f\n", poff, soff, rval);
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + poff) * *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + poff) + *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + poff) / *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + poff) - *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
intt garith_t(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, sytet tp_arith)
{
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);
	if(tp_arith == AR_T_BRO && p_size > r_size) {//pre����� ret������� ū���� ��ε��ɽ�Ʈ ������ ���ۿ� ����.
		dim3 block(WIDTH_BLOCK3(p_size));			//�̰�� pre�� �������� ������ �����Ѵ�.
		dim3 grid((p_size + block.x - 1) / block.x);
		karith_f2_bwprem<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
	} else {
		intt bsz = (tp_arith == AR_T_ONEBRO ? SMALL_BLOCK : BLOCK_SIZE);
		dim3 block(WIDTH_BLOCK2(bsz));
		dim3 grid((idx_width + block.x - 1) / block.x);
		//karith_f<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
		switch(tp_arith) {
		case AR_T_O2O:
			karith_f1<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		case AR_T_BRO:
			karith_f2<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		case AR_T_BROLC:
			karith_f2_lc<T> << <grid, block >> > (pcxt, m_sari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
			break;
		case AR_T_BRORC:
			karith_f2_rc<T> << <grid, block >> > (pcxt, m_pari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
			break;
		case AR_T_ONEBRO:
			karith_f3<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		}
	}
	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt garith_f(void *pcxt, floatt *m_pari, floatt *m_sari, floatt *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, floatt sval, sytet aop, floatt rplus, sytet tp_arith)
{
	return garith_t<floatt>(pcxt, m_pari, m_sari, m_rari, p_size, r_size,
		idx_origin, idx_width, sval, aop, rplus, tp_arith);
}
intt garith_f(void *pcxt, intt *m_pari, intt *m_sari, intt *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, intt sval, sytet aop, intt rplus, sytet tp_arith)
{
	return garith_t<intt>(pcxt, m_pari, m_sari, m_rari, p_size, r_size,
		idx_origin, idx_width, sval, aop, rplus, tp_arith);
}

template<typename T>
__global__ void ktrans_f(void *pcxt, T *m_strs, T *m_rtrs, intt idx_origin, intt idx_width, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	TransVar *tsvar = (TransVar *)pcxt;
	intt ndims = tsvar->ntrDims, *rrank = tsvar->trRankRet, i, ridx[MX_DIM];
	TransRank *tmap = tsvar->tspmap;

	_offset2idx(ndims, rrank, roff, ridx);
	for(i = 0;i < ndims; i++) {
		m_strs += (ridx[i] * tmap[i].trssz);
	}
	if(bw) *(m_rtrs + roff) += *m_strs;
	else *(m_rtrs + roff) = *m_strs;
}
template<typename T>
intt gtrans_t(void *pcxt, T *m_strs, T *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ktrans_f<T> << <grid, block >> > (pcxt, m_strs, m_rtrs, idx_origin, idx_width, bw, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gtrans_f(void *pcxt, floatt *m_strs, floatt *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	return gtrans_t<floatt>(pcxt, m_strs, m_rtrs, r_size, idx_origin, idx_width, bw);
}
intt gtrans_f(void *pcxt, intt *m_strs, intt *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	return gtrans_t<intt>(pcxt, m_strs, m_rtrs, r_size, idx_origin, idx_width, bw);
}
template<typename T>
__device__ void datomic_max_f(T *pmax, const T value)
{
	if(*pmax >= value) return;

	intt * const imax = (int *)pmax;
	intt old = *imax, vcmp;

	do
	{
		vcmp = old;
		if(__int_as_float(vcmp) >= value) break;

		old = atomicCAS(imax, vcmp, __float_as_int(value));
	} while(vcmp != old);
}
template<typename T>
__global__ void ksoftx_prob_f(void *pcxt, T *m_rsfx, T *m_sum, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	*(m_rsfx + roff) = *(m_rsfx + roff) / (*(m_sum + roff / f_size) + 1e-8);
}
template<typename T>
__global__ void ksoftx_sum_f(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_sum, T *m_max, intt f_size, sytet db, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	T v;
	if(roff >= n) return;

	if(db) v = std::exp((doublet)(*(m_ssfx + roff) - *(m_max + roff / f_size)));
	else v = std::exp((floatt)(*(m_ssfx + roff) - *(m_max + roff / f_size)));
	atomicAdd(m_sum + roff / f_size, v);
	*(m_rsfx + roff) = v;
	
}
template<typename T>
__global__ void ksoftx_max_f(void *pcxt, T *m_ssfx, T *m_max, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;
	
	datomic_max_f(m_max + roff / f_size, *(m_ssfx + roff));
}
template<typename T>
intt gsoftx_t(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_sum, T *m_max, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksoftx_max_f<T> << <grid, block >> > (pcxt, m_ssfx, m_max, f_size, idx_origin, idx_width, n);
	ksoftx_sum_f<T> << <grid, block >> > (pcxt, m_ssfx, m_rsfx, m_sum, m_max, f_size, db, idx_origin, idx_width, n);
	ksoftx_prob_f<T> << <grid, block >> > (pcxt, m_rsfx, m_sum, f_size, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gsoftx_f(void *pcxt, floatt *m_ssfx, floatt *m_rsfx, floatt *m_sum, floatt *m_max, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	return gsoftx_t<floatt>(pcxt, m_ssfx, m_rsfx, m_sum, m_max, r_size, f_size, db, idx_origin, idx_width);
}
intt gsoftx_f(void *pcxt, intt *m_ssfx, intt *m_rsfx, intt *m_sum, intt *m_max, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	return gsoftx_t<intt>(pcxt, m_ssfx, m_rsfx, m_sum, m_max, r_size, f_size, db, idx_origin, idx_width);
}

template<typename T>
__global__ void ksoftx_cross_e_f(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_tsfx, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(m_rsfx + (roff / f_size), -1.0f * std::log(*(m_ssfx + roff) + 1e-8) * *(m_tsfx + roff));
	//printf("%f %f %f\n", *(m_rsfx + (roff / f_size)), *(m_ssfx + roff), *(m_tsfx + roff));
}
template<typename T>
intt gsoftx_cross_e_t(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksoftx_cross_e_f<T> << <grid, block >> > (pcxt, m_ssfx, m_rsfx, m_tsfx, f_size, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gsoftx_cross_e_f(void *pcxt, floatt *m_ssfx, floatt *m_rsfx, floatt *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	return gsoftx_cross_e_t<floatt>(pcxt, m_ssfx, m_rsfx, m_tsfx, r_size, f_size, idx_origin, idx_width);
}
intt gsoftx_cross_e_f(void *pcxt, intt *m_ssfx, intt *m_rsfx, intt *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	return gsoftx_cross_e_t<intt>(pcxt, m_ssfx, m_rsfx, m_tsfx, r_size, f_size, idx_origin, idx_width);
}

template<typename T>
__global__ void ksum_f(void *pcxt, T *m_smet, T *m_rmet, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(m_rmet, *(m_smet + roff));
}
template<typename T>
__global__ void kmean_f(T *m_rmet, T *cmul, bool mean, intt r_size)
{
	if(cmul) *m_rmet *= *(T *)cmul;
	if(mean) *m_rmet /= r_size;
	//printf("%p %d %f\n", cmul, mean, *m_rmet);
}
template<typename T>
intt gsum_t(void *pcxt, T *m_smet, T *m_rmet, intt r_size, intt idx_origin, intt idx_width, T *cmul, bool mean)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksum_f<T> << <grid, block >> > (pcxt, m_smet, m_rmet, idx_origin, idx_width, n);
	if(n == r_size && (cmul || mean)) kmean_f<T> << <1, 1 >> > (m_rmet, cmul, mean, r_size);//������ ���ҿ��� ��ձ��ϱ� ����.

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gsum_f(void *pcxt, floatt *m_smet, floatt *m_rmet, intt r_size, intt idx_origin, intt idx_width, floatt *cmul, bool mean)
{
	return gsum_t<floatt>(pcxt, m_smet, m_rmet, r_size, idx_origin, idx_width, cmul, mean);
}
intt gsum_f(void *pcxt, intt *m_smet, intt *m_rmet, intt r_size, intt idx_origin, intt idx_width, intt *cmul, bool mean)
{
	return gsum_t<intt>(pcxt, m_smet, m_rmet, r_size, idx_origin, idx_width, cmul, mean);
}

template<typename T>
__global__ void koptadm_f(T *mm, T *mv, T *mg, T *mr, intt idx_origin,
	intt idx_width, T beta1, T beta2, T lr, T ep, intt dec, sytet db, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mm[roff] += (1.0f - beta1) * (mg[roff] - mm[roff]);//����: mm[roff] + (l - beta1)*mg[roff] - mm[roff] + mm[roff]*beta1
													//	  = mm[roff]*beta1 + (l - beta1)*mg[roff]
	mv[roff] += (1.0f - beta2) * (mg[roff] * mg[roff] - mv[roff]);
	mr[roff] += dec * lr * mm[roff] / (std::sqrt(db ? (doublet)mv[roff] : (floatt)mv[roff]) + ep);
}
template<typename T>
intt goptadm_t(void *pcxt, T *mm, T *mv, T *mg, T *mr, intt r_size, intt idx_origin,
	intt idx_width, T beta1, T beta2, T lr, T ep, intt dec, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	koptadm_f<T> << <grid, block >> > (mm, mv, mg, mr, idx_origin, idx_width, beta1, beta2, lr, ep, dec, db, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt goptadm_f(void *pcxt, floatt *mm, floatt *mv, floatt *mg, floatt *mr, intt r_size, intt idx_origin,
	intt idx_width, floatt beta1, floatt beta2, floatt lr, floatt ep, intt dec, sytet db)
{
	return goptadm_t<floatt>(pcxt, mm, mv, mg, mr, r_size, idx_origin,
		idx_width, beta1, beta2, lr, ep, dec, db);
}
intt goptadm_f(void *pcxt, intt *mm, intt *mv, intt *mg, intt *mr, intt r_size, intt idx_origin,
	intt idx_width, intt beta1, intt beta2, intt lr, intt ep, intt dec, sytet db)
{
	return goptadm_t<intt>(pcxt, mm, mv, mg, mr, r_size, idx_origin,
		idx_width, beta1, beta2, lr, ep, dec, db);
}

template<typename T>
__global__ void koptsgd_f(T *mg, T *mr, intt idx_origin, intt idx_width, T lr, intt dec, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mr[roff] += dec * lr * mg[roff];
}
template<typename T>
intt goptsgd_t(void *pcxt, T *mg, T *mr, intt r_size, intt idx_origin, intt idx_width, T lr, intt dec)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	koptsgd_f<T> << <grid, block >> > (mg, mr, idx_origin, idx_width, lr, dec, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt goptsgd_f(void *pcxt, floatt *mg, floatt *mr, intt r_size, intt idx_origin, intt idx_width, floatt lr, intt dec)
{
	return goptsgd_t<floatt>(pcxt, mg, mr, r_size, idx_origin, idx_width, lr, dec);
}
intt goptsgd_f(void *pcxt, intt *mg, intt *mr, intt r_size, intt idx_origin, intt idx_width, intt lr, intt dec)
{
	return goptsgd_t<intt>(pcxt, mg, mr, r_size, idx_origin, idx_width, lr, dec);
}
template<typename T>
__device__ __forceinline__ T mat_sqrt(T a, sytet db)
{
	return std::sqrt(db ? (doublet)a : (floatt)a);
}
template<typename T>
__device__ __forceinline__ T mat_exp(T a, sytet db)
{
	return std::exp(db ? (doublet)a : (floatt)a);
}
template<typename T>
__device__ __forceinline__ float mat_log(T a, sytet db) {
	return std::log(db ? (doublet)a : (floatt)a);
}
template<typename T>
__global__ void kactf_f(T *mpre, T *mret, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	T v;

	if(roff >= n) return;

	switch(aop2) {
	case ACTF_TANH:
		mret[roff] = std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]);
		break;
	case DACTF_TANH:
		if(rplus) mret[roff] += (1.0f - std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]) *
					std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]));
		else mret[roff] = 1.0f - std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]) * 
					std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]);
		break;
	case ACTF_RELU:
		mret[roff] = mpre[roff] > 0.0f ? mpre[roff] : 0.0f;
		break;
	case DACTF_RELU:
		if(rplus) mret[roff] += (mpre[roff] > 0.0f ? 1.0f : 0.0f);
		else mret[roff] = mpre[roff] > 0.0f ? 1.0f : 0.0f;
		break;
	case ACTF_SIGM:
		mret[roff] = 1.0 / (1.0 + mat_exp(-mpre[roff], db));//1.0f/(1.0f + std::exp(-a));
		break;
	case DACTF_SIGM:
		v = 1.0 / (1.0 + mat_exp(-mpre[roff], db));
		if(rplus) mret[roff] += (1.0 - v) * v;
		else mret[roff] = (1.0 - v) * v;
		break;
	case MATH_SQRT:
		mret[roff] = mat_sqrt(mpre[roff], db);
		break;
	case DMATH_SQRT:
		if(rplus) mret[roff] += 0.5 * 1.0 / mat_sqrt(mpre[roff], db);//0.5 * pow(mpre[roff], -0.5f)
		else mret[roff] = 0.5 * 1.0 / mat_sqrt(mpre[roff], db);//0.5 * pow(mpre[roff], -0.5f)
		break;
	case JUST_COPY:
		break;
	case DJUST_COPY://�ܼ��� �����Ķ� ���⸦ ���ϱ����� ���.
		if(rplus) mret[roff] += mpre[roff];
		else mret[roff] = mpre[roff];
		break;
	case MATH_LOG:
		mret[roff] = mat_log(mpre[roff], db);
		break;
	}
}
template<typename T>
intt gactf_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kactf_f<T> << <grid, block >> > (mpre, mret, idx_origin, idx_width, aop2, rplus, db, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gactf_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, aop2, rplus, db);
}
intt gactf_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, aop2, rplus, db);
}
template<typename T>
__global__ void kactf2_f(T *mpre, T *msuf, T *mret, T *rsuf, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	switch(aop2) {
	case ACTF_PRELU:
		mret[roff] = mpre[roff] > 0.0 ? mpre[roff] : msuf[roff] * mpre[roff];// x > 0.0f ? x : a * x
		break;
	case DACTF_PRELU:
		if(rplus) {
			mret[roff] += (mpre[roff] > 0.0 ? 1.0 : msuf[roff]);//dst[idx] = src[idx] > 0.0f ? 1.0f : a[idx];
			rsuf[roff] += (mpre[roff] > 0.0 ? 0.0 : mpre[roff]);//da[idx] = src[idx] > 0.0f ? 0.0f : src[idx];
		} else {
			mret[roff] = mpre[roff] > 0.0 ? 1.0 : msuf[roff];//dst[idx] = src[idx] > 0.0f ? 1.0f : a[idx];
			rsuf[roff] = mpre[roff] > 0.0 ? 0.0 : mpre[roff];//da[idx] = src[idx] > 0.0f ? 0.0f : src[idx];
		}
		break;
	}
}
template<typename T>
intt gactf2_t(void *pcxt, T *mpre, T *msuf, T *mret, T *rsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kactf2_f<T> << <grid, block >> > (mpre, msuf, mret, rsuf, idx_origin, idx_width, aop2, rplus, db, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gactf2_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, floatt *rsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf2_t<floatt>(pcxt, mpre, msuf, mret, rsuf, r_size, idx_origin, idx_width, aop2, rplus, db);
}
intt gactf2_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt *rsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf2_t<intt>(pcxt, mpre, msuf, mret, rsuf, r_size, idx_origin, idx_width, aop2, rplus, db);
}
template<typename T>
__global__ void kembedding_f(T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	intt idx;

	if(roff >= n) return;

	int_val_type(idx, &msuf[roff / sz_embed], stp);
	if(bw) {//msuf - input, mret - lookup table, mpre - embeded, roff�� mpre ����, roff�� mret������ �ƴϹǷ�
			//������(cpu������� �ƴ϶� gpu Ŀ�� �����嵵)�� mret�� ����� ��ø�ɼ��־� ��Ÿó�� �Ѵ�.
		atomicAdd(&mret[idx*sz_embed + roff % sz_embed], mpre[roff]);
	} else {//msuf - input, mret - embeded, mpre - lookup table, roff�� mret ����
		mret[roff] = mpre[idx*sz_embed + roff % sz_embed];
	}
}
template<typename T>
intt gembedding_t(T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kembedding_f<T> << <grid, block >> > (mpre, msuf, mret, idx_origin, idx_width, sz_embed, stp, bw, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gembedding_f(floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt bw)
{
	return gembedding_t<floatt>(mpre, msuf, mret, r_size, idx_origin, idx_width, sz_embed, stp, bw);
}
intt gembedding_f(intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt bw)
{
	return gembedding_t<intt>(mpre, msuf, mret, r_size, idx_origin, idx_width, sz_embed, stp, bw);
}
template<typename T>
__global__ void konehot_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, intt n)
{
	OneVar *ovar = (OneVar *)pcxt;
	intt poff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x, depth = ovar->idxOne[5];

	if(poff >= n || *(mpre + poff) >= depth || *(mpre + poff) < 0) return;
	
	intt roff, psz = ovar->idxOne[4];

	roff = (poff / psz) * depth * psz + *(mpre + poff) * psz + poff % psz;
	*(mret + roff) = (T)*(doublet *)ovar->idxOne;
}
template<typename T>
intt gonehot_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	konehot_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gonehot_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gonehot_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
intt gonehot_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gonehot_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
template<typename T>
__global__ void kslice_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, bool bw, sytet rplus, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	SliceRank *slicer = (SliceRank *)ovar->idxOne;
	intt *srank, ndims = ovar->nrkPre;
	intt ridx[MX_DIM], i;
	T *cmem, *smem;

	srank = ovar->rankOut;
	if(bw) {
		cmem = mret;
		smem = mpre;//slice matrix
	} else {
		cmem = mpre;
		smem = mret;//slice matrix
	}
	_offset2idx(ndims, srank, roff, ridx);
	for(i = 0;i < ndims; i++) {
		cmem += (slicer[i].slbase + ridx[i] * slicer[i].slsz);
	}
	if(bw) {
		if(rplus) *cmem += *(smem + roff);
		else *cmem = *(smem + roff);
	} else *(smem + roff) = *cmem;
}
template<typename T>
intt gslice_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kslice_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, bw, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gslice_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	return gslice_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, bw, rplus);
}
intt gslice_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	return gslice_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, bw, rplus);
}
template<typename T>
__global__ void kargmax_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	intt poff, i, naxis = ovar->nrkPre, inner_sz = ovar->nrkSuf, outer_sz = ovar->nrkOut;
	T vmax;

	for(;roff < n; roff++) {
		poff = (roff / inner_sz) * outer_sz + roff % inner_sz;
		for(i = 0, vmax = 0;i < naxis; i++, poff += inner_sz) {
			if(vmax < *(mpre + poff)) {
				vmax = *(mpre + poff);
				*(mret + roff) = i;
			}
		}
	}
}
template<typename T>
intt gargmax_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kargmax_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gargmax_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gargmax_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
intt gargmax_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gargmax_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}

template<typename T>
__global__ void kequal_f(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	bool eq = ovar->idxOne[0], cscalr = ovar->idxOne[1];
	T csv = *(T *)&ovar->idxOne[2];

	for(;roff < n; roff++) {
		if(cscalr) {
			if(eq) {
				if(mpre[roff] == csv) mret[roff] = (T)1;
				else mret[roff] = (T)0;
			} else {
				if(mpre[roff] == csv) mret[roff] = (T)0;
				else mret[roff] = (T)1;
			}
		} else {
			if(eq) {
				if(mpre[roff] == msuf[roff]) mret[roff] = (T)1;
				else mret[roff] = (T)0;
			} else {
				if(mpre[roff] == msuf[roff]) mret[roff] = (T)0;
				else mret[roff] = (T)1;
			}
		}
	}
}
template<typename T>
intt gequal_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kequal_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gequal_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gequal_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width);
}
intt gequal_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gequal_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width);
}
template<typename T>
__global__ void ktype1_t(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt aop2, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	
	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	doublet low = *(doublet *)&ovar->idxOne[0], high = *(doublet *)&ovar->idxOne[2];

	switch(aop2) {
	case TYPE1_CLIP:
		if(mpre[roff] < low) mret[roff] = low;
		else if(mpre[roff] > high) mret[roff] = high;
		else mret[roff] = mpre[roff];
		break;
	}
}
template<typename T>
intt gtype1_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ktype1_t<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, aop2, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gtype1_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtype1_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2);
}
intt gtype1_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtype1_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2);
}
#include <hiprand/hiprand_kernel.h>
template<typename T>
__global__ void knormal_t(T *mpre, hiprandState *cust, intt n)
{
	intt roff = blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mpre[roff] = hiprand_normal(&cust[roff]);
}
template<typename T>
__global__ void kuniform_t(T *mpre, hiprandState *cust, intt n)
{
	intt roff = blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mpre[roff] = hiprand_uniform(&cust[roff]);
}
__global__ void seed_random(hiprandState *cus, intt seed, intt n)
{
	intt roff = blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	hiprand_init(seed, roff, 0, &cus[roff]);
}
template<typename T>
intt grandom_t(T *mpre, intt idx_width, intt aop2)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	dim3 grid((idx_width + block.x - 1) / block.x);

	hiprandState *cust;
	hipMalloc((void **)&cust, idx_width * sizeof(hiprandState));
	seed_random<< <grid, block >> > (cust, 0, idx_width);

	switch(aop2) {
	case RAND_T_N:
		knormal_t<T> << <grid, block >> > (mpre, cust, idx_width);
		break;
	case RAND_T_U:
		kuniform_t<T> << <grid, block >> > (mpre, cust, idx_width);
		break;
	case RAND_T_L:
		break;
	case RAND_T_P:
		break;
	}
	hipFree(cust);
	hipDeviceSynchronize();
	return idx_width;
}
intt grandom_f(floatt *mpre, intt r_size, intt aop2)
{
	return grandom_t<floatt>(mpre, r_size, aop2);
}
intt grandom_f(intt *mpre, intt r_size, intt aop2)
{
	return grandom_t<intt>(mpre, r_size, aop2);
}