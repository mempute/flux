#include "hip/hip_runtime.h"

#include "mgpu.h"
#include "matrix.h"
#include <stdio.h>

/*//dot version 1
template<typename T>
__global__ void kdot_f(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt pidx[MX_DIM], sidx[MX_DIM], ridx[MX_DIM], tmp_idx[MX_DIM], poff, soff;
	intt sz_suf_shrink = dotv->szShrinkSuf, jo_top_axis_p = dotv->joTopAxisPre, jo_top_axis_s = dotv->joTopAxisSuf;
	intt pdim = dotv->ndimPre, sdim = dotv->ndimSuf, n_joint_axis = dotv->nJointAxis;
	intt *prank = dotv->szRankPre, *srank = dotv->szRankSuf, i, j;
	T sum = 0;

	if(dotv->bwGetOri == BWDIV_PREF) {//A*B=C, A(ret)=C*B',�����Ķ� A�� ������� �����Ķ� C�� ������̵ǰ� �����Ķ� A�� ��������
		doffset2idx(dotv->ndimRet, dotv->szRankRet, roff, ridx);//�����Ķ� B�� ������̵ȴ�.���⼭ B��C�� ������� ���ϰ� �ؿ���
		if(dotv->noutPre > 1 || dotv->noutRet > 1) {			//�������� ���� ����
			soff = dsparse_idx2offset(dotv->noutRet, dotv->outRankRet, ridx, dotv->outAxisRet);//A�� ��·�ũ�� �ɼº�ȯ��
			doffset2idx(dotv->noutPre, dotv->outRankPre, soff, tmp_idx);//��ȯ �ɼ°� C�� ��� ��ũ�� �ε��� �Ի�
			for(j = 0;j < dotv->noutPre; j++) pidx[dotv->outAxisPre[j]] = tmp_idx[j];//C�� ��� �ε��� ����
		} else {//A�� ��� �ε����� C�� ��� �ε����� �ٷ� ����.
			for(j = 0;j < dotv->noutPre; j++) pidx[dotv->outAxisPre[j]] = ridx[dotv->outAxisRet[j]];//��ü �ε����� ������� �ε��� ����.
		}
		if(dotv->noutSuf > 1 || dotv->njoRet > 1) {
			soff = dsparse_idx2offset(dotv->njoRet, dotv->joRankRet, ridx, dotv->joAxisRet);//A�� ���η�ũ�� �ɼº�ȯ��
			doffset2idx(dotv->noutSuf, dotv->outRankSuf, soff, tmp_idx);//��ȯ �ɼ°� B�� ��� ��ũ�� �ε��� �Ի�
			for(j = 0;j < dotv->noutSuf; j++) sidx[dotv->outAxisSuf[j]] = tmp_idx[j];//B�� ��� �ε��� ����
		} else {//A�� ���� �ε����� B�� ��� �ε����� �ٷ� ����.
			for(j = 0;j < dotv->noutSuf; j++) sidx[dotv->outAxisSuf[j]] = ridx[dotv->joAxisRet[j]];
		}
	} else if(dotv->bwGetOri == BWDIV_SUFF) {//A*B=C, B(ret)=A'*C,�����Ķ� B�� �������� �����Ķ� A�� ������̵ǰ� �����Ķ�B�� �������
		doffset2idx(dotv->ndimRet, dotv->szRankRet, roff, ridx);//�����Ķ� C�� ������̵ȴ�.���⼭ A��C�� ������� ���ϰ� �ؿ��� ������
		if(dotv->noutPre > 1 || dotv->njoRet > 1) {				//�� ���� ����.
			soff = dsparse_idx2offset(dotv->njoRet, dotv->joRankRet, ridx, dotv->joAxisRet);//B�� ���η�ũ�� �ɼº�ȯ��
			doffset2idx(dotv->noutPre, dotv->outRankPre, soff, tmp_idx);//��ȯ �ɼ°� A�� ��� ��ũ�� �ε��� �Ի�
			for(j = 0;j < dotv->noutPre; j++) pidx[dotv->outAxisPre[j]] = tmp_idx[j];//A�� ��� �ε��� ����
		} else {//B�� ���� �ε����� A�� ��� �ε����� �ٷ� ����.
			for(j = 0;j < dotv->noutPre; j++) pidx[dotv->outAxisPre[j]] = ridx[dotv->joAxisRet[j]];//��ü �ε����� ������� �ε��� ����.
		}
		if(dotv->noutSuf > 1 || dotv->noutRet > 1) {
			soff = dsparse_idx2offset(dotv->noutRet, dotv->outRankRet, ridx, dotv->outAxisRet);//B�� ��·�ũ�� �ɼ� ��ȯ��
			doffset2idx(dotv->noutSuf, dotv->outRankSuf, soff, tmp_idx);//��ȯ �ɼ°� C�� ��� ��ũ�� �ε��� ���
			for(j = 0;j < dotv->noutSuf; j++) sidx[dotv->outAxisSuf[j]] = tmp_idx[j];//C�� ��� �ε��� ����
		} else {//B�� ��� �ε����� C�� ��� �ε����� �ٷ� ����.
			for(j = 0;j < dotv->noutSuf; j++) sidx[dotv->outAxisSuf[j]] = ridx[dotv->outAxisRet[j]];
		}
	} else {
		poff = roff / sz_suf_shrink;//���� �����󿡼� �ΰ� ��Ʈ������ ������ ��Ʈ�����󿡼��� �ɼ��� suffix��Ʈ������
							//n_joint_axis dim�� ������ ������� ������ prefix��Ʈ�������� n_joint_axis dim�� ���ܵ� �ɼ��� ȹ���Ѵ�.
		doffset2idx(dotv->noutPre, dotv->outRankPre, poff, tmp_idx);//�ֻ��� ��������̳� �����߿��� ������� �ε��� �Ի�
		for(j = 0;j < dotv->noutPre; j++) pidx[dotv->outAxisPre[j]] = tmp_idx[j];//��ü �ε����� ������� �ε��� ����.
		soff = roff % sz_suf_shrink;//suffix��Ʈ�������� n_joint_axis dim�� ���ܵ� �ɼ��� ���
		doffset2idx(dotv->noutSuf, dotv->outRankSuf, soff, tmp_idx);
		for(j = 0;j < dotv->noutSuf; j++) sidx[dotv->outAxisSuf[j]] = tmp_idx[j];
	}
	//������ ���� ȹ��� ��Ʈ���� �ε������� n_joint_axis������ �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����.
	for(i = 0, sum = 0;i < n_joint_axis; i++) {
		doffset2idx(dotv->njoPre, dotv->joRankPre, i, tmp_idx);
		for(j = 0;j < dotv->njoPre; j++) pidx[dotv->joAxisPre[j]] = tmp_idx[j];
		doffset2idx(dotv->njoSuf, dotv->joRankSuf, i, tmp_idx);
		for(j = 0;j < dotv->njoSuf; j++) sidx[dotv->joAxisSuf[j]] = tmp_idx[j];
		poff = didx2offset(pdim, prank, pidx);
		soff = didx2offset(sdim, srank, sidx);
		//printf("%f %f\n", *(m_pdot + poff), *(m_sdot + soff));
		sum += *(m_pdot + poff) * *(m_sdot + soff);
	}
	if(dotv->bwGetOri) {
		if(rplus != 1) *(m_rdot + roff) *= rplus;
	} else {
		if(rplus == 0) *(m_rdot + roff) = 0;
		else *(m_rdot + roff) *= rplus;
	}
	*(m_rdot + roff) += sum;
	//printf("%d %f\n", roff, *(m_rdot + roff));
}*/
/*
template<typename T>
__global__ void kdot_f2(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt range, intt n)
{//�׸��� ��� �ȴ� ����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x * range + threadIdx.x * range;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;
	if(n > roff + range) n = roff + range;

	intt ridx[MX_DIM], tmp_idx[MX_DIM], soff;
	intt rdim = dotv->ndimRet, *out_rank = dotv->outRank;
	intt *rrank = dotv->szRankRet;
	intt nout_pre = dotv->noutPre, nout = dotv->noutRank;
	intt njo_pre = dotv->njoPre, njo_suf = dotv->njoSuf;
	intt nout_suf = dotv->noutSuf, njo_ret = dotv->njoRet, *out_axis_ret = dotv->outAxisRet;
	intt *jo_axis_ret = dotv->joAxisRet;
	intt *out_rank_pre = dotv->outRankPre, *out_rank_suf = dotv->outRankSuf, *Jo_rank_ret = dotv->joRankRet;
	const bool bw_get_ori = dotv->bwGetOri, jdim_equal = dotv->jdimEqual, interv_out = dotv->intervOut, ret_first = dotv->retFirst;
	SparseRank *spr_pre_out = dotv->sprPreOut, *spr_pre_jo = dotv->sprPreJo, *spr_suf_out = dotv->sprSufOut, *spr_suf_jo = dotv->sprSufJo;
	intt po_idx[MX_DIM], pj_idx[MX_DIM], so_idx[MX_DIM], sj_idx[MX_DIM], i;
	T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx2(nout_pre, nout, out_rank, roff, po_idx, so_idx);
	for(i = 0;i < nout_pre; i++) m_pdot += (po_idx[i] * spr_pre_out[i].rksz);
	for(i = 0;i < nout_suf; i++) m_sdot += (so_idx[i] * spr_suf_out[i].rksz);
	if(jdim_equal) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < njo_pre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < njo_pre; i++) pj_idx[i] = 0;
		for(i = 0;i < njo_suf; i++) sj_idx[i] = 0;
	}
	const intt nout_pre2 = nout_pre - 1, nout_suf2 = nout_suf - 1, njo_pre2 = njo_pre - 1, njo_suf2 = njo_suf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	const intt podim = spr_pre_out[nout_pre2].rkdim, posz = spr_pre_out[nout_pre2].rksz;
	const intt sodim = spr_suf_out[nout_suf2].rkdim, sosz = spr_suf_out[nout_suf2].rksz;
	intt i_pj = 0, i_sj = 0, i_po = po_idx[nout_pre2], i_so = so_idx[nout_suf2];
	for(;roff < n; roff++) {
		for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%f %f\n", *(m_pdot + poff), *(m_sdot + soff));
			sum += *m_pdot * *m_sdot;
			if(jdim_equal) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J0:;					m_pdot -= spr_pre_jo[i].rktsz;
						m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {//�߰� ���� ����
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J0;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								m_sdot += spr_suf_jo[i].rksz;
								break;
							}
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J1:;					m_pdot -= spr_pre_jo[i].rktsz;
						if(--i < 0) break;
						else {
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J1;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						i_sj = 0;
						i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {
							if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
								sj_idx[i] = 0;
								goto J2;
							} else {
								m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(bw_get_ori) {//������, �����Ķ��� ������ ��� axis�� interval�� ����ų� ������ �����ǰų� �������� ������ Ʋ���� 
			if(interv_out) {//��� ��Ʈ������ ret��Ʈ������ out axis�� �����Ǵ� ��ũ�� Ʋ�����ְ� out axis rank���� idx��
				//ret��Ʈ���� ��ũ�� ����Ͽ� ret��Ʈ���� �ɼ����� ��ȯ�Ѵ�. �ɼº�ȯ�ǹǷ� �����Ͽ� �ѹ��� device mem���� �����Ѵ�.
				po_idx[nout_pre2] = i_po;//������ �ؿ��� �ε��� �������� �����Ƿ� ���⼭ ����
				so_idx[nout_suf2] = i_so;
				if(ret_first) {//ret��Ʈ������ dot�� ù��°�� ��ġ�ϴ� ���(�����Ķ� �ش� ��Ʈ����(pre or suf)�� first�� ���ε��� ���� ���
					for(i = 0;i < nout_pre; i++) {//pref out idx�� ret��Ʈ������ ��� axis idx�� ����
						ridx[out_axis_ret[i]] = po_idx[i];
					}
					if(njo_ret == 1 && nout_suf == 1) ridx[jo_axis_ret[0]] = so_idx[0];//��ȯ���� �ٷ� ����
					else {//suf out idx�� ret��Ʈ������ ���� axis�� �ȴ�. suf out idx�� suf out��ũ�� ����Ͽ� �ɼ����� ��ȯ�� 
						_sparse_idx2offset(nout_suf, out_rank_suf, so_idx, soff);//�� �ɼ���
						_offset2idx(njo_ret, Jo_rank_ret, soff, tmp_idx);//ret��Ʈ������ ���� ��ũ�� ����Ͽ� ���� axis��ġ��
						for(i = 0;i < njo_ret; i++) ridx[jo_axis_ret[i]] = tmp_idx[i];//�ε����Ѵ�.
					}
				} else {
					for(i = 0;i < nout_suf; i++) {//suf out idx�� ret��Ʈ������ ��� axis idx�� ����
						ridx[out_axis_ret[i]] = so_idx[i];
					}
					if(njo_ret == 1 && nout_pre == 1) ridx[jo_axis_ret[0]] = po_idx[0];//��ȯ���� �ٷ� ����
					else {//pref out idx�� ret��Ʈ������ ���� axis�� �ȴ�. pref out idx�� pref out��ũ�� ����Ͽ� �ɼ����� ��ȯ�� 
						_sparse_idx2offset(nout_pre, out_rank_pre, po_idx, soff);//�� �ɼ���
						_offset2idx(njo_ret, Jo_rank_ret, soff, tmp_idx);//ret��Ʈ������ ���� ��ũ�� ����Ͽ� ���� axis��ġ��
						for(i = 0;i < njo_ret; i++) ridx[jo_axis_ret[i]] = tmp_idx[i];//�ε����Ѵ�.
					}
				}
				_idx2offset(rdim, rrank, ridx, soff);//���� ret��Ʈ������ �ε��̵� idx�� ret��Ʈ���� �ɼ����� ��ȯ�Ѵ�.
				if(rplus != 1) *(m_rdot + soff) *= rplus;
				*(m_rdot + soff) += sum;
			} else {//���� ��ȯ���� �ٷ� �����Ѵ�.
				if(rplus != 1) *(m_rdot + roff) *= rplus;
				*(m_rdot + roff) += sum;
			}
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
		for(;;) {//��� ���� �ε����� ���� ���� suf out ���� �ε��� ����
			if(sodim == ++i_so) {
				i_so = 0;//suf out ���� ���� ���� �ø�
				i = nout_suf2;
O1:				m_sdot -= spr_suf_out[i].rktsz;
				if(--i < 0) break;//���� ���� ������
				else {
					if(spr_suf_out[i].rkdim == ++so_idx[i]) {
						so_idx[i] = 0;//suf out �߰� ���� ���� �ø�
						goto O1;
					} else {
						m_sdot += spr_suf_out[i].rksz;//���� ���� ����(����) �ּҰ� ����
						goto LA;//suf out �߰� ���� ���� ����
					}
				}
			} else {
				m_sdot += sosz;//���� ����(����) �ּҰ� ����
				goto LA;//suf out ���� ���� ����
			}
		}
		for(;;) {//��� ���� �ε����� ���� ���� pre out ���� �ε��� ����
			if(podim == ++i_po) {
				i_po = 0;
				i = nout_pre2;
O2:				m_pdot -= spr_pre_out[i].rktsz;
				if(--i < 0) goto LB2;
				else {
					if(spr_pre_out[i].rkdim == ++po_idx[i]) {
						po_idx[i] = 0;
						goto O2;
					} else {
						m_pdot += spr_pre_out[i].rksz;//���� ���� ����(����)�� ����
						break;//suf out �߰� ���� ���� ����
					}
				}
			} else {
				m_pdot += posz;
				break;
			}
		}
LA:;
	}
LB2:;
}
template<typename T>
intt gdot_t2(void *pcxt_dev, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));//dot t2���� ����
	intt range = idx_width / block.x;
	dim3 grid(idx_width % (block.x * range) ? 2 : 1);//idx_origin�� �� ������ ������ �ε����ϹǷ� �׸���� 1���̴�.
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f2<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, range, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
template<typename T>
__global__ void kdot_f2(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//�׸��� ��� ����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	intt ridx[MX_DIM], tmp_idx[MX_DIM], soff;
	intt rdim = dotv->ndimRet, *out_rank = dotv->outRank;
	intt *rrank = dotv->szRankRet;
	intt nout_pre = dotv->noutPre, nout = dotv->noutRank;
	intt njo_pre = dotv->njoPre, njo_suf = dotv->njoSuf;
	intt nout_suf = dotv->noutSuf, njo_ret = dotv->njoRet, *out_axis_ret = dotv->outAxisRet;
	intt *jo_axis_ret = dotv->joAxisRet;
	intt *out_rank_pre = dotv->outRankPre, *out_rank_suf = dotv->outRankSuf, *Jo_rank_ret = dotv->joRankRet;
	const bool bw_get_ori = dotv->bwGetOri, jdim_equal = dotv->jdimEqual, interv_out = dotv->intervOut, ret_first = dotv->retFirst;
	SparseRank *spr_pre_out = dotv->sprPreOut, *spr_pre_jo = dotv->sprPreJo, *spr_suf_out = dotv->sprSufOut, *spr_suf_jo = dotv->sprSufJo;
	intt po_idx[MX_DIM], pj_idx[MX_DIM], so_idx[MX_DIM], sj_idx[MX_DIM], i;
	T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx2(nout_pre, nout, out_rank, roff, po_idx, so_idx);
	for(i = 0;i < nout_pre; i++) m_pdot += (po_idx[i] * spr_pre_out[i].rksz);
	for(i = 0;i < nout_suf; i++) m_sdot += (so_idx[i] * spr_suf_out[i].rksz);
	if(jdim_equal) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < njo_pre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < njo_pre; i++) pj_idx[i] = 0;
		for(i = 0;i < njo_suf; i++) sj_idx[i] = 0;
	}
	const intt nout_pre2 = nout_pre - 1, nout_suf2 = nout_suf - 1, njo_pre2 = njo_pre - 1, njo_suf2 = njo_suf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt i_pj = 0, i_sj = 0, i_po = po_idx[nout_pre2], i_so = so_idx[nout_suf2];
	for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
		//printf("%f %f\n", *(m_pdot + poff), *(m_sdot + soff));
		sum += *m_pdot * *m_sdot;
		if(jdim_equal) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J0:;				m_pdot -= spr_pre_jo[i].rktsz;
					m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {//�߰� ���� ����
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J0;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							m_sdot += spr_suf_jo[i].rksz;
							break;
						}
					}
				} else {//���� ���� ����
					m_pdot += pjsz;//�� ����(����)�� ����
					m_sdot += sjsz;
					break;
				}
			}
		} else {
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J1:;				m_pdot -= spr_pre_jo[i].rktsz;
					if(--i < 0) break;
					else {
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J1;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_pdot += pjsz;//�� ����(����)�� ����
					break;
				}
			}
			for(;;) {//suf �������� �ε��� ����
				if(sjdim == ++i_sj) {
					i_sj = 0;
					i = njo_suf2;
J2:;				m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {
						if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
							sj_idx[i] = 0;
							goto J2;
						} else {
							m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_sdot += sjsz;
					break;
				}
			}
		}
	}
LB1:;
	if(bw_get_ori) {//������, �����Ķ��� ������ ��� axis�� interval�� ����ų� ������ �����ǰų� �������� ������ Ʋ���� 
		if(interv_out) {//��� ��Ʈ������ ret��Ʈ������ out axis�� �����Ǵ� ��ũ�� Ʋ�����ְ� out axis rank���� idx��
			//ret��Ʈ���� ��ũ�� ����Ͽ� ret��Ʈ���� �ɼ����� ��ȯ�Ѵ�. �ɼº�ȯ�ǹǷ� �����Ͽ� �ѹ��� device mem���� �����Ѵ�.
			po_idx[nout_pre2] = i_po;//������ �ؿ��� �ε��� �������� �����Ƿ� ���⼭ ����
			so_idx[nout_suf2] = i_so;
			if(ret_first) {//ret��Ʈ������ dot�� ù��°�� ��ġ�ϴ� ���(�����Ķ� �ش� ��Ʈ����(pre or suf)�� first�� ���ε��� ���� ���
				for(i = 0;i < nout_pre; i++) {//pref out idx�� ret��Ʈ������ ��� axis idx�� ����
					ridx[out_axis_ret[i]] = po_idx[i];
				}
				if(njo_ret == 1 && nout_suf == 1) ridx[jo_axis_ret[0]] = so_idx[0];//��ȯ���� �ٷ� ����
				else {//suf out idx�� ret��Ʈ������ ���� axis�� �ȴ�. suf out idx�� suf out��ũ�� ����Ͽ� �ɼ����� ��ȯ�� 
					_sparse_idx2offset(nout_suf, out_rank_suf, so_idx, soff);//�� �ɼ���
					_offset2idx(njo_ret, Jo_rank_ret, soff, tmp_idx);//ret��Ʈ������ ���� ��ũ�� ����Ͽ� ���� axis��ġ��
					for(i = 0;i < njo_ret; i++) ridx[jo_axis_ret[i]] = tmp_idx[i];//�ε����Ѵ�.
				}
			} else {
				for(i = 0;i < nout_suf; i++) {//suf out idx�� ret��Ʈ������ ��� axis idx�� ����
					ridx[out_axis_ret[i]] = so_idx[i];
				}
				if(njo_ret == 1 && nout_pre == 1) ridx[jo_axis_ret[0]] = po_idx[0];//��ȯ���� �ٷ� ����
				else {//pref out idx�� ret��Ʈ������ ���� axis�� �ȴ�. pref out idx�� pref out��ũ�� ����Ͽ� �ɼ����� ��ȯ�� 
					_sparse_idx2offset(nout_pre, out_rank_pre, po_idx, soff);//�� �ɼ���
					_offset2idx(njo_ret, Jo_rank_ret, soff, tmp_idx);//ret��Ʈ������ ���� ��ũ�� ����Ͽ� ���� axis��ġ��
					for(i = 0;i < njo_ret; i++) ridx[jo_axis_ret[i]] = tmp_idx[i];//�ε����Ѵ�.
				}
			}
			_idx2offset(rdim, rrank, ridx, soff);//���� ret��Ʈ������ �ε��̵� idx�� ret��Ʈ���� �ɼ����� ��ȯ�Ѵ�.
			if(rplus) *(m_rdot + soff) += sum;
			else *(m_rdot + soff) = sum;
		} else {//���� ��ȯ���� �ٷ� �����Ѵ�.
			if(rplus) *(m_rdot + roff) += sum;
			else *(m_rdot + roff) = sum;
		}
	} else {
		if(rplus) *(m_rdot + roff) += sum;
		else *(m_rdot + roff) = sum;
	}
}
template<typename T>
intt gdot_t2(void *pcxt_dev, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));//dot t2���� ����
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f2<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}
intt gdot_f2(void *pcxt_dev, floatt *pdot_mdev, floatt *sdot_mdev, floatt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, floatt rplus)
{
	return gdot_t2<floatt>(pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
intt gdot_f2(void *pcxt_dev, intt *pdot_mdev, intt *sdot_mdev, intt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, intt rplus)
{
	return gdot_t2<intt>(pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
