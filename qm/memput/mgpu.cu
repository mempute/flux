#include "hip/hip_runtime.h"

#include "mgpu.h"
#include "matrix.h"
#include <stdio.h>
#include <assert.h>
#include <npp.h>

void cudaerror(intt ecd, const bytet *emsg)
{
	hipError_t error = (hipError_t)ecd;
	if(error != hipSuccess) {
		bytet ebuf[1024];
		sprintf(ebuf, "cuda error throw: %d %s\n %s\n", ecd, hipGetErrorString(error), emsg);
		throwFault(-1, ebuf);
	}
}
void cuda_error_check(intt ecd) //-27���� ������
{
	hipError_t error = hipGetLastError();
	if(error != hipSuccess) {
		printf("cuda error: %d %s\n", ecd, hipGetErrorString(error));
		*(bytex *)0 = 1;
		throwFault(ecd, hipGetErrorString(error));
	}
}
void CudaDevSet(intt gid)
{
	//printf("cuda device set %d\n", gid);
	hipError_t error = hipSetDevice(gid);
	if(error != hipSuccess) {
		printf("cuda error: %s\n", hipGetErrorString(error));
		throwFault(-1, hipGetErrorString(error));
	}
}
size_t getmfreegpu(intt gid)
{
	size_t free_t, total_t;

	if(gid >= 0) CudaDevSet(gid);//�� �Լ��� �׷��� ���� ������� ��� �����Ǵ� ���������� ȣ��ȴ�.
	hipMemGetInfo(&free_t, &total_t);

	return free_t;
}
__device__ intt didx2offset(intt ndim, intt *srank, intt *idx)
{
	intt off = 0, j = 0;

	for(intt i = 1;i < ndim; i++) {
		off += MRANK_SIZE(srank, i) * *(idx + j++);
	}
	off += *(idx + j);

	return off;
}
__device__ void doffset2idx2(intt out_axis[], intt n_preout_axis, intt rdim, intt rrank[], intt off, intt pidx[], intt sidx[])
{
	intt j = 0, k;

	for(intt i = 1;i < rdim; i++, j++) {//�ش� �ε����� �ϳ� �Ʒ� ��ũ�� ������� ���� ���̹Ƿ� i�� 1����
		k = out_axis[j];
		if(j < n_preout_axis) {
			if(rrank[j] < 0) pidx[k] = 0;
			else {
				pidx[k] = off / MRANK_SIZE(rrank, i);
				off %= MRANK_SIZE(rrank, i);
			}
		} else {
			if(rrank[j] < 0) sidx[k] = 0;
			else {
				sidx[k] = off / MRANK_SIZE(rrank, i);
				off %= MRANK_SIZE(rrank, i);
			}
		}
	}
	sidx[out_axis[j]] = off;//������ �ε����� suf matrix�� ���� �ǰ� ���� �ɼ��� �ȴ�.
}
__device__ intt dsparse_idx2offset(intt ndim, intt *srank, intt *idx, intt *axis)
{
	intt off = 0, j = 0;

	for(intt i = 1;i < ndim; i++) {
		off += MRANK_SIZE(srank, i) * *(idx + *(axis + j++));
	}
	off += *(idx + *(axis + j));

	return off;
}
__device__ void doffset2idx(intt ndim, intt *srank, intt off, intt *idx)
{
	intt j = 0;

	for(intt i = 1;i < ndim; i++) {
		if(*(srank + j) < 0) *(idx + j++) = 0;
		else {
			*(idx + j++) = off / MRANK_SIZE(srank, i);
			off %= MRANK_SIZE(srank, i);
		}
	}
	*(idx + j) = off;
}
__device__ intt dmoff2soff(intt mdim, intt *mrank, intt sdim, intt *srank, intt moff, intt *idx)
{
	intt i = mdim - 1, j = sdim - 1;

	doffset2idx(mdim, mrank, moff, idx);
	for(;j >= 0; i--, j--) {
		if(*(srank + j) < 0) *(idx + i) = 0;
	}
	return didx2offset(sdim, srank, idx + ++i);
}
__device__ void dlead_offset2idx(intt nbro, intt cdim, intt ndim, intt *srank, intt off, intt cidx[])
{
	if(nbro) {
		intt i = 0;
		for(;i < cdim - ndim; i++) cidx[i] = 0;
		doffset2idx(ndim, srank, off, &cidx[i]);
	} else cidx[0] = off;

	cidx[MX_DIM - 1] = 0;//bro_offset�� ��)���� ����üũ�� ���
}
__device__ intt dbro_offset(intt nbro, intt bro_dim[], intt bro_idx[], intt cdim, intt *crank, intt cidx[])
{
	intt i = nbro - 1, off;

	if(cidx[MX_DIM - 1] == 1) return -1;//��.

	if(nbro) {
		off = didx2offset(cdim, crank, cidx);

		for(;i >= 0; i--) {
			if(++cidx[bro_idx[i]] == bro_dim[i]) cidx[bro_idx[i]] = 0;
			else break;
		}
		if(i < 0) cidx[MX_DIM - 1] = 1;
		return off;
	} else {
		cidx[MX_DIM - 1] = 1;
		return cidx[0];
	}
}
/*__device__ intt count_over_axis(intt idx[], intt axis)
{
	if(axis == 0) return -1;

	intt n = 1;
	for(intt i = 0;i < axis; i++) n *= idx[i];
	return n;
}*/
template<typename T>
__global__ void ksplit_f(void *pcxt, T *m_split, intt pdim, intt sdim, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt *prank;
	T *rsplit_mdev, *rsplit_mdevs;
	ConcatVar *ccv = (ConcatVar *)pcxt;

	prank = P_LINK_VAR2(intt, pcxt, ccv->szRankPrimary);
	rsplit_mdevs = P_LINK_VAR2(T, pcxt, ccv->mptrDevSecondary);
	intt outer_sz = MRANK_SIZE(prank, axis), inner_sz = (axis == pdim - 1 ? 1 : MRANK_SIZE(prank, axis + 1)) * nstep;
	intt si, soff;

	soff = roff % outer_sz;//split�� ��ũ���� ���� �ɼ�
	if(nsplit > 0) {
		si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
		soff = (roff / outer_sz) * inner_sz + soff % inner_sz;//���ҹ�°���� �ɼ�
	} else if(nstep) {//each map
		si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
		intt *sbase = P_LINK_VAR2(intt, pcxt, ccv->sbaseCat);
		intt *sdim = P_LINK_VAR2(intt, pcxt, ccv->sdimCat);
		soff = (roff / outer_sz) * *(sdim + si) + (soff - *(sbase + si));//���ҹ�°���� �ɼ�
	} else {
		intt *sbase = P_LINK_VAR2(intt, pcxt, ccv->sbaseCat);
		intt *sdim = P_LINK_VAR2(intt, pcxt, ccv->sdimCat);
		for(si = 0;*(sbase + si) + *(sdim + si) <= soff; si++);//split�� ���� ���� �̹� ���ҹ�° find
		soff = (roff / outer_sz) * *(sdim + si) + (soff - *(sbase + si));//split�� ���� ���� �̹� ���ҹ�°���� �ɼ�
	}
	rsplit_mdev = *((T **)rsplit_mdevs + si);//�̹� ���� �޸�

	if(bw) *(rsplit_mdev + soff) += *(m_split + roff);
	else *(rsplit_mdev + soff) = *(m_split + roff);
	//printf("(%p) %d %d %f %f\n", rsplit_mdev, roff, soff, *(rsplit_mdev + soff), *(m_split + roff));
}

template<typename T>
intt gsplit_t(void *pcxt, T *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rsize > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rsize);

	ksplit_f<T> << <grid, block >> > (pcxt, m_split, pdim, sdim, idx_origin, idx_width, nsplit, nstep, axis, bw, n);
	
	hipDeviceSynchronize();
	cuda_error_check(-2);
	return n - idx_origin * idx_width;
}
intt gsplit_f(void *pcxt, floatt *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt nsplit, intt nstep, intt axis, bool bw)
{
	return gsplit_t<floatt>(pcxt, m_split, pdim, sdim, rsize, idx_origin, idx_width, nsplit, nstep, axis, bw);
}
intt gsplit_f(void *pcxt, intt *m_split, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width, 
	intt nsplit, intt nstep, intt axis, bool bw)
{
	return gsplit_t<intt>(pcxt, m_split, pdim, sdim, rsize, idx_origin, idx_width, nsplit, nstep, axis, bw);
}

template<typename T>
__global__ void kconcat_f(void *pcxt, T *m_rcat, intt pdim, intt sdim, intt idx_origin, intt idx_width, 
	intt ncat, intt nstep, intt axis, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt *prank;
	T *pcat_mdev, *pcat_mdevs;
	ConcatVar *ccv = (ConcatVar *)pcxt;

	prank = P_LINK_VAR2(intt, pcxt, ccv->szRankPrimary);
	pcat_mdevs = P_LINK_VAR2(T, pcxt, ccv->mptrDevSecondary);
	intt outer_sz = MRANK_SIZE(prank, axis), inner_sz = (axis == pdim - 1 ? 1 : MRANK_SIZE(prank, axis + 1)) * nstep;
	intt si, soff;

	soff = roff % outer_sz;//split�� ��ũ���� ���� �ɼ�
	if(ncat > 0) {
		si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
		soff = (roff / outer_sz) * inner_sz + soff % inner_sz;//���ҹ�°���� �ɼ�
	} else if(nstep) {//each map
		si = soff / inner_sz;//split�� ���� ���� �̹� ���ҹ�°
		intt *sbase = P_LINK_VAR2(intt, pcxt, ccv->sbaseCat);
		intt *sdim = P_LINK_VAR2(intt, pcxt, ccv->sdimCat);
		soff = (roff / outer_sz) * *(sdim + si) + (soff - *(sbase + si));//���ҹ�°���� �ɼ�
	} else {
		intt *sbase = P_LINK_VAR2(intt, pcxt, ccv->sbaseCat);
		intt *sdim = P_LINK_VAR2(intt, pcxt, ccv->sdimCat);
		for(si = 0;*(sbase + si) + *(sdim + si) <= soff; si++);//split�� ���� ���� �̹� ���ҹ�° find
		//intt save = soff;
		soff = (roff / outer_sz) * *(sdim + si) + (soff - *(sbase + si));//split�� ���� ���� �̹� ���ҹ�°���� �ɼ�
		//pcat_mdev = *((T **)pcat_mdevs + si);
		//printf("%d %d %d %d %d [%f]\n", save, si, *(sbase + si), *(sdim + si), soff, *(pcat_mdev + soff));
	}
	pcat_mdev = *((T **)pcat_mdevs + si);
	if(bw) *(m_rcat + roff) += *(pcat_mdev + soff);
	else *(m_rcat + roff) = *(pcat_mdev + soff);
	//printf("[%d](%p) %d %d %d %d %f %f\n", roff, pcat_mdev, i, is, idx[axis], cat_off, *(pcat_mdev + cat_off), *(m_rcat + roff));
}
template<typename T>
intt gconcat_t(void *pcxt, T *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width, 
	intt ncat, intt nstep, intt axis, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rsize > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rsize);

	kconcat_f<T> << <grid, block >> > (pcxt, m_rcat, pdim, sdim, idx_origin, idx_width, ncat, nstep, axis, bw, n);
	hipDeviceSynchronize();
	cuda_error_check(-3);

	return n - idx_origin * idx_width;
}
intt gconcat_f(void *pcxt, floatt *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt ncat, intt nstep, intt axis, bool bw)
{
	return gconcat_t<floatt>(pcxt, m_rcat, pdim, sdim, rsize, idx_origin, idx_width, ncat, nstep, axis, bw);
}
intt gconcat_f(void *pcxt, intt *m_rcat, intt pdim, intt sdim, intt rsize, intt idx_origin, intt idx_width,
	intt ncat, intt nstep, intt axis, bool bw)
{
	return gconcat_t<intt>(pcxt, m_rcat, pdim, sdim, rsize, idx_origin, idx_width, ncat, nstep, axis, bw);
}
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt range, intt n)
{//������� ���� ���� ��� ���� ������ ����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x * range + threadIdx.x * range;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;
	if(n > roff + range) n = roff + range;
	
	intt *out_rank = dotv->outRank, nout = dotv->noutRank;
	intt njo_pre = dotv->njoPre, njo_suf = dotv->njoSuf;
	const bool bw_get_ori = dotv->bwGetOri, jdim_equal = dotv->jdimEqual;
	SparseRank *spr_out = dotv->sprPreOut, *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], ret_idx[MX_DIM], i;
	register T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx(nout, out_rank, roff, ret_idx);
	for(i = 0;i < nout; i++) {
		if(spr_out[i].rkPref) m_pdot += (ret_idx[i] * spr_out[i].rksz);
		else m_sdot += (ret_idx[i] * spr_out[i].rksz);
	}
	if(jdim_equal) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < njo_pre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < njo_pre; i++) pj_idx[i] = 0;
		for(i = 0;i < njo_suf; i++) sj_idx[i] = 0;
	}
	nout--;
	const intt njo_pre2 = njo_pre - 1, njo_suf2 = njo_suf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	const intt podim = spr_out[nout].rkdim, posz = spr_out[nout].rksz;
	bool lastout_is_pref = spr_out[nout].rkPref;
	intt i_po = ret_idx[nout], i_pj = 0, i_sj = 0;
	
	for(;roff < n; roff++) {
		for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%f %f\n", *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(jdim_equal) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J0:;					m_pdot -= spr_pre_jo[i].rktsz;
						m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {//�߰� ���� ����
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J0;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								m_sdot += spr_suf_jo[i].rksz;
								break;
							}
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J1:;					m_pdot -= spr_pre_jo[i].rktsz;
						if(--i < 0) break;
						else {
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J1;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						i_sj = 0;
						i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {
							if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
								sj_idx[i] = 0;
								goto J2;
							} else {
								m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(bw_get_ori) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
		for(;;) {//��� ���� �ε��� ����(pre�� suf�� ��� �� �����Ͽ�)
			if(podim == ++i_po) {//podim�� pre�� suf�� ���� �������� �� ������ ��ġ�ϴ� ���� �����, i_po, posz, lastout_is_pref�� ��������)
				i_po = 0;
				i = nout;
O2:				if(spr_out[i].rkPref) m_pdot -= spr_out[i].rktsz;
				else m_sdot -= spr_out[i].rktsz;
				if(--i < 0) goto LB2;
				else {
					if(spr_out[i].rkdim == ++ret_idx[i]) {
						ret_idx[i] = 0;
						goto O2;
					} else {
						if(spr_out[i].rkPref) m_pdot += spr_out[i].rksz;//���� ���� ����(����)�� ����
						else m_sdot += spr_out[i].rksz;
						break;//suf out �߰� ���� ���� ����
					}
				}
			} else {
				if(lastout_is_pref) m_pdot += posz;
				else m_sdot += posz;
				break;
			}
		}
	}
LB2:;
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
			intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));//�� dot�� BLOCK_SIZE �̴� idx_width�̴� 512�� �Ѿ�� �� ���� ���� ������ ������
	intt range = idx_width / block.x;//512�� �Ѿ�� ���� �޸�(����) ����� �Ѱ� �ʰ��Ǿ� ������� �ʴ´�. 
	dim3 grid(idx_width % (block.x * range) ? 2 : 1);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, range, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *m_pdot, T *m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//�������� one thread one out����, ���� grid����
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], i;
	register T sum;
	//�� roff�� ret��Ʈ������ ������ �ɼ��̰� �̰��� �̹� ��Ʈ���� ���� ���� out axis rank�������� ��ȯ�Ѵ�.
	_offset2idx(dotv->noutRank, dotv->outRank, roff, pj_idx);
	for(i = 0;i < dotv->noutRank; i++) {
		if(dotv->sprPreOut[i].rkPref) m_pdot += (pj_idx[i] * dotv->sprPreOut[i].rksz);
		else m_sdot += (pj_idx[i] * dotv->sprPreOut[i].rksz);
	}
	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt i_pj = 0, i_sj = 0;

	for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
		//printf("%f %f\n", *m_pdot, *m_sdot);
		sum += *m_pdot * *m_sdot;
		if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J0:;				m_pdot -= spr_pre_jo[i].rktsz;
					m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {//�߰� ���� ����
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J0;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							m_sdot += spr_suf_jo[i].rksz;
							break;
						}
					}
				} else {//���� ���� ����
					m_pdot += pjsz;//�� ����(����)�� ����
					m_sdot += sjsz;
					break;
				}
			}
		} else {
			for(;;) {//pre �������� �ε��� ����
				if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
					i_pj = 0;
					i = njo_pre2;
J1:;				m_pdot -= spr_pre_jo[i].rktsz;
					if(--i < 0) break;
					else {
						if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
							pj_idx[i] = 0;
							goto J1;
						} else {
							m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_pdot += pjsz;//�� ����(����)�� ����
					break;
				}
			}
			for(;;) {//suf �������� �ε��� ����
				if(sjdim == ++i_sj) {
					i_sj = 0;
					i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
					if(--i < 0) goto LB1;
					else {
						if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
							sj_idx[i] = 0;
							goto J2;
						} else {
							m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
							break;
						}
					}
				} else {
					m_sdot += sjsz;
					break;
				}
			}
		}
	}
LB1:;
	if(dotv->bwGetOri) {//������
		if(rplus != 1) *(m_rdot + roff) *= rplus;
		*(m_rdot + roff) += sum;
	} else {
		if(rplus == 0) *(m_rdot + roff) = 0;
		else *(m_rdot + roff) *= rplus;
		*(m_rdot + roff) += sum;
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);

	kdot_f<T> << <grid, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//���� �� �޸� ���� ������ ����: ȿ������, ���߿� Ȥ�� ����ҷ��� jdimEqual�ƴҶ� ��� Ʋ���Ƿ� jdimEqual�϶��� 
	//�������� �����ϰ� jdimEqual�ƴҶ��� ���� ��ȷ���ϰ� ���� �ڵ� ����, ����� ������ �׽�Ʈ ������.
	//__shared__ T cache_bank[SM_SIZE];
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], tmp_idx[MX_DIM], i, i_pj, i_sj;
	bool pover = 0, sover = 0;
	register T sum;

	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	T *m_pdot, *m_sdot;
	for(;roff < n; roff += blockDim.x) {
		//__syncthreads();
		_offset2idx(dotv->noutRank, dotv->outRank, roff, tmp_idx);
		for(i = 0, m_pdot = _m_pdot, m_sdot = _m_sdot;i < dotv->noutRank; i++) {
			if(dotv->sprPreOut[i].rkPref) m_pdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
			else m_sdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
		}
		i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
		if(i_pj == 0) pover = 1;
		if(dotv->jdimEqual) i_sj = i_pj;
		else {
			i_sj = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
			if(i_sj == 0) sover = 1;
		}
		m_pdot += i_pj * pjsz;
		m_sdot += i_sj * sjsz;
		for(sum = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//if(threadIdx.x == 2) printf("%d %d %d %f %f\n", threadIdx.x, i_pj, pjdim, *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						if(pover) {
							m_pdot += (pjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							m_sdot += (sjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							pjdim = spr_pre_jo[njo_pre2].rkdim;
							pover = 0;
							i = njo_pre2;
J0:;						m_pdot -= spr_pre_jo[i].rktsz;
							m_sdot -= spr_suf_jo[i].rktsz;
							if(--i < 0) goto LB1;
							else {//�߰� ���� ����
								if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
									pj_idx[i] = 0;
									goto J0;
								} else {
									i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
									if(i_pj == 0) pover = 1;
									m_pdot += (spr_pre_jo[i].rksz + i_pj * pjsz);//���� ���� ����(����)�� ����
									m_sdot += (spr_suf_jo[i].rksz + i_pj * sjsz);
									break;
								}
							}
						} else {
							i_pj = 0;
							pover = 1;
							pjdim = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
							m_pdot -= spr_pre_jo[njo_pre2].rktsz;//m_pdot -= (spr_pre_jo[njo_pre2].rktsz - (pjdim * pjsz));
							m_sdot -= spr_suf_jo[njo_pre2].rktsz;//m_sdot -= (spr_suf_jo[njo_pre2].rktsz - (pjdim * sjsz));
							break;
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						if(pover) {
							m_pdot += (pjsz * (spr_pre_jo[njo_pre2].rkdim - pjdim));
							pjdim = spr_pre_jo[njo_pre2].rkdim;
							pover = 0;
							i = njo_pre2;
J1:;						m_pdot -= spr_pre_jo[i].rktsz;
							if(--i < 0) break;
							else {
								if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
									pj_idx[i] = 0;
									goto J1;
								} else {
									i_pj = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
									if(i_pj == 0) pover = 1;
									m_pdot += (spr_pre_jo[i].rksz + i_pj * pjsz);//���� ���� ����(����)�� ����
									break;
								}
							}
						} else {
							i_pj = 0;
							pover = 1;
							pjdim = (blockDim.x < pjdim ? threadIdx.x : threadIdx.x % pjdim);
							m_pdot -= spr_pre_jo[njo_pre2].rktsz;//m_pdot -= (spr_pre_jo[njo_pre2].rktsz - (pjdim * pjsz));
							break;
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						if(sover) {
							m_sdot += (sjsz * (spr_suf_jo[njo_suf2].rkdim - sjdim));
							sjdim = spr_suf_jo[njo_suf2].rkdim;
							sover = 0;
							i = njo_suf2;
J2:;						m_sdot -= spr_suf_jo[i].rktsz;
							if(--i < 0) goto LB1;
							else {
								if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
									sj_idx[i] = 0;
									goto J2;
								} else {
									i_sj = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
									if(i_sj == 0) sover = 1;
									m_sdot += (spr_suf_jo[i].rksz + i_sj * sjsz);//���� ���� ����(����)�� ����
									break;
								}
							}
						} else {
							i_sj = 0;
							sover = 1;
							sjdim = (blockDim.x < sjdim ? threadIdx.x : threadIdx.x % sjdim);
							m_sdot -= spr_suf_jo[njo_suf2].rktsz;//m_sdot -= (spr_suf_jo[njo_suf2].rktsz - (sjdim * sjsz));
							break;
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ�� ����.
	//__shared__ T cache_bank[SM_SIZE];
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("%d %d %d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if(roff >= n) return;

	SparseRank *spr_pre_jo = dotv->sprPreJo, *spr_suf_jo = dotv->sprSufJo;
	const intt njo_pre2 = dotv->njoPre - 1, njo_suf2 = dotv->njoSuf - 1;
	const intt pjdim = spr_pre_jo[njo_pre2].rkdim, pjsz = spr_pre_jo[njo_pre2].rksz;
	const intt sjdim = spr_suf_jo[njo_suf2].rkdim, sjsz = spr_suf_jo[njo_suf2].rksz;
	intt pj_idx[MX_DIM], sj_idx[MX_DIM], tmp_idx[MX_DIM], i, i_pj, i_sj;
	register T sum;

	if(dotv->jdimEqual) {//���� ���� ��ũ�� ������ �ѹ��� �ʱ�ȭ
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = sj_idx[i] = 0;
	} else {
		for(i = 0;i < dotv->njoPre; i++) pj_idx[i] = 0;
		for(i = 0;i < dotv->njoSuf; i++) sj_idx[i] = 0;
	}
	T *m_pdot, *m_sdot;
	for(;roff < n; roff += blockDim.x) {
		//__syncthreads();
		_offset2idx(dotv->noutRank, dotv->outRank, roff, tmp_idx);
		for(i = 0, m_pdot = _m_pdot, m_sdot = _m_sdot;i < dotv->noutRank; i++) {
			if(dotv->sprPreOut[i].rkPref) m_pdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
			else m_sdot += (tmp_idx[i] * dotv->sprPreOut[i].rksz);
		}
		for(sum = 0, i_pj = i_sj = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%f %f\n", *m_pdot, *m_sdot);
			sum += *m_pdot * *m_sdot;
			if(dotv->jdimEqual) {//���� ���� �ε����� ������ ��ǥ�� pre join�ε����� ����
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J0:;					m_pdot -= spr_pre_jo[i].rktsz;
						m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {//�߰� ���� ����
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J0;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								m_sdot += spr_suf_jo[i].rksz;
								break;
							}
						}
					} else {//���� ���� ����
						m_pdot += pjsz;//�� ����(����)�� ����
						m_sdot += sjsz;
						break;
					}
				}
			} else {
				for(;;) {//pre �������� �ε��� ����
					if(pjdim == ++i_pj) {//����(����) �ø��� ���ÿ� ���������� ���� ��� �ʱ�ȭ�� �ڵ����� �ȴ�.
						i_pj = 0;
						i = njo_pre2;
J1:;					m_pdot -= spr_pre_jo[i].rktsz;
						if(--i < 0) break;
						else {
							if(spr_pre_jo[i].rkdim == ++pj_idx[i]) {
								pj_idx[i] = 0;
								goto J1;
							} else {
								m_pdot += spr_pre_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_pdot += pjsz;//�� ����(����)�� ����
						break;
					}
				}
				for(;;) {//suf �������� �ε��� ����
					if(sjdim == ++i_sj) {
						i_sj = 0;
						i = njo_suf2;
J2:;					m_sdot -= spr_suf_jo[i].rktsz;
						if(--i < 0) goto LB1;
						else {
							if(spr_suf_jo[i].rkdim == ++sj_idx[i]) {
								sj_idx[i] = 0;
								goto J2;
							} else {
								m_sdot += spr_suf_jo[i].rksz;//���� ���� ����(����)�� ����
								break;
							}
						}
					} else {
						m_sdot += sjsz;
						break;
					}
				}
			}
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(rplus == 0) *(m_rdot + roff) = 0;
			else *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		}
	}
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
/*
template<typename T>
__global__ void kdot_f(void *pcxt, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ��, �����޸𸮻��, �Ѱ� �� ���� ����.
	__shared__ T _cache_bank[SM_SIZE];
	T *cache_bank;
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * blockDim.x + threadIdx.x;
	intt roff_ori = roff;
	//printf("qq %d %d %d %d %d %d\n", idx_width, idx_origin, blockIdx.x, blockDim.x, threadIdx.x, roff);
	if(roff >= n) return;

	SharePoint *sp = (SharePoint *)&_cache_bank[0];
	if(threadIdx.x == 0) {
		memcpy(sp->spr_out, dotv->sprPreOut, sizeof(SparseRank) * dotv->noutRank);
		sp->cache_axis = dotv->axisCache;
		sp->okfit = dotv->fitOutKernel;
		sp->jkfit = dotv->fitJoKernel;
		sp->oksz = blockDim.x;//��� ��Ʈ���� ũ�Ⱑ ��� Ŀ�� ������� ������ ��� ��Ʈ���� ũ��� ������.
		sp->jksz = dotv->szJoKernel;
		sp->nrecycc = dotv->nrecycCache;
		sp->ncycj = dotv->ncycJo;
		sp->njoint = dotv->nJointAxis;
		sp->nout = dotv->noutRank;
		memcpy(sp->out_rank, dotv->outRank, sizeof(intt) * dotv->noutRank);
		if(sp->spr_out[sp->cache_axis].rkPref) {//�����ϴ� �� �������� �ٷ����� (���)������ �� ������ ���� ���� ���ҵ��� 
			memcpy(sp->spr_jcache, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);	//�����޸𸮿� ĳ���Ͽ� �� ����(���)���� �Ѱ� ���Ҹ��� ���� ������ �����Ѵ�.
			memcpy(sp->jrank_cache, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_cache = dotv->njoPre;
			memcpy(sp->spr_jleaf, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);//��¿��� �����ϴ� �� ���� �ܸ� ����(�������� ������ pre�� suf�� �Ѱ� ����)
			memcpy(sp->jrank_leaf, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_leaf = dotv->njoSuf;
		} else {
			memcpy(sp->spr_jcache, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);
			memcpy(sp->jrank_cache, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_cache = dotv->njoSuf;
			memcpy(sp->spr_jleaf, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);
			memcpy(sp->jrank_leaf, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_leaf = dotv->njoPre;
		}
	}
	__syncthreads();
	//printf("zz %d %d\n", sp->jksz, dotv->szJoKernel);
	cache_bank = (T *)((bytet *)_cache_bank + sizeof(SharePoint));
	cache_bank = (T *)ALIGN_UNIT((divadx)cache_bank);
	const intt njo_leaf2 = sp->njo_leaf - 1;
	intt ljdim = sp->spr_jleaf[njo_leaf2].rkdim;
	const intt ljdim_ori = ljdim, ljsz = sp->spr_jleaf[njo_leaf2].rksz;
	intt leaf_idx[MX_DIM], tmp_idx[MX_DIM], i, i_lj, j;
	register T sum;

	T *m_pdot, *m_sdot, *m_cache, *m_leaf;
	intt inc = blockDim.x, itime_ok = sp->nrecycc, itime_jk = 0;//�ʱ� Ƚ���� ����.
	//intt iloop = 0;
	for(;; roff += inc) {
LP:;	//printf("aa %d %d %d\n", roff, itime_ok, sp->nrecycc);
		cache_load(n, roff_ori, roff, sp->okfit, sp->jkfit, sp->oksz, sp->jksz, sp->nrecycc, sp->ncycj, 
			sp->njoint, itime_ok, itime_jk, threadIdx.x, blockDim.x, inc, sp->cache_axis, sp->nout, sp->out_rank, tmp_idx,
			sp->spr_out, sp->spr_jcache, sp->spr_jleaf, sp->njo_cache, sp->jrank_cache, sp->njo_leaf, 
			sp->jrank_leaf,	_m_pdot, m_pdot, _m_sdot, m_sdot, m_cache, m_leaf, leaf_idx, cache_bank, LP);
		i_lj = leaf_idx[njo_leaf2];//�̹� ���� Ŀ�� �ָ��� ���� �ε���
		//printf("sss %d: %d %d %d\n", threadIdx.x, ljdim, i_lj);
		if(sp->jksz < ljdim - i_lj) i_lj = ljdim - sp->jksz;//���εǴ� ������ ����Ŀ�� ������(jksz)���� ���� ���� ��)���� üũ�ǹǷ� 
		//��������ʰ� ���λ���� Ŀ�� ���ҵǾ� jksz�� ���ϴܸ� �������� ������� ��)���� üũ�ǰ� ����.
		for(sum = 0, j = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%d: %d %d %f %f\n", threadIdx.x, sp->jksz, ljsz, *m_cache, *m_leaf);
			sum += *m_cache * *m_leaf;
			//printf("ss %d: %d %d\n", threadIdx.x, ljdim, i_lj);
			if(ljdim == ++i_lj) {//��.���� �ܸ� ���� �ε��� ����
				if(sp->jksz + leaf_idx[njo_leaf2] < (++j + 1) * ljdim) {//����Ŀ�λ���� �ʱ���� �������� �����Ͽ� ������ ���ϸ���
					i_lj = (j +1) * ljdim - (sp->jksz + leaf_idx[njo_leaf2]);//üũ������� ������ ���ϸ��� ������� 
					if(i_lj >= ljdim) break;									//������ŭ �����Ͽ� ������ ��)���� üũ�ǰ� �Ѵ�.
				} else i_lj = 0;
				i = njo_leaf2;
J2:;			m_leaf -= sp->spr_jleaf[i].rktsz;
				if(--i < 0) {
					//printf("vv %d: %d %d\n", threadIdx.x, ljdim, i_lj);
					goto LB1;//��.
				} else {
					if(sp->spr_jleaf[i].rkdim == ++leaf_idx[i]) {
						leaf_idx[i] = 0;
						goto J2;
					} else m_leaf += sp->spr_jleaf[i].rksz;//���� ���� ����(����)�� ����
				}
			} else m_leaf += ljsz;
			m_cache++;
		}
LB1:;
		if(dotv->bwGetOri) {//������
			if(itime_jk == 0 && rplus != 1) *(m_rdot + roff) *= rplus;
			*(m_rdot + roff) += sum;
		} else {
			if(itime_jk == 0) {
				if(rplus == 0) *(m_rdot + roff) = 0;
				else *(m_rdot + roff) *= rplus;
			}
			*(m_rdot + roff) += sum;
			//printf("%f\n", sum);
		}
		//iloop++;
	}
	//printf("## %d\n", iloop);
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(oksz));//������ ��� Ŀ�� ������� �� ����� �� ũ�� �������� �ʴ´�.
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <1, block >> > (pcxt_dev, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);

	hipDeviceSynchronize();
	return n - idx_origin * idx_width;
}*/
template<typename T>
__global__ void kdot_f(void *pcxt, intt share_unit, T *_m_pdot, T *_m_sdot, T *m_rdot, intt idx_origin, intt idx_width, T rplus, intt n)
{//������� ��� �� �Ѱ� �ּ� ������, ������� �� ���� ���� ���� ����, ��ǥ ��ȯ���� ȹ��, �����޸𸮻��, ���� �� ���� ����.
	__shared__ T _cache_bank[SM_SIZE];//���߿� �����޸� �뷮�� Ŀ���� m_leaf��Ʈ���� ��ä�� leaf_idx, tmp_idx�� �����޸𸮿� �����Ѵ�.
	T *cache_bank;
	DotVar *dotv = (DotVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x * share_unit + threadIdx.x;
	intt roff_ori = roff;
	//printf("qq %d %d %d %d %d %d\n", idx_width, idx_origin, blockIdx.x, blockDim.x, threadIdx.x, roff);
	if(roff >= n) return;
	
	share_unit = roff - threadIdx.x + share_unit;//share_unit�� �ٷ� �Ʒ��� ����ҷ��� ����, ���Ŀ��� ������ �����Ƿ�
	if(n > share_unit) n = share_unit;

	SharePoint *sp = (SharePoint *)&_cache_bank[0];
	if(threadIdx.x == 0) {
		memcpy(sp->spr_out, dotv->sprPreOut, sizeof(SparseRank) * dotv->noutRank);
		sp->cache_axis = dotv->axisCache;
		sp->okfit = dotv->fitOutKernel;
		sp->jkfit = dotv->fitJoKernel;
		sp->oksz = blockDim.x;//��� ��Ʈ���� ũ�Ⱑ ��� Ŀ�� ������� ������ ��� ��Ʈ���� ũ��� ������.
		sp->jksz = dotv->szJoKernel;
		sp->nrecycc = dotv->nrecycCache;
		sp->ncycj = dotv->ncycJo;
		sp->njoint = dotv->nJointAxis;
		sp->nout = dotv->noutRank;
		memcpy(sp->out_rank, dotv->outRank, sizeof(intt) * dotv->noutRank);
		if(sp->spr_out[sp->cache_axis].rkPref) {//�����ϴ� �� �������� �ٷ����� (���)������ �� ������ ���� ���� ���ҵ��� 
			memcpy(sp->spr_jcache, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);	//�����޸𸮿� ĳ���Ͽ� �� ����(���)���� �Ѱ� ���Ҹ��� ���� ������ �����Ѵ�.
			memcpy(sp->jrank_cache, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_cache = dotv->njoPre;
			memcpy(sp->spr_jleaf, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);//��¿��� �����ϴ� �� ���� �ܸ� ����(�������� ������ pre�� suf�� �Ѱ� ����)
			memcpy(sp->jrank_leaf, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_leaf = dotv->njoSuf;
		} else {
			memcpy(sp->spr_jcache, dotv->sprSufJo, sizeof(SparseRank) * dotv->njoSuf);
			memcpy(sp->jrank_cache, dotv->joRankSuf, sizeof(intt) * dotv->njoSuf);
			sp->njo_cache = dotv->njoSuf;
			memcpy(sp->spr_jleaf, dotv->sprPreJo, sizeof(SparseRank) * dotv->njoPre);
			memcpy(sp->jrank_leaf, dotv->joRankPre, sizeof(intt) * dotv->njoPre);
			sp->njo_leaf = dotv->njoPre;
		}
	}
	__syncthreads();
	//printf("zz %d %d\n", sp->jksz, dotv->szJoKernel);
	cache_bank = (T *)((bytet *)_cache_bank + sizeof(SharePoint));
	cache_bank = (T *)ALIGN_UNIT((divadx)cache_bank);
	const intt njo_leaf2 = sp->njo_leaf - 1;
	intt ljdim = sp->spr_jleaf[njo_leaf2].rkdim;
	const intt ljsz = sp->spr_jleaf[njo_leaf2].rksz;
	intt leaf_idx[MX_DIM], tmp_idx[MX_DIM], i, i_lj, j;
	register T sum;

	T *m_pdot, *m_sdot, *m_cache, *m_leaf;
	intt inc = blockDim.x, itime_ok = sp->nrecycc, itime_jk = 0;//�ʱ� Ƚ���� ����.
	//intt iloop = 0;
	for(;; roff += inc) {
LP:;	//printf("aa %d %d %d\n", roff, itime_ok, sp->nrecycc);
		cache_load(n, roff_ori, roff, sp->okfit, sp->jkfit, sp->oksz, sp->jksz, sp->nrecycc, sp->ncycj,
			sp->njoint, itime_ok, itime_jk, threadIdx.x, blockDim.x, inc, sp->cache_axis, sp->nout, sp->out_rank, tmp_idx,
			sp->spr_out, sp->spr_jcache, sp->spr_jleaf, sp->njo_cache, sp->jrank_cache, sp->njo_leaf,
			sp->jrank_leaf, _m_pdot, m_pdot, _m_sdot, m_sdot, m_cache, m_leaf, leaf_idx, cache_bank, LP);
		i_lj = leaf_idx[njo_leaf2];//�̹� ���� Ŀ�� �ָ��� ���� �ε���
		//printf("sss %d: %d %d %d\n", threadIdx.x, ljdim, i_lj);
		if(sp->jksz < ljdim - i_lj) i_lj = ljdim - sp->jksz;//���εǴ� ������ ����Ŀ�� ������(jksz)���� ���� ���� ��)���� üũ�ǹǷ� 
		//��������ʰ� ���λ���� Ŀ�� ���ҵǾ� jksz�� ���ϴܸ� �������� ������� ��)���� üũ�ǰ� ����.
		for(sum = 0, j = 0;;) {//���������� ���, ���� �ε����� ���������� �����ϸ� ���� ���� ����, ����Ʈ ���� �ε��� ����
			//printf("%d: %d %d %f %f\n", threadIdx.x, sp->jksz, ljsz, *m_cache, *m_leaf);
			sum += *m_cache * *m_leaf;
			//printf("ss %d: %d %d\n", threadIdx.x, ljdim, i_lj);
			if(ljdim == ++i_lj) {//��.���� �ܸ� ���� �ε��� ����
				if(sp->jksz + leaf_idx[njo_leaf2] < (++j + 1) * ljdim) {//����Ŀ�λ���� �ʱ���� �������� �����Ͽ� ������ ���ϸ���
					i_lj = (j + 1) * ljdim - (sp->jksz + leaf_idx[njo_leaf2]);//üũ������� ������ ���ϸ��� ������� 
					if(i_lj >= ljdim) break;									//������ŭ �����Ͽ� ������ ��)���� üũ�ǰ� �Ѵ�.
				} else i_lj = 0;
				i = njo_leaf2;
J2:;			m_leaf -= sp->spr_jleaf[i].rktsz;
				if(--i < 0) {
					//printf("vv %d: %d %d\n", threadIdx.x, ljdim, i_lj);
					goto LB1;//��.
				} else {
					if(sp->spr_jleaf[i].rkdim == ++leaf_idx[i]) {
						leaf_idx[i] = 0;
						goto J2;
					} else m_leaf += sp->spr_jleaf[i].rksz;//���� ���� ����(����)�� ����
				}
			} else m_leaf += ljsz;
			m_cache++;
		}
LB1:;
		if(itime_jk) *(m_rdot + roff) += sum;
		else {
			if(rplus) *(m_rdot + roff) += sum;
			else *(m_rdot + roff) = sum;
		}
		//iloop++;
	}
	//printf("## %d\n", iloop);
}
template<typename T>
intt gdot_t(void *pcxt_dev, intt oksz, intt share_unit, T *pdot_mdev, T *sdot_mdev, T *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, T rplus)
{
	//dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	//dim3 block(WIDTH_BLOCK);
	dim3 block(WIDTH_BLOCK2(oksz));//������ ��� Ŀ�� ������� �� ����� �� ũ�� �������� �ʴ´�.
	dim3 grid((idx_width + share_unit - 1) / share_unit);//�׸��� ����� ���Ϸ��� �Ʒ����� share_unit ����� 
	//grid((idx_width + share_unit *2 - 1) / share_unit *2);//���(x����� �����ϸ� ���������� share_unit x���� ó���ȴ�.)�ϸ� �ǳ� policyTrack������ ���� ó���Ǿ� �ϹǷ� �� share_unit�� x�谡 �ǰ��Ѵ�.
	intt n = (rdot_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : rdot_size);
	kdot_f<T> << <grid, block >> > (pcxt_dev, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, idx_origin, idx_width, rplus, n);
	hipDeviceSynchronize();
	cuda_error_check(-4);
	return n - idx_origin * idx_width;
}
intt gdot_f(void *pcxt_dev, intt oksz, intt share_unit, floatt *pdot_mdev, floatt *sdot_mdev, floatt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, floatt rplus)
{
	return gdot_t<floatt>(pcxt_dev, oksz, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
intt gdot_f(void *pcxt_dev, intt oksz, intt share_unit, intt *pdot_mdev, intt *sdot_mdev, intt *rdot_mdev, intt rdot_size,
	intt idx_origin, intt idx_width, intt rplus)
{
	return gdot_t<intt>(pcxt_dev, oksz, share_unit, pdot_mdev, sdot_mdev, rdot_mdev, rdot_size,
		idx_origin, idx_width, rplus);
}
template<typename T>
__global__ void karith_f(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt *mrank = arv->arRankMast, *prank = arv->arRankPre, *srank = arv->arRankSuf, *rrank = arv->arRankRet;
	intt npre = arv->narPre, nsuf = arv->narSuf, nmast = arv->narMast;
	intt off, cidx[MX_DIM], tmp_idx[MX_DIM], coff;
	T rval, *ppre, *psuf;

	if(arv->bwGetOri) {
		dlead_offset2idx(arv->narBro, npre, arv->narRet, rrank, roff, cidx);
		for(;;) {//rrank�� �Ѱ� ���ҿ� ���Ͽ� rrank�� ������ 1�� ��ũ���� ���������� ��ȸ�Ͽ� ��ε�ĳ���õ� �͵��� pre�� suf ����
			coff = dbro_offset(arv->narBro, arv->broDimRet, arv->broIdxRet, nmast, mrank, cidx);
			if(coff < 0) break;
			if(m_pari) {//�����Ŀ��� pref�� �����Ŀ��� ���ϵǴ� ��ε�ĳ��Ʈ�� ��Ʈ������ ������ mrank�� �ɼ°��ǹǷ�
				ppre = m_pari + coff;//�ٷ� ���.
				//printf("%d ", coff);
			} else ppre = &sval;
			if(m_sari) {
				off = dmoff2soff(nmast, mrank, nsuf, srank, coff, tmp_idx);
				psuf = m_sari + off;
				//printf("%d\n", off);
			} else psuf = &sval;
			switch(aop) {
			case AOP_MUL:
				rval = *ppre * *psuf;
				break;
			case AOP_PLUS:
				rval = *ppre;
				break;
			case AOP_DIV:
				break;
			case AOP_MINUS:
				break;
			case ABP_MINUS_PREF:
				rval = *ppre;
				break;
			case ABP_MINUS_SUFF:
				rval = *ppre * -1;
				break;
			case ABP_DIV_PREF:
				rval = *ppre * (1 / *psuf);
				break;
			case ABP_DIV_SUFF:
				rval = *ppre * (1 / (*psuf * *psuf) * -1);
				break;
			case ABP_BWTEST:
				rval = *ppre / *psuf;
				break;
			}
			if(aop == ABP_BWTEST) {
				if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
				else if(*(m_rari + roff) != rval) printf("xxx\n");
			} else {
				if(rplus) *(m_rari + roff) += rval;
				else *(m_rari + roff) = rval;
			}
		}
	} else {
		if(m_pari) {
			off = dmoff2soff(nmast, mrank, npre, prank, roff, tmp_idx);
			ppre = m_pari + off;
			//printf("%d ", off);
		} else ppre = &sval;
		if(m_sari) {
			off = dmoff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx);
			psuf = m_sari + off;
			//printf("%d\n", off);
		} else psuf = &sval;
		switch(aop) {
		case AOP_MUL:
			rval = *ppre * *psuf;
			break;
		case AOP_PLUS:
			rval = *ppre + *psuf;
			break;
		case AOP_DIV:
			rval = *ppre / *psuf;
			break;
		case AOP_MINUS:
			rval = *ppre - *psuf;
			break;
		}
		if(rplus) {
			*(m_rari + roff) *= rplus;
			*(m_rari + roff) += rval;
		} else *(m_rari + roff) = rval;
	}
}
template<typename T>
__global__ void karith_f1(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��ε� �ɽ�Ʈ�� ���� ���� ��Ʈ���� �ϴ��� ���� 
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;//pre�� suf�� ret�� ��ġ�� pre�� suf�� ���Ǳ����� �����Ǳ� ���� ���µǹǷ� ���� �Ի��ϰ� �����ϱ�����
	if(arv->bwGetOri) {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * *(m_sari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF:
			rval = *(m_pari + roff) * -1;
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			rval = *(m_pari + roff) * (1 / *(m_sari + roff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			rval = *(m_pari + roff) * (1 / (*(m_sari + roff) * *(m_sari + roff)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = *(m_pari + roff) / *(m_sari + roff);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff) + *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff) / *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff) - *(m_sari + roff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_bwprem(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��� ������ ���� ��ε� �ɽ�Ʈ(������� 1�� ������ ����) �����, m_par����� m_rari������ ���� Ŀ�� m_par�� �׸��������
	intt poff = blockIdx.x*blockDim.x + threadIdx.x;
	ArithVar *arv = (ArithVar *)pcxt;
	intt osz = arv->zarOut, ssz = arv->zarSuf;
	intt roff = poff % osz;

	if(roff < idx_width * idx_origin || roff >= n) return;//���� ��Ʈ������ ���ҵǾ��� ��� pre�κ��� ������ roff�� ���� üũ

	T rval;
	switch(aop) {
	case AOP_MUL:
		rval = *(m_pari + poff) * *(m_sari + poff % ssz);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_PLUS:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_DIV:
		break;
	case AOP_MINUS:
		break;
	case ABP_MINUS_PREF:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_MINUS_SUFF:
		rval = *(m_pari + poff) * -1;
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_PREF:
		rval = *(m_pari + poff) * (1 / *(m_sari + poff % ssz));
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_SUFF:
		rval = *(m_pari + poff) * (1 / (*(m_sari + poff % ssz) * *(m_sari + poff % ssz)) * -1);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_BWTEST:
		rval = *(m_pari + poff) / *(m_sari + poff % ssz);
		if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
		else if(*(m_rari + roff) != rval) printf("xxx\n");
		break;
	}
}
template<typename T>
__global__ void karith_f2(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��� ������ ���� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt ssz = arv->zarSuf, psz = arv->zarPre;
	T rval;
	if(arv->bwGetOri) {//pre�� out�� ����� ���� ���, pre����� out���� ū ���� �� �Լ����� ����, pre�� �����Ŀ���
		switch(aop) {	//���� ��Ʈ�����̹Ƿ� pre�� �� ���� ���� ����.
		case AOP_MUL:
			rval = *(m_pari + roff) * *(m_sari + roff % ssz);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF:
			rval = *(m_pari + roff) * -1;
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			rval = *(m_pari + roff) * (1 / *(m_sari + roff % ssz));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			rval = *(m_pari + roff) * (1 / (*(m_sari + roff % ssz) * *(m_sari + roff % ssz)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = *(m_pari + roff) / *(m_sari + roff % ssz);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval) printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff % psz) * *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff % psz) + *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff % psz) / *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff % psz) - *(m_sari + roff % ssz);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_lc(void *pcxt, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{//������ ����� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;
	if(arv->bwGetOri) {//m_sari�� �����Ķ��� ���
		switch(aop) {
		case AOP_MUL:
			rval = (sval * *(m_sari + roff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_sari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF://������ ����̹Ƿ� pref�� ����.
			break;
		case ABP_MINUS_SUFF:
			if(rplus) *(m_rari + roff) += *(m_sari + roff) * -1;
			else *(m_rari + roff) = *(m_sari + roff) * -1;
			break;
		case ABP_DIV_PREF://������ ����̹Ƿ� pref�� ����.
			break;
		case ABP_DIV_SUFF:
			rval = *(m_sari + roff) * (1 / (*(m_rari + roff) * *(m_rari + roff)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = sval / *(m_sari + roff);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = (sval * *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = (sval + *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = (sval / *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = (sval - *(m_sari + roff));
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f2_rc(void *pcxt, T *m_pari, T *m_rari, intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, intt n)
{//������ ����� ��ε� �ɽ�Ʈ(������� 1�� ������ ����)
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	T rval;
	if(arv->bwGetOri) {
		switch(aop) {
		case AOP_MUL:
			rval = (*(m_pari + roff) * sval);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF://������ ����̹Ƿ� suff�� ����.
			//rval = *(m_pari + roff) * -1;
			//if(rplus) *(m_rari + roff) += rval;
			//else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			rval = *(m_pari + roff) * (1 / sval);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF://������ ����̹Ƿ� suff�� ����.
			//rval = *(m_pari + roff) * (1 / (sval * sval) * -1);
			//if(rplus) *(m_rari + roff) += rval;
			//else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			rval = *(m_pari + roff) / sval;
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + roff) * sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff) + sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + roff) / sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + roff) - sval;
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
__global__ void karith_f3_bwprem(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//��� ������ ���� ��ε� �ɽ�Ʈ(������� 1�� ������ ����) �����, m_par����� m_rari������ ���� Ŀ�� m_par�� �׸��������
	intt poff = blockIdx.x*blockDim.x + threadIdx.x;
	ArithVar *arv = (ArithVar *)pcxt;
	intt *mrank = arv->arRankMast, *srank = arv->arRankSuf, *rrank = arv->arRankRet;
	intt nsuf = arv->narSuf, nmast = arv->narMast, nret = arv->narRet;
	intt tmp_idx[MX_DIM], soff, roff;
	T rval;

	_moff2soff(nmast, mrank, nret, rrank, poff, tmp_idx, roff);

	if(roff < idx_width * idx_origin || roff >= n) return;//���� ��Ʈ������ ���ҵǾ��� ��� pre�κ��� ������ roff�� ���� üũ

	switch(aop) {
	case AOP_MUL:
		_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
		rval = *(m_pari + poff) * *(m_sari + soff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_PLUS:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case AOP_DIV:
		break;
	case AOP_MINUS:
		break;
	case ABP_MINUS_PREF:
		rval = *(m_pari + poff);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_MINUS_SUFF:
		rval = *(m_pari + poff) * -1;
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_PREF:
		_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
		rval = *(m_pari + poff) * (1 / *(m_sari + soff));
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_DIV_SUFF:
		_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
		rval = *(m_pari + poff) * (1 / (*(m_sari + soff) * *(m_sari + soff)) * -1);
		if(rplus) atomicAdd(m_rari + roff, rval);
		else *(m_rari + roff) = rval;
		break;
	case ABP_BWTEST:
		_moff2soff(nmast, mrank, nsuf, srank, poff, tmp_idx, soff);
		rval = *(m_pari + poff) / *(m_sari + soff);
		if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
		else if(*(m_rari + roff) != rval)  printf("xxx\n");
		break;
	}
}
template<typename T>
__global__ void karith_f3(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt idx_origin, intt idx_width, sytet aop, T rplus, intt n)
{//������� 1�� ������ �����ϴ� ��ε� �ɽ�Ʈ, ��� ������ ����̸� ����� ������� 1�� ������ �ǹ̰� �������Ƿ� Ÿ�� 2�� ���̽��� �����.
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	ArithVar *arv = (ArithVar *)pcxt;
	intt *mrank = arv->arRankMast, *prank = arv->arRankPre, *srank = arv->arRankSuf;
	intt npre = arv->narPre, nsuf = arv->narSuf, nmast = arv->narMast;
	intt tmp_idx[MX_DIM], poff, soff;
	T rval;

	if(arv->bwGetOri) {//pre�� out�� ����� ���� ���, pre����� out���� ū ���� �� �Լ����� ����, pre�� �����Ŀ���
		switch(aop) {	//���� ��Ʈ�����̹Ƿ� pre�� �� ���� ���� ����.
		case AOP_MUL:
			_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
			rval = *(m_pari + roff) * *(m_sari + soff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			break;
		case AOP_MINUS:
			break;
		case ABP_MINUS_PREF:
			rval = *(m_pari + roff);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_MINUS_SUFF:
			rval = *(m_pari + roff) * -1;
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_PREF:
			_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
			rval = *(m_pari + roff) * (1 / *(m_sari + soff));
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_DIV_SUFF:
			_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
			rval = *(m_pari + roff) * (1 / (*(m_sari + soff) * *(m_sari + soff)) * -1);
			if(rplus) *(m_rari + roff) += rval;
			else *(m_rari + roff) = rval;
			break;
		case ABP_BWTEST:
			_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
			rval = *(m_pari + roff) / *(m_sari + soff);
			if(*(m_rari + roff) == 0) *(m_rari + roff) = rval;
			else if(*(m_rari + roff) != rval)  printf("xxx\n");
			break;
		}
	} else {
		_moff2soff(nmast, mrank, npre, prank, roff, tmp_idx, poff);
		_moff2soff(nmast, mrank, nsuf, srank, roff, tmp_idx, soff);
		//printf("11111111 %d %d %f\n", poff, soff, rval);
		switch(aop) {
		case AOP_MUL:
			rval = *(m_pari + poff) * *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_PLUS:
			rval = *(m_pari + poff) + *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_DIV:
			rval = *(m_pari + poff) / *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		case AOP_MINUS:
			rval = *(m_pari + poff) - *(m_sari + soff);
			if(rplus) {
				*(m_rari + roff) *= rplus;
				*(m_rari + roff) += rval;
			} else *(m_rari + roff) = rval;
			break;
		}
	}
}
template<typename T>
intt garith_t(void *pcxt, T *m_pari, T *m_sari, T *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, T sval, sytet aop, T rplus, sytet tp_arith, sytet bw)
{
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);
	if(tp_arith >= AR_T_BRO && p_size > r_size) {//pre����� ret������� ū���� ��ε��ɽ�Ʈ ������ ���ۿ� ����.
		dim3 block(WIDTH_BLOCK3(p_size));			//�̰�� pre�� �������� ������ �����Ѵ�.
		dim3 grid((p_size + block.x - 1) / block.x);
		if(tp_arith == AR_T_BRO) karith_f2_bwprem<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
		else karith_f3_bwprem<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
	} else {
		intt bsz = (tp_arith == AR_T_ONEBRO ? SMALL_BLOCK : BLOCK_SIZE);
		dim3 block(WIDTH_BLOCK2(bsz));
		dim3 grid((idx_width + block.x - 1) / block.x);
		//karith_f<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
		switch(tp_arith) {
		case AR_T_O2O:
			karith_f1<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		case AR_T_BROLC:
			karith_f2_lc<T> << <grid, block >> > (pcxt, bw ? m_pari : m_sari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
			break;
		case AR_T_BRORC:
			karith_f2_rc<T> << <grid, block >> > (pcxt, m_pari, m_rari, idx_origin, idx_width, sval, aop, rplus, n);
			break;
		case AR_T_BRO:
			karith_f2<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		case AR_T_ONEBRO:
			karith_f3<T> << <grid, block >> > (pcxt, m_pari, m_sari, m_rari, idx_origin, idx_width, aop, rplus, n);
			break;
		}
	}
	hipDeviceSynchronize();
	cuda_error_check(-5);
	return n - idx_origin * idx_width;
}
intt garith_f(void *pcxt, floatt *m_pari, floatt *m_sari, floatt *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, floatt sval, sytet aop, floatt rplus, sytet tp_arith, sytet bw)
{
	return garith_t<floatt>(pcxt, m_pari, m_sari, m_rari, p_size, r_size,
		idx_origin, idx_width, sval, aop, rplus, tp_arith, bw);
}
intt garith_f(void *pcxt, intt *m_pari, intt *m_sari, intt *m_rari, intt p_size, intt r_size,
	intt idx_origin, intt idx_width, intt sval, sytet aop, intt rplus, sytet tp_arith, sytet bw)
{
	return garith_t<intt>(pcxt, m_pari, m_sari, m_rari, p_size, r_size,
		idx_origin, idx_width, sval, aop, rplus, tp_arith, bw);
}

template<typename T>
__global__ void ktrans_f(void *pcxt, T *m_strs, T *m_rtrs, intt idx_origin, intt idx_width, bool bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	TransVar *tsvar = (TransVar *)pcxt;
	intt ndims = tsvar->ntrDims, *rrank = tsvar->trRankRet, i, ridx[MX_DIM];
	TransRank *tmap = tsvar->tspmap;

	_offset2idx(ndims, rrank, roff, ridx);
	for(i = 0;i < ndims; i++) {
		m_strs += (ridx[i] * tmap[i].trssz);
	}
	if(bw) *(m_rtrs + roff) += *m_strs;
	else *(m_rtrs + roff) = *m_strs;
}
template<typename T>
intt gtrans_t(void *pcxt, T *m_strs, T *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ktrans_f<T> << <grid, block >> > (pcxt, m_strs, m_rtrs, idx_origin, idx_width, bw, n);
	hipDeviceSynchronize();
	cuda_error_check(-6);
	return n - idx_origin * idx_width;
}
intt gtrans_f(void *pcxt, floatt *m_strs, floatt *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	return gtrans_t<floatt>(pcxt, m_strs, m_rtrs, r_size, idx_origin, idx_width, bw);
}
intt gtrans_f(void *pcxt, intt *m_strs, intt *m_rtrs, intt r_size, intt idx_origin, intt idx_width, bool bw)
{
	return gtrans_t<intt>(pcxt, m_strs, m_rtrs, r_size, idx_origin, idx_width, bw);
}
/*template<typename T>
__device__ void datomic_max_f(T *pmax, const T value)
{
	if(*pmax >= value) return;

	intt * const imax = (int *)pmax;
	intt old = *imax, vcmp;

	do
	{
		vcmp = old;
		if(__int_as_float(vcmp) >= value) break;

		old = atomicCAS(imax, vcmp, __float_as_int(value));
	} while(vcmp != old);
}*/
template<typename T>
__device__ void datomic_max_f(T *pmax, const T value)
{
	intt * const imax = (int *)pmax;
	intt old = *imax, vcmp;

	while(value > __int_as_float(old)) {
		//printf("%f %f %d\n", value, __int_as_float(old), old);
		vcmp = old;
		old = atomicCAS(imax, vcmp, __float_as_int(value));
	}
}
template<typename T>
__global__ void ksoftx_prob_f(void *pcxt, T *m_rsfx, T *m_sum, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	*(m_rsfx + roff) = *(m_rsfx + roff) / (*(m_sum + roff / f_size) + 1e-8);
}
template<typename T>
__global__ void ksoftx_sum_f(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_sum, T *m_max, intt f_size, sytet db, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	T v;
	if(roff >= n) return;

	if(db) v = std::exp((doublet)(*(m_ssfx + roff) - *(m_max + roff / f_size)));
	else v = std::exp((floatt)(*(m_ssfx + roff) - *(m_max + roff / f_size)));
	atomicAdd(m_sum + roff / f_size, v);
	*(m_rsfx + roff) = v;
	
}
template<typename T>
void ksoftx_max_f(intt tp, T *m_ssfx, T *m_max, T *m_buf, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin;
	intt max_off = roff / f_size;

	switch(tp) {
	case tfloat:
		for(;roff < n; roff += f_size, max_off++) {
			nppsMax_32f((floatt *)m_ssfx + roff, (const intt)f_size, (floatt *)m_max + max_off, (ubytet *)m_buf);
		}
		break;
	}
}
template<typename T>
intt gsoftx_t(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_sum, T *m_max, T *m_buf, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	//dim3 grid2((idx_width / f_size + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksoftx_max_f<T>(tfloat, m_ssfx, m_max, m_buf, f_size, idx_origin, idx_width, n);
	ksoftx_sum_f<T> << <grid, block >> > (pcxt, m_ssfx, m_rsfx, m_sum, m_max, f_size, db, idx_origin, idx_width, n);
	ksoftx_prob_f<T> << <grid, block >> > (pcxt, m_rsfx, m_sum, f_size, idx_origin, idx_width, n);
	hipDeviceSynchronize();
	cuda_error_check(-24);
	return n - idx_origin * idx_width;
}
intt gsoftx_f(void *pcxt, floatt *m_ssfx, floatt *m_rsfx, floatt *m_sum, floatt *m_max, floatt *m_buf, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	return gsoftx_t<floatt>(pcxt, m_ssfx, m_rsfx, m_sum, m_max, m_buf, r_size, f_size, db, idx_origin, idx_width);
}
intt gsoftx_f(void *pcxt, intt *m_ssfx, intt *m_rsfx, intt *m_sum, intt *m_max, intt *m_buf, intt r_size, intt f_size, sytet db, intt idx_origin, intt idx_width)
{
	return gsoftx_t<intt>(pcxt, m_ssfx, m_rsfx, m_sum, m_max, m_buf, r_size, f_size, db, idx_origin, idx_width);
}

template<typename T>
__global__ void ksoftx_cross_e_f(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_tsfx, intt f_size, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(m_rsfx + (roff / f_size), -1.0f * std::log(*(m_ssfx + roff) + 1e-8) * *(m_tsfx + roff));
	//printf("%f %f %f\n", *(m_rsfx + (roff / f_size)), *(m_ssfx + roff), *(m_tsfx + roff));
}
template<typename T>
intt gsoftx_cross_e_t(void *pcxt, T *m_ssfx, T *m_rsfx, T *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksoftx_cross_e_f<T> << <grid, block >> > (pcxt, m_ssfx, m_rsfx, m_tsfx, f_size, idx_origin, idx_width, n);
	hipDeviceSynchronize();
	cuda_error_check(-7);
	return n - idx_origin * idx_width;
}
intt gsoftx_cross_e_f(void *pcxt, floatt *m_ssfx, floatt *m_rsfx, floatt *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	return gsoftx_cross_e_t<floatt>(pcxt, m_ssfx, m_rsfx, m_tsfx, r_size, f_size, idx_origin, idx_width);
}
intt gsoftx_cross_e_f(void *pcxt, intt *m_ssfx, intt *m_rsfx, intt *m_tsfx, intt r_size, intt f_size, intt idx_origin, intt idx_width)
{
	return gsoftx_cross_e_t<intt>(pcxt, m_ssfx, m_rsfx, m_tsfx, r_size, f_size, idx_origin, idx_width);
}
template<typename T>
__global__ void kmean_f(T *m_rmet, T *cmul, bool mean, intt r_size)
{
	if(cmul) *m_rmet *= *(T *)cmul;
	if(mean) *m_rmet /= r_size;
	//printf("%p %d %f\n", cmul, mean, *m_rmet);
}
template<typename T>
__global__ void kmse_f(void *pcxt, T *m_smet, T *m_tmet, T *m_rmet, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(m_rmet, (m_smet[roff] - m_tmet[roff]) * (m_smet[roff] - m_tmet[roff]));
}
template<typename T>
__global__ void kmse_f2(void *pcxt, T *m_smet, T *m_tmet, T *m_rmet, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	m_rmet[roff] = (m_smet[roff] - m_tmet[roff]) * (m_smet[roff] - m_tmet[roff]);
}
template<typename T>
intt gmse_t(void *pcxt, T *m_smet, T *m_tmet, T *m_rmet, intt r_size, intt idx_origin, intt idx_width, bool mean)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	if(mean) {//��Ʈ���� ��ü ���� ���� ���
		kmse_f<T> << <grid, block >> > (pcxt, m_smet, m_tmet, m_rmet, idx_origin, idx_width, n);
		hipDeviceSynchronize();
		if(n == r_size) kmean_f<T> << <1, 1 >> > (m_rmet, nullx, 1, r_size);//������ ���ҿ��� ��ձ��ϱ� ����.
		n = 1;
	} else {//��ġ���� ���� ���������� ���
		kmse_f2<T> << <grid, block >> > (pcxt, m_smet, m_tmet, m_rmet, idx_origin, idx_width, n);
	}
	hipDeviceSynchronize();
	cuda_error_check(-8);
	return n;
}
intt gmse_f(void *pcxt, floatt *m_smet, floatt *m_tmet, floatt *m_rmet, intt r_size, intt idx_origin, intt idx_width, bool mean)
{
	return gmse_t<floatt>(pcxt, m_smet, m_tmet, m_rmet, r_size, idx_origin, idx_width, mean);
}
intt gmse_f(void *pcxt, intt *m_smet, intt *m_tmet, intt *m_rmet, intt r_size, intt idx_origin, intt idx_width, bool mean)
{
	return gmse_t<intt>(pcxt, m_smet, m_tmet, m_rmet, r_size, idx_origin, idx_width, mean);
}
template<typename T>
__global__ void ksum_f(void *pcxt, T *m_smet, T *m_rmet, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(m_rmet, *(m_smet + roff));
}
template<typename T>
intt gsum_t(void *pcxt, T *m_smet, T *m_rmet, intt r_size, intt idx_origin, intt idx_width, T *cmul, bool mean)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ksum_f<T> << <grid, block >> > (pcxt, m_smet, m_rmet, idx_origin, idx_width, n);
	if(n == r_size && (cmul || mean)) kmean_f<T> << <1, 1 >> > (m_rmet, cmul, mean, r_size);//������ ���ҿ��� ��ձ��ϱ� ����.
	hipDeviceSynchronize();
	cuda_error_check(-9);
	return n - idx_origin * idx_width;
}
intt gsum_f(void *pcxt, floatt *m_smet, floatt *m_rmet, intt r_size, intt idx_origin, intt idx_width, floatt *cmul, bool mean)
{
	return gsum_t<floatt>(pcxt, m_smet, m_rmet, r_size, idx_origin, idx_width, cmul, mean);
}
intt gsum_f(void *pcxt, intt *m_smet, intt *m_rmet, intt r_size, intt idx_origin, intt idx_width, intt *cmul, bool mean)
{
	return gsum_t<intt>(pcxt, m_smet, m_rmet, r_size, idx_origin, idx_width, cmul, mean);
}

template<typename T>
__global__ void kbmean_f(T *mret, intt beg, intt n, intt sum_sz)
{
	intt roff = beg + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mret[roff] /= sum_sz;
}
template<typename T>
__global__ void kbsum_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, intt n, bool bw, sytet rplus, intt sum_sz)
{
	OneVar *ovar = (OneVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	
	if(roff >= n) return;

	if(bw) {//roff�� mret�ɼ�
		floatt div = *(floatt *)&ovar->idxOne[2];
		if(ovar->idxOne[0]) {//�����Ķ� ��� �������� ���� ���⿡ ���Ͽ� �����Ķ� �Կ� �� ����� �����Ѵ�.
			if(rplus) mret[roff] += mpre[roff / sum_sz] * div;
			else mret[roff] = mpre[roff / sum_sz] * div;
		} else {
			if(rplus) mret[roff] += mpre[roff / sum_sz];
			else mret[roff] = mpre[roff / sum_sz];
		}
	} else {
		atomicAdd(&mret[roff / sum_sz], mpre[roff]);//roff�� mpre�ɼ�
	}
}
template<typename T>
intt gbsum_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width, intt sum_sz, bool mean, bool bw, sytet rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kbsum_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, n, bw, rplus, sum_sz);
	if(bw == 0 && mean) {//������ ���ҿ��� ��ձ��ϱ� ����.
		intt beg = (idx_width * idx_origin) / sum_sz;//�̹� ������ ��ġ���� ��� ���� ��ġ �ɼ� ����
		intt end = (idx_width * idx_origin + n) / sum_sz;//�̹� ������ ��ġ���� ��� ���� ��ġ �ɼ� ����
		intt n2 = end - beg;
		dim3 grid2((n2 + block.x - 1) / block.x);
		kbmean_f<T> << <grid2, block >> > (mret, beg, n2, sum_sz);
	}
	hipDeviceSynchronize();
	cuda_error_check(-26);
	return n - idx_origin * idx_width;
}
intt gbsum_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt sum_sz, bool mean, bool bw, sytet rplus)
{
	return gbsum_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, sum_sz, mean, bw, rplus);
}
intt gbsum_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt sum_sz, bool mean, bool bw, sytet rplus)
{
	return gbsum_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, sum_sz, mean, bw, rplus);
}

template<typename T>
__global__ void koptadm_f(T *mm, T *mv, T *mg, T *mr, intt idx_origin,
	intt idx_width, T beta1, T beta2, T lr, T ep, intt dec, sytet db, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mm[roff] += (1.0f - beta1) * (mg[roff] - mm[roff]);//����: mm[roff] + (l - beta1)*mg[roff] - mm[roff] + mm[roff]*beta1
													//	  = mm[roff]*beta1 + (l - beta1)*mg[roff]
	mv[roff] += (1.0f - beta2) * (mg[roff] * mg[roff] - mv[roff]);
	mr[roff] += dec * lr * mm[roff] / (std::sqrt(db ? (doublet)mv[roff] : (floatt)mv[roff]) + ep);
}
template<typename T>
intt goptadm_t(void *pcxt, T *mm, T *mv, T *mg, T *mr, intt r_size, intt idx_origin,
	intt idx_width, T beta1, T beta2, T lr, T ep, intt dec, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	koptadm_f<T> << <grid, block >> > (mm, mv, mg, mr, idx_origin, idx_width, beta1, beta2, lr, ep, dec, db, n);
	hipDeviceSynchronize();
	cuda_error_check(-10);
	return n - idx_origin * idx_width;
}
intt goptadm_f(void *pcxt, floatt *mm, floatt *mv, floatt *mg, floatt *mr, intt r_size, intt idx_origin,
	intt idx_width, floatt beta1, floatt beta2, floatt lr, floatt ep, intt dec, sytet db)
{
	return goptadm_t<floatt>(pcxt, mm, mv, mg, mr, r_size, idx_origin,
		idx_width, beta1, beta2, lr, ep, dec, db);
}
intt goptadm_f(void *pcxt, intt *mm, intt *mv, intt *mg, intt *mr, intt r_size, intt idx_origin,
	intt idx_width, intt beta1, intt beta2, intt lr, intt ep, intt dec, sytet db)
{
	return goptadm_t<intt>(pcxt, mm, mv, mg, mr, r_size, idx_origin,
		idx_width, beta1, beta2, lr, ep, dec, db);
}

template<typename T>
__global__ void koptsgd_f(T *mg, T *mr, intt idx_origin, intt idx_width, T lr, intt dec, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mr[roff] += dec * lr * mg[roff];
}
template<typename T>
intt goptsgd_t(void *pcxt, T *mg, T *mr, intt r_size, intt idx_origin, intt idx_width, T lr, intt dec)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	koptsgd_f<T> << <grid, block >> > (mg, mr, idx_origin, idx_width, lr, dec, n);
	hipDeviceSynchronize();
	cuda_error_check(-11);
	return n - idx_origin * idx_width;
}
intt goptsgd_f(void *pcxt, floatt *mg, floatt *mr, intt r_size, intt idx_origin, intt idx_width, floatt lr, intt dec)
{
	return goptsgd_t<floatt>(pcxt, mg, mr, r_size, idx_origin, idx_width, lr, dec);
}
intt goptsgd_f(void *pcxt, intt *mg, intt *mr, intt r_size, intt idx_origin, intt idx_width, intt lr, intt dec)
{
	return goptsgd_t<intt>(pcxt, mg, mr, r_size, idx_origin, idx_width, lr, dec);
}
template<typename T>
__device__ __forceinline__ T mat_sqrt(T a, sytet db)
{
	return std::sqrt(db ? (doublet)a : (floatt)a);
}
template<typename T>
__device__ __forceinline__ T mat_exp(T a, sytet db)
{
	return std::exp(db ? (doublet)a : (floatt)a);
}
template<typename T>
__device__ __forceinline__ float mat_log(T a, sytet db) {
	return std::log(db ? (doublet)a : (floatt)a);
}
template<typename T>
__global__ void kactf_f(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db, intt n)
{//msuf�� ���� ����
	OneVar *ovar = (OneVar *)pcxt;
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	T v;

	if(roff >= n) return;

	switch(aop2) {
	case ACTF_TANH:
		mret[roff] = std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]);
		//printf("%f %f\n", mret[roff], mpre[roff]);
		break;
	case DACTF_TANH:
		if(rplus) mret[roff] += ((1.0f - std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]) *
			std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff])) * msuf[roff]);
		else mret[roff] = (1.0f - std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff]) *
			std::tanh(db ? (doublet)mpre[roff] : (floatt)mpre[roff])) * msuf[roff];
		break;
	case ACTF_RELU:
		mret[roff] = mpre[roff] > 0.0f ? mpre[roff] : 0.0f;
		//if(mpre[roff] > 0.0f) {
		//	mret[roff] = mpre[roff];
		//	mpre[roff] = 1;
		//} else mpre[roff] = 0;
		break;
	case DACTF_RELU:
		if(rplus) mret[roff] += (mpre[roff] > 0.0 ? msuf[roff] : 0.0);
		else mret[roff] = (mpre[roff] > 0.0 ? msuf[roff] : 0.0);
		break;
	case ACTF_SIGM:
		mret[roff] = 1.0 / (1.0 + mat_exp(-mpre[roff], db));//1.0f/(1.0f + std::exp(-a));
		break;
	case DACTF_SIGM:
		v = 1.0 / (1.0 + mat_exp(-mpre[roff], db));
		if(rplus) mret[roff] += (1.0 - v) * v * msuf[roff];
		else mret[roff] = (1.0 - v) * v * msuf[roff];
		break;
	case ACTF_LRELU:
		mret[roff] = mpre[roff] > 0.0f ? mpre[roff] : *(T *)&ovar->idxOne[0] * mpre[roff];
		break;
	case DACTF_LRELU:
		if(rplus) mret[roff] += ((mpre[roff] > 0.0 ? 1.0 : *(T *)&ovar->idxOne[0]) * msuf[roff]);
		else mret[roff] = (mpre[roff] > 0.0 ? 1.0 : *(T *)&ovar->idxOne[0]) * msuf[roff];
		break;
	case MATH_SQRT:
		mret[roff] = mat_sqrt(mpre[roff], db);
		break;
	case DMATH_SQRT:
		if(rplus) mret[roff] += ((0.5 * 1.0 / (mat_sqrt(mpre[roff], db) + 1e-9)) * msuf[roff]);//0.5 * pow(mpre[roff], -0.5f)
		else mret[roff] = (0.5 * 1.0 / (mat_sqrt(mpre[roff], db) + 1e-9)) * msuf[roff];//0.5 * pow(mpre[roff], -0.5f)
		break;
	case DJUST_COPY://�ܼ��� �����Ķ� ���⸦ ���ϱ����� ���.
		if(msuf) {
			if(rplus) mret[roff] += mpre[roff] * msuf[roff];
			else mret[roff] = mpre[roff] * msuf[roff];
		} else {
			if(rplus) mret[roff] += mpre[roff];
			else mret[roff] = mpre[roff];
		}
		break;
	case DJUST_COPY2://mpre, msuf�� ���� 1��
		if(msuf) {
			if(rplus) mret[roff] += mpre[0] * msuf[0];
			else mret[roff] = mpre[0] * msuf[0];
		} else {
			if(rplus) mret[roff] += mpre[0];
			else mret[roff] = mpre[0];
		}
		break;
	case MATH_LOG:
		mret[roff] = mat_log(mpre[roff], db);
		break;
	case DMATH_LOG:
		mret[roff] = (1.0 / (mpre[roff] + 1e-9)) * msuf[roff];
		break;
	case DLOSS_FUNC:
		mret[roff] += ((mpre[roff] - msuf[roff]) / *(T *)ovar->idxOne);//��ġ������� ����
		break;
	case SCOOP_UP:
	{
		intt slidex, slidey, stridex, stridey, outx, outy, ibatch, rest;
		intt prey = ovar->idxOne[6], prex = ovar->idxOne[7], sz_feat = ovar->idxOne[8], d2;
		slidey = ovar->idxOne[0]; slidex = ovar->idxOne[1] * sz_feat; stridey = ovar->idxOne[2];
		stridex = ovar->idxOne[3]; outy = ovar->idxOne[4]; outx = ovar->idxOne[5];
		intt n_derive = outx * outy, sz_slide = slidex * slidey, irow, icol, i, j;
		intt sz_derive = n_derive * sz_slide, sz_derive_row = outx * sz_slide;
		T *px, *py, *pbatch, *pslide;
		//roff�� ��Ʈ���̵忡 ���� Ȯ�� �Ļ��� ��Ʈ������ �ɼ��̹Ƿ� �̷μ� �ε����� ȹ���ϰ� �̷κ��� �ҽ��� �ɼ��� ����Ѵ�.
		ibatch = roff / sz_derive;//��Ʈ���̵忡 ���� �Ļ��� ��Ʈ���������� �̹� �ɼ��� ���� ��ġ�ο� �ε���
		rest = roff - (ibatch * sz_derive);//�Ļ��� ��Ʈ������ �̹� ���� ��ġ�ο��� ���� ������
		//��Ʈ���̵忡 ���� Ȯ��� �Ļ� ��Ʈ�������� �� �ɼ��� �Ļ� ��Ʈ�����󿡼��� ��ġ�ε����� �����̵��� ���� �𼭸���
		//���������� �Ͽ� ��ġ�������� �ο�� �÷� �ε����� ��ȯ�Ѵ�.
		if(outy != 1) {//2d
			irow = rest / sz_derive_row;//�Ļ���Ʈ������ ���� �ϳ��� ��ġ���� �ɼ��� �Ļ���Ʈ���� �� ���� ������� ����
			rest -= irow * sz_derive_row;//�Ļ���Ʈ������ �� ��ġ�� �ο� �ε����� ���ϰ� �� �ο��� ���� �ɼ����κ��� ����
			d2 = 1;						//����� ���Ѵ�.
		} else {
			irow = 0;//1d	
			d2 = 0;
		}
		icol = rest / sz_slide;//�Ļ���Ʈ������ ���� ������κ��� �����̵� ��������� �÷� �ε����� ���
		rest -= icol * sz_slide;

		prex *= sz_feat;
		stridey *= prex;//�ο� ��Ʈ���̵带 mpre��Ʈ�������� ������ ��ȯ
		stridex *= sz_feat;//�÷� ��Ʈ���̵带 mpre��Ʈ�������� ������ ��ȯ
		outy *= stridey;//�Ѱ谪�� mpre��Ʈ�������� ������ ��ȯ, �����е� �ȴٸ� prey�� ��Ʈ���̵尡 �ѹ� �� ������ ���� �ϼ��ִ�. 
		outx *= stridex;//�Ѱ谪�� mpre��Ʈ�������� ������ ��ȯ, �����е� �ȴٸ� prex�� ��Ʈ���̵尡 �ѹ� �� ������ ���� �ϼ��ִ�.
		irow *= stridey;//�Ѱ谪�� mpre��Ʈ�������� �������� ��ȯ�����Ƿ� �ʱⰪ�� ��ȯ
		icol *= stridex;//�Ѱ谪�� mpre��Ʈ�������� �������� ��ȯ�����Ƿ� �ʱⰪ�� ��ȯ

		if(slidey != 1) {//2d
			i = rest / slidex;//�Ļ���Ʈ������ �����̵峻�� �ο� �ε��� ��� 
			//j = rest % slidex;//�Ļ���Ʈ������ �����̵峻�� �÷� �ε��� ���
			j = rest - i * slidex;//��.gpu ���� �����忡���� roff�� ���Ĵ����� �ƴ� ���İ��� ������ �ǹǷ�
									//���� ������ ���ϰ� ���� j�� ���� �������� �ɼ����� �Ѵ�.
		} else {//1d
			i = 0;
			j = rest;
		}
		slidey *= prex;//�Ѱ谪�� mpre��Ʈ�������� ������ ��ȯ
		i *= prex;//�Ѱ谪�� mpre��Ʈ�������� �������� ��ȯ�����Ƿ� �ʱⰪ�� ��ȯ
		//j *= sz_feat;//��)���� ���� ���� �����̹Ƿ� ���� ������ ���� �;���.
		prey *= prex;//1�� ��ġ������

		pbatch = mpre + ibatch * prey;//mpre��Ʈ�������� �̹� ��ġ ���� ������ ���.
		py = pbatch + d2 * irow;//d2�� 1d�̸� 0, mpre��Ʈ�������� �ο������� ����.
		pslide = py + icol;//�����̵� ������(���� ����) ������ ����
		px = pslide + i;//mpre������ �� ������ ������ �����ؾ��Ѵ�.
		mret[roff++] = (irow + i >= prey || icol + j >= prex ? 0 : *(px + j));//�����е�
	}
		break;
	case DSCOOP_UP:
		break;
	case MINMAX_NORMAL:
		mret[roff] = (mpre[roff] - (T)*(doublet *)ovar->idxOne[0]) / (T)*(doublet *)ovar->idxOne[2];
		break;
	case DMINMAX_NORMAL:
		break;
	}
}
template<typename T>
intt gactf_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kactf_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, aop2, rplus, db, n);
	hipDeviceSynchronize();
	cuda_error_check(-12);
	return n - idx_origin * idx_width;
}
intt gactf_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2, rplus, db);
}
intt gactf_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2, rplus, db);
}
template<typename T>
__global__ void kactf2_f(void *pcxt, T *mpre, T *msuf, T *mret, T *m1, T *m2, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	switch(aop2) {
	case ACTF_PRELU://msuf�� prelu ����ġ ������
		mret[roff] = mpre[roff] > 0.0 ? mpre[roff] : msuf[roff] * mpre[roff];// x > 0.0f ? x : a * x
		break;
	case DACTF_PRELU://mpre�� ���� ����, msuf�� �����Ķ� ���, mret�� �����Ķ� �Է� ����
	{				//m1 - prelu ����ġ ������, m2 - prelu ����ġ ����
		OneVar *ovar = (OneVar *)pcxt;
		if(rplus) {
			mret[roff] += ((msuf[roff] > 0.0 ? 1.0 : m1[roff]) * mpre[roff]);//dst[idx] = src[idx] > 0.0f ? 1.0f : a[idx];
			m2[roff] += ((msuf[roff] > 0.0 ? 0.0 : msuf[roff]) * mpre[roff]);//da[idx] = src[idx] > 0.0f ? 0.0f : src[idx];
		} else {
			mret[roff] = (msuf[roff] > 0.0 ? 1.0 : m1[roff]) * mpre[roff];//dst[idx] = src[idx] > 0.0f ? 1.0f : a[idx];
			m2[roff] = (msuf[roff] > 0.0 ? 0.0 : msuf[roff]) * mpre[roff];//da[idx] = src[idx] > 0.0f ? 0.0f : src[idx];
		}
	}
		break;
	}
}
template<typename T>
intt gactf2_t(void *pcxt, T *mpre, T *msuf, T *mret, T *m1, T *m2, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kactf2_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, m1, m2, idx_origin, idx_width, aop2, rplus, db, n);
	hipDeviceSynchronize();
	cuda_error_check(-13);
	return n - idx_origin * idx_width;
}
intt gactf2_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, floatt *m1, floatt *m2, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf2_t<floatt>(pcxt, mpre, msuf, mret, m1, m2, r_size, idx_origin, idx_width, aop2, rplus, db);
}
intt gactf2_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt *m1, intt *m2, intt r_size, intt idx_origin, intt idx_width, intt aop2, sytet rplus, sytet db)
{
	return gactf2_t<intt>(pcxt, mpre, msuf, mret, m1, m2, r_size, idx_origin, idx_width, aop2, rplus, db);
}
template<typename T>
__global__ void ktwo_f(T *mpre, T *msuf, T *mret, T *bpre, T *bsuf, intt idx_origin, intt idx_width, intt aop2, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	switch(aop2) {
	case TWOF_SQDIFF:
		mret[roff] = (mpre[roff] - msuf[roff]) * (mpre[roff] - msuf[roff]);
		break;
	case DTWOF_SQDIFF:
		mpre[roff] += 2 * mret[roff];
		msuf[roff] += -2 * mret[roff];
		break;
	}
}
template<typename T>
intt gtwo_t(void *pcxt, T *mpre, T *msuf, T *mret, T *bpre, T *bsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ktwo_f<T> << <grid, block >> > (mpre, msuf, mret, bpre, bsuf, idx_origin, idx_width, aop2, n);
	hipDeviceSynchronize();
	cuda_error_check(-14);
	return n - idx_origin * idx_width;
}
intt gtwo_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, floatt *bpre, floatt *bsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtwo_t<floatt>(pcxt, mpre, msuf, mret, bpre, bsuf, r_size, idx_origin, idx_width, aop2);
}
intt gtwo_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt *bpre, intt *bsuf, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtwo_t<intt>(pcxt, mpre, msuf, mret, bpre, bsuf, r_size, idx_origin, idx_width, aop2);
}
template<typename T>
__global__ void kembedding_f(T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt sz_embed, 
	intt stp, intt etable_sz, intt bw, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	intt idx;

	if(roff >= n) return;

	int_val_type(idx, &msuf[roff / sz_embed], stp);
	assert(idx < etable_sz);
	if(bw) {//msuf - input, mret - lookup table, mpre - embeded grad, roff�� mpre ����, roff�� mret������ �ƴϹǷ�
			//������(cpu������� �ƴ϶� gpu Ŀ�� �����嵵)�� mret�� ����� ��ø�ɼ��־� ��Ÿó�� �Ѵ�.
		atomicAdd(&mret[idx*sz_embed + roff % sz_embed], mpre[roff]);
	} else {//msuf - input, mret - embeded, mpre - lookup table, roff�� mret ����
		mret[roff] = mpre[idx*sz_embed + roff % sz_embed];
	}
}
template<typename T>
intt gembedding_t(T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt etable_sz, intt bw)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kembedding_f<T> << <grid, block >> > (mpre, msuf, mret, idx_origin, idx_width, sz_embed, stp, etable_sz, bw, n);
	hipDeviceSynchronize();
	cuda_error_check(-15);
	return n - idx_origin * idx_width;
}
intt gembedding_f(floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt etable_sz, intt bw)
{
	return gembedding_t<floatt>(mpre, msuf, mret, r_size, idx_origin, idx_width, sz_embed, stp, etable_sz, bw);
}
intt gembedding_f(intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt sz_embed, intt stp, intt etable_sz, intt bw)
{
	return gembedding_t<intt>(mpre, msuf, mret, r_size, idx_origin, idx_width, sz_embed, stp, etable_sz, bw);
}
template<typename T>
__global__ void konehot_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, intt n)
{
	OneVar *ovar = (OneVar *)pcxt;
	intt poff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x, depth = ovar->idxOne[5];

	if(poff >= n || *(mpre + poff) >= depth || *(mpre + poff) < 0) return;
	
	intt roff, psz = ovar->idxOne[4];

	roff = (poff / psz) * depth * psz + *(mpre + poff) * psz + poff % psz;
	*(mret + roff) = (T)*(doublet *)ovar->idxOne;
	//printf("%d %d %f\n", poff, roff, *(mret + roff));
}
template<typename T>
intt gonehot_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	konehot_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, n);
	hipDeviceSynchronize();
	cuda_error_check(-16);
	return n - idx_origin * idx_width;
}
intt gonehot_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gonehot_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
intt gonehot_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gonehot_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
template<typename T>
__global__ void kslice_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, bool bw, sytet rplus, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	SliceRank *slicer = (SliceRank *)ovar->idxOne;
	intt *srank, ndims = ovar->nrkPre;
	intt ridx[MX_DIM], i;
	T *cmem, *smem;

	srank = ovar->rankOut;
	if(bw) {
		cmem = mret;
		smem = mpre;//slice matrix
	} else {
		cmem = mpre;
		smem = mret;//slice matrix
	}
	_offset2idx(ndims, srank, roff, ridx);
	for(i = 0;i < ndims; i++) {
		cmem += (slicer[i].slbase + ridx[i] * slicer[i].slsz);
	}
	if(bw) {
		if(rplus) *cmem += *(smem + roff);
		else *cmem = *(smem + roff);
	} else *(smem + roff) = *cmem;
}
template<typename T>
intt gslice_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kslice_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, bw, rplus, n);
	hipDeviceSynchronize();
	cuda_error_check(-17);
	return n - idx_origin * idx_width;
}
intt gslice_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	return gslice_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, bw, rplus);
}
intt gslice_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width, bool bw, sytet rplus)
{
	return gslice_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width, bw, rplus);
}
template<typename T>
__global__ void kargmax_f(void *pcxt, T *mpre, T *mret, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	intt poff, i, naxis = ovar->nrkPre, inner_sz = ovar->nrkSuf, outer_sz = ovar->nrkOut;
	T vmax;

	poff = (roff / inner_sz) * outer_sz + roff % inner_sz;
	vmax = *(mpre + poff);
	*(mret + roff) = 0;
	for(i = 0;i < naxis; i++, poff += inner_sz) {
		if(vmax < *(mpre + poff)) {
			vmax = *(mpre + poff);
			*(mret + roff) = i;
		}
	}
}
template<typename T>
intt gargmax_t(void *pcxt, T *mpre, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kargmax_f<T> << <grid, block >> > (pcxt, mpre, mret, idx_origin, idx_width, n);
	hipDeviceSynchronize();
	cuda_error_check(-18);
	return n - idx_origin * idx_width;
}
intt gargmax_f(void *pcxt, floatt *mpre, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gargmax_t<floatt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
intt gargmax_f(void *pcxt, intt *mpre, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gargmax_t<intt>(pcxt, mpre, mret, r_size, idx_origin, idx_width);
}
template<typename T>
__global__ void kvmax_f(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt bw, intt rplus, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	intt poff, i, naxis = ovar->nrkPre, inner_sz = ovar->nrkSuf, outer_sz = ovar->nrkOut;

	if(bw) {//mpre�� arg max index map, msuf�� ������ ����, �Ѵ� poff(��� �ɼ�)���� ����, mret�� �ҽ�, roff�� Ǯ�� �ҽ� �ɼ�
		poff = (roff / outer_sz) * inner_sz + roff % inner_sz;//�ɼ� ���
		if(*(mpre + poff) == (roff % outer_sz) / inner_sz) *(mret + roff) = *(msuf + poff);
		else *(mret + roff) = 0;
	} else {//mpre�� �ҽ�, msuf�� arg max index map, mret�� max ���, roff�� Ǯ�� ��� �ɼ�, poff�� Ǯ�� �ҽ� Ȯ�� �ɼ�
		poff = (roff / inner_sz) * outer_sz + roff % inner_sz;//�ɼ� Ȯ��
		*(mret + roff) = *(mpre + poff);
		*(msuf + roff) = 0;
		for(i = 0; i < naxis; i++, poff += inner_sz) {
			if(*(mret + roff) < *(mpre + poff)) {
				*(mret + roff) = *(mpre + poff);
				*(msuf + roff) = i;
			}
		}
	}
}
template<typename T>
intt gvmax_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt bw, sytet rplus)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kvmax_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, bw, rplus, n);
	hipDeviceSynchronize();
	cuda_error_check(-27);
	return n - idx_origin * idx_width;
}
intt gvmax_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt bw, sytet rplus)
{
	return gvmax_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, bw, rplus);
}
intt gvmax_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt bw, sytet rplus)
{
	return gvmax_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, bw, rplus);
}

template<typename T>
__global__ void kequal_f(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;
	bool eq = ovar->idxOne[0], cscalr = ovar->idxOne[1];
	T csv = *(T *)&ovar->idxOne[2];

	if(cscalr) {
		if(eq) {
			if(mpre[roff] == csv) mret[roff] = (T)1;
			else mret[roff] = (T)0;
		} else {
			if(mpre[roff] == csv) mret[roff] = (T)0;
			else mret[roff] = (T)1;
		}
	} else {
		if(eq) {
			if(mpre[roff] == msuf[roff]) mret[roff] = (T)1;
			else mret[roff] = (T)0;
		} else {
			if(mpre[roff] == msuf[roff]) mret[roff] = (T)0;
			else mret[roff] = (T)1;
		}
	}
}
template<typename T>
intt gequal_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	kequal_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, n);
	hipDeviceSynchronize();
	cuda_error_check(-19);
	return n - idx_origin * idx_width;
}
intt gequal_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gequal_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width);
}
intt gequal_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width)
{
	return gequal_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width);
}
template<typename T>
__global__ void ktype1_f(void *pcxt, T *mpre, T *msuf, T *mret, intt idx_origin, intt idx_width, intt aop2, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	
	if(roff >= n) return;

	OneVar *ovar = (OneVar *)pcxt;

	switch(aop2) {
	case TYPE1_CLIP:
	{
		doublet low = *(doublet *)&ovar->idxOne[0], high = *(doublet *)&ovar->idxOne[2];
		if(mpre[roff] < low) mret[roff] = low;
		else if(mpre[roff] > high) mret[roff] = high;
		else mret[roff] = mpre[roff];
	}
		break;
	case DIAGO_MUL:
	{
		intt dimen = ovar->idxOne[0];
		intt d = roff / dimen, r = roff % dimen;
		intt poff = d * dimen * dimen + r * dimen + r;
		mret[roff] = mpre[poff] * msuf[poff];
	}
		break;
	case DIAGO_FILL:
	{
		intt dimen = ovar->idxOne[0];
		intt d = roff / dimen, r = roff % dimen;
		intt poff = d * dimen * dimen + r * dimen + r;
		mret[poff] = mpre[roff];
	}
		break;
	}
}
template<typename T>
intt gtype1_t(void *pcxt, T *mpre, T *msuf, T *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	ktype1_f<T> << <grid, block >> > (pcxt, mpre, msuf, mret, idx_origin, idx_width, aop2, n);
	hipDeviceSynchronize();
	cuda_error_check(-20);
	return (n - idx_origin * idx_width) * (aop2 == DIAGO_FILL ? -1 : 1);
}
intt gtype1_f(void *pcxt, floatt *mpre, floatt *msuf, floatt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtype1_t<floatt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2);
}
intt gtype1_f(void *pcxt, intt *mpre, intt *msuf, intt *mret, intt r_size, intt idx_origin, intt idx_width, intt aop2)
{
	return gtype1_t<intt>(pcxt, mpre, msuf, mret, r_size, idx_origin, idx_width, aop2);
}
#include <hiprand/hiprand_kernel.h>
template<typename T>
__global__ void knormal_t(void *pcxt, T *mpre, hiprandState *cust, intt idx_origin, intt idx_width, intt n)
{
	OneVar *ovar = (OneVar *)pcxt;
	doublet a = *(doublet *)&ovar->idxOne[2];//variance
	intt soff = blockIdx.x*blockDim.x + threadIdx.x;
	intt roff = idx_width * idx_origin + soff;

	if(roff >= n) return;

	mpre[roff] = hiprand_normal(&cust[soff]) * a;
}
template<typename T>
__global__ void kuniform_t(void *pcxt, T *mpre, hiprandState *cust, intt idx_origin, intt idx_width, intt n)
{
	OneVar *ovar = (OneVar *)pcxt;
	doublet a = *(doublet *)&ovar->idxOne[2];//variance
	intt soff = blockIdx.x*blockDim.x + threadIdx.x;
	intt roff = idx_width * idx_origin + soff;

	if(roff >= n) return;

	mpre[roff] = hiprand_uniform(&cust[soff]) * a;
}
__global__ void seed_random(hiprandState *cust, uintt seed, intt n)
{
	intt roff = blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	hiprand_init(seed, roff, 0, &cust[roff]);
}
template<typename T>
intt grandom_t(void *pcxt, T *mpre, intt r_size, intt idx_origin, intt idx_width, intt aop2, intt seed)
{
	dim3 block(WIDTH_BLOCK2(SMALL_BLOCK));
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	hiprandState *cust;
	intt cu_len = n - idx_origin * idx_width;
	hipMalloc((void **)&cust, cu_len * sizeof(hiprandState));

	seed_random << <grid, block >> > (cust, seed < 0 ? time(nullx) : seed, cu_len);

	switch(aop2) {
	case RAND_T_N:
		knormal_t<T> << <grid, block >> > (pcxt, mpre, cust, idx_origin, idx_width, n);
		break;
	case RAND_T_U:
		kuniform_t<T> << <grid, block >> > (pcxt, mpre, cust, idx_origin, idx_width, n);
		break;
	case RAND_T_L:
		break;
	case RAND_T_P:
		break;
	}
	hipFree(cust);//����� ũ�� ���� �߻��Ѵ�.
	cuda_error_check(-21);
	hipDeviceSynchronize();
	return cu_len;
}
intt grandom_f(void *pcxt, floatt *mpre, intt r_size, intt idx_origin, intt idx_width, intt aop2, intt seed)
{
	return grandom_t<floatt>(pcxt, mpre, r_size, idx_origin, idx_width, aop2, seed);
}
intt grandom_f(void *pcxt, intt *mpre, intt r_size, intt idx_origin, intt idx_width, intt aop2, intt seed)
{
	return grandom_t<intt>(pcxt, mpre, r_size, idx_origin, idx_width, aop2, seed);
}

template<typename T>
__global__ void kln_sum_f(T *mi, T *sum, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	atomicAdd(sum + (roff / dsz), mi[roff]);
	//printf("111 %f %f %d\n", sum + (roff / dsz), mi[roff], roff);
}
template<typename T>
__global__ void kln_mean_f(T *mean, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mean[roff] /= dsz;
	//printf("222 %f\n", mean[roff]);
}
template<typename T>
__global__ void kln_var_f(T *mi, T *mean, T *md, T *mv, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;
	T v;

	if(roff >= n) return;

	intt q = roff / dsz;
	md[roff] = v = (mi[roff] - mean[q]);//��� ����
	atomicAdd(&mv[q], v * v);//�л�, ������� ���� ��
	//printf("333 %f %f\n", md[roff], mv[q]);
}
template<typename T>
__global__ void kln_sdev_f(T *mv, intt dsz, intt idx_origin, intt idx_width, intt n, bool db)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mv[roff] = 1.0 / std::sqrt((db ? (doublet)mv[roff] : (floatt)mv[roff]) / dsz + 1e-9);//ǥ������ ����
}
template<typename T>
__global__ void kln_zval_f(T *md, T *mv, T *mz, T *ga, T *be, T *mr, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mz[roff] = md[roff] * mv[roff / dsz];//�Է°� ��� ������ ǥ������ ������ ���Ͽ� ǥ�ذ� ����
	intt r = roff % dsz;
	mr[roff] = mz[roff] * ga[r] + be[r];//ǥ�ذ��� ������ ���ϰ� ��Ÿ�� ���Ͽ� ǥ�� ��°� ����.
	//printf("444 %f %f %f\n", mr[roff], mz[roff], mv[roff / dsz]);
}
template<typename T>
__global__ void kln_g_zval_f(T *mi, T *ga, T *md, T *g_mz, T *var, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	g_mz[roff] = mi[roff] * ga[roff % dsz];//���� ���谪�� �������� ���Ͽ� ������ ��� ǥ�ذ��� ���� ����
	atomicAdd(&var[roff / dsz], -0.5 * g_mz[roff] * md[roff]);//ǥ�ذ� ���迡 ������ �Է°� ��������� ���� ��
}
template<typename T>
__global__ void kln_g_var_f(T *mv, T *var, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	var[roff] *= (mv[roff] * mv[roff] * mv[roff]);//�� �տ� �Է°� ǥ������ ������ ���Ͽ� �л� ���� ����
}
template<typename T>
__global__ void kln_g_mean_sum_f(T *g_mz, T *mv, T *mean, intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt q = roff / dsz;
	atomicAdd(mean + q, -1.0 * g_mz[roff] * mv[q]);
}
template<typename T>
__global__ void kln_g_mean_f(T *mean, T *var, T *mdmean, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	mean[roff] += -2.0 * var[roff] * mdmean[roff];
}
template<typename T>
__global__ void kln_g_i_f(T *g_mz, T *mi, T *md, T *mv, T *mz, T *var, T *mean, T *mr, T *g_gm, T *g_be, 
	intt dsz, intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt q = roff / dsz;
	mr[roff] = g_mz[roff] * mv[q] + (2.0 / dsz) * var[q] * md[roff] + (1.0 / dsz) * mean[q];
	intt r = roff % dsz;
	atomicAdd(&g_gm[r], mi[roff] * mz[roff]);//���� ���谪�� ������ ��� ǥ�ذ��� ���� ���� ���Ͽ� ���� ���� ����
	atomicAdd(&g_be[r], mi[roff]);//���� ���谪�� ���Ͽ� ��Ÿ ���� ����
}
template<typename T>
intt glayer_norm_t(void *pcxt, T *mi, T *mr, T *md, T *mz, T *mv, T *mean, T *g_mz, T *var, T *mdmean,
	T *ga, T *be, T *g_gm, T *g_be, intt r_size, intt idx_origin, intt idx_width, intt dsz, bool bw, bool db)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);
	dim3 grid2((idx_width / dsz + block.x - 1) / block.x);
	intt n2 = n / dsz;

	if(bw) {
		kln_g_zval_f<T> << <grid, block >> > (mi, ga, md, g_mz, var, dsz, idx_origin, idx_width, n);
		kln_g_var_f<T> << <grid2, block >> > (mv, var, dsz, idx_origin, idx_width, n2);
		kln_sum_f<T> << <grid, block >> > (md, mdmean, dsz, idx_origin, idx_width, n);//md sum
		kln_mean_f<T> << <grid2, block >> > (mdmean, dsz, idx_origin, idx_width, n2);//md mean
		kln_g_mean_sum_f<T> << <grid, block >> > (g_mz, mv, mean, dsz, idx_origin, idx_width, n);
		kln_g_mean_f<T> << <grid2, block >> > (mean, var, mdmean, idx_origin, idx_width, n2);
		kln_g_i_f<T> << <grid, block >> > (g_mz, mi, md, mv, mz, var, mean, mr, g_gm, g_be, dsz, idx_origin, idx_width, n);
	} else {
		kln_sum_f<T> << <grid, block >> > (mi, mean, dsz, idx_origin, idx_width, n);
		kln_mean_f<T> << <grid2, block >> > (mean, dsz, idx_origin, idx_width, n2);
		kln_var_f<T> << <grid, block >> > (mi, mean, md, mv, dsz, idx_origin, idx_width, n);
		kln_sdev_f<T> << <grid2, block >> > (mv, dsz, idx_origin, idx_width, n2, db);
		kln_zval_f<T> << <grid, block >> > (md, mv, mz, ga, be, mr, dsz, idx_origin, idx_width, n);
	}
	hipDeviceSynchronize();
	cuda_error_check(-22);
	return n - idx_origin * idx_width;
}
intt glayer_norm_f(void *pcxt, floatt *mi, floatt *mr, floatt *md, floatt *mz, floatt *mv, floatt *mean, 
	floatt *g_mz, floatt *var, floatt *mdmean, floatt *ga, floatt *be, floatt *g_gm, floatt *g_be, 
	intt r_size, intt idx_origin, intt idx_width, intt dsz, bool bw, bool db)
{
	return glayer_norm_t<floatt>(pcxt, mi, mr, md, mz, mv, mean, g_mz, var, mdmean, ga, be, g_gm, g_be,
		r_size, idx_origin, idx_width, dsz, bw, db);
}
intt glayer_norm_f(void *pcxt, intt *mi, intt *mr, intt *md, intt *mz, intt *mv, intt *mean,
	intt *g_mz, intt *var, intt *mdmean, intt *ga, intt *be, intt *g_gm, intt *g_be,
	intt r_size, intt idx_origin, intt idx_width, intt dsz, bool bw, bool db)
{
	return glayer_norm_t<intt>(pcxt, mi, mr, md, mz, mv, mean, g_mz, var, mdmean, ga, be, g_gm, g_be,
		r_size, idx_origin, idx_width, dsz, bw, db);
}
/*
#define CACHE_F_N	9 //aa)�� �ε���
template<typename DT, typename OT>
__global__ void kmatmul_sm_f(void *pcxt, DT mpre[], DT msuf[], DT mret[], intt M, intt K, intt N, intt T, bool rplus,
	intt idx_origin, intt idx_width, intt n, intt n_batch_capable)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	__shared__ DT cache_mat[SM_SIZE];
	OT *base_batch, *pbase, *sbase, *n_batch_sm, *cache_sz, *psz, *ssz, *rsz, *pre_csz, *rest;
	base_batch = (OT *)&cache_mat[0];
	pbase = (OT *)&cache_mat[1];
	sbase = (OT *)&cache_mat[2];
	n_batch_sm = (OT *)&cache_mat[3];
	cache_sz = (OT *)&cache_mat[4];
	psz = (OT *)&cache_mat[5];
	ssz = (OT *)&cache_mat[6];
	rsz = (OT *)&cache_mat[7];
	rest = (OT *)&cache_mat[8];
	pre_csz = (OT *)&cache_mat[CACHE_F_N];//aa.
	DT *p_pre = &cache_mat[CACHE_F_N +1], *p_suf;

	if(blockIdx.x == 0) {
		*psz = M * K;
		*ssz = K * N;
		*rsz = M * N;//��ġ�� ��� K, N�� ��ġ�� ���� �ε����̰� suf��Ʈ������ ��ġ���� ���� ������ ��Ʈ����(N, K)
		intt obsz = *psz + *ssz;//�Ѱ� ��ġ ó���� �ҿ�Ǵ� pre�� suf�� ���� ������
		*n_batch_sm = n_batch_capable;//ĳ���޸𸮷� ó���ϴ� ��ġ ����
		*cache_sz = *n_batch_sm * obsz;//�ѹ��� ĳ�� �ε��ϴ� pre�� suf�� ���� ��ġ ������
		*pre_csz = *psz * *n_batch_sm;//pre��Ʈ ��ġ ������
		p_suf = p_pre + *pre_csz;//suf��Ʈ ĳ�� ������ ��� ������
		*base_batch = 0;
	}
	intt i;
	if(blockIdx.x == 0 || (idx_width * idx_origin + (blockIdx.x +1)*blockDim.x) / *rsz >= *base_batch + *n_batch_sm) {
		*base_batch = (idx_width * idx_origin + blockIdx.x*blockDim.x) / *rsz;//�̹� �׸����� ù��° �����尡 �������� ��ġ�ε���
		*pbase = *base_batch * *psz;//�̹� �׸����� ù��° ��ġ ���� pre �ɼ�
		*sbase = *base_batch * *ssz - *pre_csz;//�̹� �׸����� ù��° ��ġ ���� suf �ɼ�
		for(i = 0;; i += blockDim.x) {//ĳ�� ���� ��ġ ���� �׸��� ���� ��ġ ������ ũ�� �� ��ŭ �� �׸��� ���࿡�� 
			intt off = i + threadIdx.x;//�� ������ �׸��� ������ �ݺ��Ͽ� �Ѳ����� ĳ���ϰ� �� �� �׸���� �̸� �����Ѵ�.
			if(off >= *cache_sz) break;//ĳ�� ����� ��� ������� ��ŵ
			*(p_pre + off) = (off < *pre_csz ? mpre[*pbase + off] : msuf[*sbase + off]);//���ڴ� p_suf�� ����ȴ�.
		}
		__syncthreads();
	}
	roff -= (*base_batch * *rsz);
	
	intt j = (roff / *rsz) * *ssz;//��ġ�ε���(roff / rsz) * suf ��Ʈ���� ������(ssz) => suf ��ġ �ɼ�
	DT sum = 0;

	switch(T) {
	case 0://AB
		i = (roff / N) * K;
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
		for(intt k = i + K;i < k; i++, j += N) sum += p_pre[i] * p_suf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 1://A^B
		i = (roff / *rsz) * *psz + ((roff % *rsz) / N);
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
		for(intt k = i + *psz;i < k; i += M, j += N) sum += p_pre[i] * p_suf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 2://AB^
		i = (roff / N) * K;
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
		for(intt k = i + K;i < k; i++, j++) sum += p_pre[i] * p_suf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 3://A^B^
		i = (roff / *rsz) * *psz + ((roff % *rsz) / N);
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
		for(intt k = i + *psz;i < k; i += M, j++) sum += p_pre[i] * p_suf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	}
}
*/
#define OFV_CACHE	1
#ifdef OFV_CACHE
#define CACHE_F_N	12 //aa)�� �ε���
#else //���� ���� ���� �޸� ���
#define CACHE_F_N	2 //aa)�� �ε���
#endif
template<typename DT, typename OT>
__global__ void kmatmul_sm_f(void *pcxt, DT mpre[], DT msuf[], DT mret[], intt M, intt K, intt N, intt T, bool rplus,
	intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	__shared__ DT cache_mat[SM_SIZE];

	OT *psz, *ssz, *rsz;
#ifdef OFV_CACHE
	OT *base, *beg, *end, *n_batch_cache, *cache_sz, *pre_csz, *obsz, *pbase, *sbase, *sbase2;
	psz = (OT *)&cache_mat[0];
	ssz = (OT *)&cache_mat[1];
	rsz = (OT *)&cache_mat[2];
	base = (OT *)&cache_mat[3];
	beg = (OT *)&cache_mat[4];
	end = (OT *)&cache_mat[5];
	n_batch_cache = (OT *)&cache_mat[6];
	cache_sz = (OT *)&cache_mat[7];
	pre_csz = (OT *)&cache_mat[8];
	obsz = (OT *)&cache_mat[9];
	pbase = (OT *)&cache_mat[10];
	sbase = (OT *)&cache_mat[11];
	sbase2 = (OT *)&cache_mat[CACHE_F_N];//aa.
	DT *p_pre = &cache_mat[CACHE_F_N + 1];

	*psz = M * K;
	*ssz = K * N;
	*rsz = M * N;//��ġ�� ��� K, N�� ��ġ�� ���� �ε����̰� suf��Ʈ������ ��ġ���� ���� ������ ��Ʈ����(N, K)
	*base = idx_width * idx_origin + blockIdx.x*blockDim.x;//�̹� �׸��� ������ ���̽� �ɼ�
	*beg = *base / *rsz;//�̹� �׸����� ù��° �����尡 �������� ��ġ�ε���
	*end = n <= *base + blockDim.x ? (n - 1) / *rsz : (*base + blockDim.x - 1) / *rsz;
	*n_batch_cache = *end - *beg + 1;//�̹� �׸��忡 �ε��� ��ġ ����
	*obsz = *psz + *ssz;//�Ѱ� ��ġ ó���� �ҿ�Ǵ� pre�� suf�� ���� ������
	*cache_sz = *n_batch_cache * *obsz;//ĳ�� �ε��ϴ� pre�� suf�� ���� ��ġ ������
	*pre_csz = *n_batch_cache * *psz;//pre��Ʈ ��ġ ������
	DT *p_suf = p_pre + *pre_csz;//suf��Ʈ ĳ�� ������ ��� ������
	*pbase = *beg * *psz;//�̹� �׸����� ù��° ��ġ ���� pre �ɼ�
	*sbase = *beg * *ssz;//�̹� �׸����� ù��° ��ġ ���� suf �ɼ�
	*sbase2 = *sbase - *pre_csz;
	//printf("blockIdx: %d psz: %d ssz: %d rsz: %d\n", blockIdx.x, *psz, *ssz, *rsz);
	//printf("base: %d beg: %d end: %d n_batch_cache: %d cache_sz: %d pre_csz: %d pbase: %d sbase: %d\n", *base, *beg, *end, *n_batch_cache, *cache_sz, *pre_csz, *pbase, *sbase);
	intt i;
	for(i = 0;; i += blockDim.x) {//�׸��� ���� ������ ������ �̹��� �ε��� ��ġ ����� ���ڶ��������Ƿ� �ݺ��Ͽ� �ε� 
		intt off = i + threadIdx.x;
		if(off >= *cache_sz) break;//ĳ�� ����� ��� ������� ��ŵ
		*(p_pre + off) = (off < *pre_csz ? mpre[*pbase + off] : msuf[*sbase2 + off]);//���ڴ� p_suf�� ����ȴ�.
	}
	__syncthreads();
	/*if(threadIdx.x == 0) {
		for(i = 0;i < *pre_csz; i++) printf("%f\n", p_pre[i]);
		printf("\n");
		for(i = 0;i < *cache_sz - *pre_csz; i++) printf("%f\n", p_suf[i]);
	}
	__syncthreads();*/
#else //���� ���� ���� �޸� ���
	psz = (OT *)&cache_mat[0];
	ssz = (OT *)&cache_mat[1];
	rsz = (OT *)&cache_mat[CACHE_F_N];//aa.
	DT *p_pre = &cache_mat[CACHE_F_N + 1];

	*psz = M * K;
	*ssz = K * N;
	*rsz = M * N;//��ġ�� ��� K, N�� ��ġ�� ���� �ε����̰� suf��Ʈ������ ��ġ���� ���� ������ ��Ʈ����(N, K)
	intt base = idx_width * idx_origin + blockIdx.x*blockDim.x;//�̹� �׸��� ������ ���̽� �ɼ�
	intt beg = base / *rsz;//�̹� �׸����� ù��° �����尡 �������� ��ġ�ε���
	intt end = n <= base + blockDim.x ? (n - 1) / *rsz : (base + blockDim.x - 1) / *rsz;
	intt n_batch_cache = end - beg + 1;//�̹� �׸��忡 �ε��� ��ġ ����
	intt obsz = *psz + *ssz;//�Ѱ� ��ġ ó���� �ҿ�Ǵ� pre�� suf�� ���� ������
	intt cache_sz = n_batch_cache * obsz;//ĳ�� �ε��ϴ� pre�� suf�� ���� ��ġ ������
	intt pre_csz = n_batch_cache * *psz;//pre��Ʈ ��ġ ������
	DT *p_suf = p_pre + pre_csz;//suf��Ʈ ĳ�� ������ ��� ������
	intt pbase = beg * *psz;//�̹� �׸����� ù��° ��ġ ���� pre �ɼ�
	intt sbase = beg * *ssz;//�̹� �׸����� ù��° ��ġ ���� suf �ɼ�
	intt sbase2 = sbase - pre_csz;
	intt i;
	for(i = 0;; i += blockDim.x) {//�׸��� ���� ������ ������ �̹��� �ε��� ��ġ ����� ���ڶ��������Ƿ� �ݺ��Ͽ� �ε� 
		intt off = i + threadIdx.x;
		if(off >= cache_sz) break;//ĳ�� ����� ��� ������� ��ŵ
		*(p_pre + off) = (off < pre_csz ? mpre[pbase + off] : msuf[sbase2 + off]);//���ڴ� p_suf�� ����ȴ�.
	}
	__syncthreads();
#endif
	intt j = (roff / *rsz) * *ssz;//��ġ�ε���(roff / rsz) * suf ��Ʈ���� ������(ssz) => suf ��ġ �ɼ�
	DT sum = 0;

	switch(T) {
	case 0://AB
		i = (roff / N) * K;
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
#ifdef OFV_CACHE
		i -= *pbase; j -= *sbase;
#else
		i -= pbase; j -= sbase;
#endif
		for(intt k = i + K;i < k; i++, j += N) sum += p_pre[i] * p_suf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 1://A^B
		i = (roff / *rsz) * *psz + ((roff % *rsz) / N);
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
#ifdef OFV_CACHE
		i -= *pbase; j -= *sbase;
#else
		i -= pbase; j -= sbase;
#endif
		for(intt k = i + *psz;i < k; i += M, j += N) sum += p_pre[i] * p_suf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 2://AB^
		i = (roff / N) * K;
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
#ifdef OFV_CACHE
		i -= *pbase; j -= *sbase;
#else
		i -= pbase; j -= sbase;
#endif
		for(intt k = i + K;i < k; i++, j++) sum += p_pre[i] * p_suf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 3://A^B^
		i = (roff / *rsz) * *psz + ((roff % *rsz) / N);
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
#ifdef OFV_CACHE
		i -= *pbase; j -= *sbase;
#else
		i -= pbase; j -= sbase;
#endif
		for(intt k = i + *psz;i < k; i += M, j++) sum += p_pre[i] * p_suf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	}
}

template<typename DT>
__global__ void kmatmul_f(void *pcxt, DT mpre[], DT msuf[], DT mret[], intt M, intt K, intt N, intt T, bool rplus,
	intt idx_origin, intt idx_width, intt n)
{
	intt roff = idx_width * idx_origin + blockIdx.x*blockDim.x + threadIdx.x;

	if(roff >= n) return;

	intt i, sz = M * N;//��ġ�� ��� K, N�� ��ġ�� ���� �ε����̰� suf��Ʈ������ ��ġ���� ���� ������ ��Ʈ����(N, K)
	intt j = (roff / sz) * (K * N);//��ġ�ε���(roff / sz) * suf ��Ʈ���� ������(K * N) => suf ��ġ �ɼ�
	DT sum = 0;

	switch(T) {
	case 0://AB
		i = (roff / N) * K;
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
		for(intt k = i + K;i < k; i++, j += N) sum += mpre[i] * msuf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 1://A^B
		i = (roff / sz) * M * K + ((roff % sz) / N);
		j += roff % N;//suf ��ġ �ɼ� + ret/suf�÷� �ε���(roff % N) //ret�� suf�� �÷��ε����� ����
		for(intt k = i + K * M;i < k; i += M, j += N) sum += mpre[i] * msuf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 2://AB^
		i = (roff / N) * K;
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
		for(intt k = i + K;i < k; i++, j++) sum += mpre[i] * msuf[j];
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	case 3://A^B^
		i = (roff / sz) * M * K + ((roff % sz) / N);
		j += (roff % N) * K;//suf ��ġ �ɼ� + ret�÷��ε���/suf�ο��ε���(roff % N) * suf�÷�������(K) //ret�� �÷��ε����� suf�� �ο��ε���
		for(intt k = i + K * M;i < k; i += M, j++) sum += mpre[i] * msuf[j];//ret�Ѱ� ���Ұ� �ջ�
		if(rplus) mret[roff] += sum;
		else mret[roff] = sum;
		break;
	}
}
template<typename DT, typename OT>
intt gmatmul_t(void *pcxt, DT mpre[], DT msuf[], DT mret[], intt r_size, intt M, intt K, intt N, intt T, bool rplus,
				intt idx_origin, intt idx_width)
{
	dim3 block(WIDTH_BLOCK);
	dim3 grid((idx_width + block.x - 1) / block.x);
	intt n = (r_size > (idx_origin + 1) * idx_width ? (idx_origin + 1) * idx_width : r_size);

	intt obsz = M * K + K * N;//�Ѱ� ��ġ ó���� �ҿ�Ǵ� pre�� suf�� ���� ������
	intt n_ret_batch_block = (block.x + (M * N) - 1) / (M * N);//���� ret��ġ outer ����
	intt n_capable_batch_sm = ((SM_SIZE - sizeof(DT) * CACHE_F_N) / (n_ret_batch_block * obsz)) * n_ret_batch_block;//ĳ���޸𸮷� ó���ϴ� ��ġ inner ����

	//if(idx_origin != 2) return n - idx_origin * idx_width;
	//n_capable_batch_sm = 0;
	/*���� ������üũ�� �����ִ� �� ����, n_capable_batch_sm����� �߸��Ǿ� Ư����� �����޸𸮷�
	�����ϸ� �����޸� ��� ������ �ȴ�. ���߿� n_capable_batch_sm ��� ���� ������ ���� 
	if(n_capable_batch_sm) kmatmul_sm_f<DT, OT> << <grid, block >> > (pcxt, mpre, msuf, mret, M, K, N, T, rplus, idx_origin, idx_width, n);
	else*/ kmatmul_f<DT> << <grid, block >> > (pcxt, mpre, msuf, mret, M, K, N, T, rplus, idx_origin, idx_width, n);

	hipDeviceSynchronize();
	cuda_error_check(-25);
	return n - idx_origin * idx_width;
}
intt gmatmul_f(void *pcxt, floatt mpre[], floatt msuf[], floatt mret[], intt r_size, intt M, intt K, intt N, intt T, bool rplus,
	intt idx_origin, intt idx_width)
{
	return gmatmul_t<floatt, intt>(pcxt, mpre, msuf, mret, r_size, M, K, N, T, rplus, idx_origin, idx_width);
}
intt gmatmul_f(void *pcxt, intt mpre[], intt msuf[], intt mret[], intt r_size, intt M, intt K, intt N, intt T, bool rplus,
	intt idx_origin, intt idx_width)
{
	return gmatmul_t<intt, intt>(pcxt, mpre, msuf, mret, r_size, M, K, N, T, rplus, idx_origin, idx_width);
}
__global__ void DoKernel(float* data, float value)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(value != 0)
		data[y * 1280 + x] = value;
}

struct mgt {
	float *o, *a, *b, *bb;
};
void multi_gpu_test(void *p)
{
	struct mgt *mp = (struct mgt *)p;
	size_t sizeGpuMemFree;
	size_t sizeGpuMemTotal;
	hipError_t error;

	dim3 grid(1280 / 16, 720 / 16);
	dim3 block(16, 16);

	LARGE_INTEGER start, end, freq;
	QueryPerformanceFrequency(&freq);

	float* origin = mp->o;
	float* a = mp->a;
	float* b = mp->b;
	float* aa;
	float* bb = mp->bb;

	QueryPerformanceCounter(&start);

	//hipHostMalloc(&origin, sizeof(float) * 16 * 1280 * 720);
	//ZeroMemory(origin, sizeof(float) * 16 * 1280 * 720);
	//origin = (float *)malloc(sizeof(float) * 16 * 1280 * 720);
	memset(origin, 0x00, sizeof(float) * 16 * 1280 * 720);

	hipSetDevice(1);

	//error = hipMalloc(&a, sizeof(float) * 16 * 1280 * 720);
	//error = hipHostMalloc(&aa, sizeof(float) * 16 * 1280 * 720);
	aa = (float *)malloc(sizeof(float) * 16 * 1280 * 720);
	error = hipMemset(a, 0, sizeof(float) * 16 * 1280 * 720);

	error = hipMemGetInfo(&sizeGpuMemFree, &sizeGpuMemTotal);
	printf("GPU 1 memory: %I64u / %I64u\n", sizeGpuMemFree, sizeGpuMemTotal);

	hipMemcpy(a, origin, sizeof(float) * 16 * 1280 * 720, hipMemcpyHostToDevice);
	DoKernel << <grid, block >> > (a, 1);
	hipDeviceSynchronize();
	hipMemcpy(aa, a, sizeof(float) * 16 * 1280 * 720, hipMemcpyDeviceToHost);

	//hipSetDevice(1);

	//error = hipMalloc(&b, sizeof(float) * 16 * 1280 * 720);
	//error = hipHostMalloc(&bb, sizeof(float) * 16 * 1280 * 720);
	//bb = (float *)malloc(sizeof(float) * 16 * 1280 * 720);

	error = hipMemGetInfo(&sizeGpuMemFree, &sizeGpuMemTotal);
	printf("GPU 2 memory: %I64u / %I64u\n", sizeGpuMemFree, sizeGpuMemTotal);
	// ����1
	error = hipMemcpy(b, a, sizeof(float) * 16 * 1280 * 720, hipMemcpyDeviceToDevice);
	cuda_error_check(-1);
	// ����2
	//error = hipMemcpy(b, origin, sizeof(float) * 16 * 1280 * 720, hipMemcpyHostToDevice);
	DoKernel << <grid, block >> > (b, 0);
	hipDeviceSynchronize();
	hipMemcpy(bb, b, sizeof(float) * 16 * 1280 * 720, hipMemcpyDeviceToHost);
	cuda_error_check(-2);

	hipHostFree(origin);
	hipHostFree(aa);
	hipHostFree(bb);

	hipSetDevice(0);
	hipFree(a);

	hipSetDevice(1);
	hipFree(b);

	QueryPerformanceCounter(&end);
	printf("aaa %f\n", (float)(end.QuadPart - start.QuadPart) / freq.QuadPart);
}
/*
__global__ void DoKernel(float1* data, float value)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(value != 0)
		data[y * 1280 + x].x = value;
}
void multi_gpu_test(void)
{
	size_t sizeGpuMemFree;
	size_t sizeGpuMemTotal;
	hipError_t error;

	dim3 grid(1280 / 16, 720 / 16);
	dim3 block(16, 16);

	LARGE_INTEGER start, end, freq;
	QueryPerformanceFrequency(&freq);

	float1* origin;
	float1* a;
	float1* b;
	float1* aa;
	float1* bb;

	QueryPerformanceCounter(&start);

	//hipHostMalloc(&origin, sizeof(float1) * 16 * 1280 * 720);
	//ZeroMemory(origin, sizeof(float1) * 16 * 1280 * 720);
	origin = (float1 *)malloc(sizeof(float1) * 16 * 1280 * 720);
	memset(origin, 0x00, sizeof(float1) * 16 * 1280 * 720);

	hipSetDevice(1);

	error = hipMalloc(&a, sizeof(float1) * 16 * 1280 * 720);
	//error = hipHostMalloc(&aa, sizeof(float1) * 16 * 1280 * 720);
	aa = (float1 *)malloc(sizeof(float1) * 16 * 1280 * 720);
	error = hipMemset(a, 0, sizeof(float1) * 16 * 1280 * 720);

	error = hipMemGetInfo(&sizeGpuMemFree, &sizeGpuMemTotal);
	printf("GPU 1 memory: %I64u / %I64u\n", sizeGpuMemFree, sizeGpuMemTotal);

	hipMemcpy(a, origin, sizeof(float1) * 16 * 1280 * 720, hipMemcpyHostToDevice);
	DoKernel << <grid, block >> > (a, 1);
	hipDeviceSynchronize();
	hipMemcpy(aa, a, sizeof(float1) * 16 * 1280 * 720, hipMemcpyDeviceToHost);

	//hipSetDevice(1);

	error = hipMalloc(&b, sizeof(float1) * 16 * 1280 * 720);
	//error = hipHostMalloc(&bb, sizeof(float1) * 16 * 1280 * 720);
	bb = (float1 *)malloc(sizeof(float1) * 16 * 1280 * 720);

	error = hipMemGetInfo(&sizeGpuMemFree, &sizeGpuMemTotal);
	printf("GPU 2 memory: %I64u / %I64u\n", sizeGpuMemFree, sizeGpuMemTotal);
	// ����1
	error = hipMemcpy(b, a, sizeof(float1) * 16 * 1280 * 720, hipMemcpyDeviceToDevice);
	cuda_error_check(-1);
	// ����2
	//error = hipMemcpy(b, origin, sizeof(float1) * 16 * 1280 * 720, hipMemcpyHostToDevice);
	DoKernel << <grid, block >> > (b, 0);
	hipDeviceSynchronize();
	hipMemcpy(bb, b, sizeof(float1) * 16 * 1280 * 720, hipMemcpyDeviceToHost);
	cuda_error_check(-2);

	hipHostFree(origin);
	hipHostFree(aa);
	hipHostFree(bb);

	hipSetDevice(0);
	hipFree(a);

	hipSetDevice(1);
	hipFree(b);

	QueryPerformanceCounter(&end);
	printf("aaa %f\n", (float)(end.QuadPart - start.QuadPart) / freq.QuadPart);
}
*/